#include "hip/hip_runtime.h"
/*
 *
 * Copyright 2009 Phylogenetic Likelihood Working Group
 *
 * This file is part of BEAGLE.
 *
 * Use of this source code is governed by an MIT-style
 * license that can be found in the LICENSE file or at
 * https://opensource.org/licenses/MIT.
 *
 * @author Marc Suchard
 * @author Daniel Ayres
 * @author Andrew Rambaut
 */

#define STATE_COUNT 4

#ifdef CUDA
    #include "libhmsbeagle/GPU/GPUImplDefs.h"
    #include "libhmsbeagle/GPU/kernels/kernelsAll.cu" // This file includes the non-state-count specific kernels
    extern "C" {
#endif

#define multBy4(x)  ((x) << 2)
#define multBy16(x) ((x) << 4)
#define LIKE_PATTERN_BLOCK_SIZE PATTERN_BLOCK_SIZE

///////////////////////////////////////////////////////////////////////////////
// kernel macros CPU

#define DETERMINE_INDICES_4_CPU()\
    int patIdx = KW_LOCAL_ID_0;\
    int matrix = KW_GROUP_ID_1;\
    int pattern = KW_GROUP_ID_0 * KW_LOCAL_SIZE_0 + patIdx;\
    int deltaPartialsByState = pattern * PADDED_STATE_COUNT;\
    int deltaPartialsByMatrix = matrix * PADDED_STATE_COUNT * endPattern;\
    int deltaMatrix = matrix * PADDED_STATE_COUNT * PADDED_STATE_COUNT;\
    int deltaPartials = deltaPartialsByMatrix + deltaPartialsByState;

#define DETERMINE_INDICES_4_MULTI_1_CPU()\
    int opIndexPtr = (gridStartOp + KW_GROUP_ID_0) * 8;\
    int startPat   = ptrOffsets[opIndexPtr    ];\
    int endPattern = ptrOffsets[opIndexPtr + 1];\
    int patIdx = KW_LOCAL_ID_0;\
    int pattern = startPat + patIdx;

#define DETERMINE_INDICES_4_MULTI_2_CPU()\
    int matrix = KW_GROUP_ID_1;\
    int deltaPartialsByState = pattern * PADDED_STATE_COUNT;\
    int deltaPartialsByMatrix = matrix * PADDED_STATE_COUNT * totalPatterns;\
    int deltaMatrix = matrix * PADDED_STATE_COUNT * PADDED_STATE_COUNT;\
    int deltaPartials = deltaPartialsByMatrix + deltaPartialsByState;

#define DETERMINE_INDICES_4_MULTI_3_CPU()\
          KW_GLOBAL_VAR REAL* KW_RESTRICT partials3 =  partials + ptrOffsets[opIndexPtr + 4];\
    const KW_GLOBAL_VAR REAL* KW_RESTRICT matrices1 =  matrices + ptrOffsets[opIndexPtr + 5];\
    const KW_GLOBAL_VAR REAL* KW_RESTRICT matrices2 =  matrices + ptrOffsets[opIndexPtr + 6];

#define DETERMINE_INDICES_4_PART_1_CPU()\
    int patIdx = KW_LOCAL_ID_0;\
    int pattern = startPattern + KW_GROUP_ID_0 * KW_LOCAL_SIZE_0 + patIdx;

#define DETERMINE_INDICES_4_PART_2_CPU()\
    int matrix = KW_GROUP_ID_1;\
    int deltaPartialsByState = pattern * PADDED_STATE_COUNT;\
    int deltaPartialsByMatrix = matrix * PADDED_STATE_COUNT * totalPatterns;\
    int deltaMatrix = matrix * PADDED_STATE_COUNT * PADDED_STATE_COUNT;\
    int deltaPartials = deltaPartialsByMatrix + deltaPartialsByState;

#define DETERMINE_INDICES_4_EDGEPART_1_CPU()\
    int opIndexPtr = KW_GROUP_ID_0 * 5;\
    int startPat   = ptrOffsets[opIndexPtr    ];\
    int endPattern = ptrOffsets[opIndexPtr + 1];\
    int patIdx = KW_LOCAL_ID_0;\
    int pattern = startPat + patIdx;

#define SUM_PARTIALS_PARTIALS_4_CPU()\
    REAL sum1[PADDED_STATE_COUNT];\
    REAL sum2[PADDED_STATE_COUNT];\
    const KW_GLOBAL_VAR REAL* KW_RESTRICT sMatrix1 = matrices1 + deltaMatrix;\
    const KW_GLOBAL_VAR REAL* KW_RESTRICT sMatrix2 = matrices2 + deltaMatrix;\
    const KW_GLOBAL_VAR REAL* KW_RESTRICT sPartials1 = partials1 + deltaPartials;\
    const KW_GLOBAL_VAR REAL* KW_RESTRICT sPartials2 = partials2 + deltaPartials;\
    for(int i = 0; i < PADDED_STATE_COUNT; i++) {\
        sum1[i] = sMatrix1[0 * PADDED_STATE_COUNT + i] * sPartials1[0];\
        sum2[i] = sMatrix2[0 * PADDED_STATE_COUNT + i] * sPartials2[0];\
    }\
    for (int s = 1; s < PADDED_STATE_COUNT; s++) {\
        for(int i = 0; i < PADDED_STATE_COUNT; i++) {\
            FMA(sMatrix1[s * PADDED_STATE_COUNT + i],  sPartials1[s], sum1[i]);\
            FMA(sMatrix2[s * PADDED_STATE_COUNT + i],  sPartials2[s], sum2[i]);\
        }\
    }

#define SUM_STATES_PARTIALS_4_CPU()\
    REAL sum1[PADDED_STATE_COUNT];\
    REAL sum2[PADDED_STATE_COUNT];\
    KW_GLOBAL_VAR REAL* KW_RESTRICT sMatrix1 = matrices1 + deltaMatrix;\
    KW_GLOBAL_VAR REAL* KW_RESTRICT sMatrix2 = matrices2 + deltaMatrix;\
    KW_GLOBAL_VAR REAL* KW_RESTRICT sPartials2 = partials2 + deltaPartials;\
    int state1 = states1[pattern];\
    if (state1 < PADDED_STATE_COUNT) {\
        for(int i = 0; i < PADDED_STATE_COUNT; i++) {\
            sum1[i] = sMatrix1[state1 * PADDED_STATE_COUNT + i];\
        }\
    } else {\
        for(int i = 0; i < PADDED_STATE_COUNT; i++) {\
            sum1[i] = 1.0;\
        }\
    }\
    for(int i = 0; i < PADDED_STATE_COUNT; i++) {\
        sum2[i] = sMatrix2[0 * PADDED_STATE_COUNT + i] * sPartials2[0];\
    }\
    for (int s = 1; s < PADDED_STATE_COUNT; s++) {\
        for(int i = 0; i < PADDED_STATE_COUNT; i++) {\
            FMA(sMatrix2[s * PADDED_STATE_COUNT + i],  sPartials2[s], sum2[i]);\
        }\
    }


#define SUM_STATES_STATES_4_CPU()\
    KW_GLOBAL_VAR REAL* KW_RESTRICT sMatrix1 = matrices1 + deltaMatrix;\
    KW_GLOBAL_VAR REAL* KW_RESTRICT sMatrix2 = matrices2 + deltaMatrix;\
    int state1 = states1[pattern];\
    int state2 = states2[pattern];\
    if (state1 < PADDED_STATE_COUNT && state2 < PADDED_STATE_COUNT) {\
        for(int i = 0; i < PADDED_STATE_COUNT; i++) {\
            partials3[deltaPartials + i] = sMatrix1[state1 * 4 + i] * sMatrix2[state2 * 4 + i];\
        }\
    } else if (state1 < PADDED_STATE_COUNT) {\
        for(int i = 0; i < PADDED_STATE_COUNT; i++) {\
            partials3[deltaPartials + i] = sMatrix1[state1 * 4 + i];\
        }\
    } else if (state2 < PADDED_STATE_COUNT) {\
        for(int i = 0; i < PADDED_STATE_COUNT; i++) {\
            partials3[deltaPartials + i] = sMatrix2[state2 * 4 + i];\
        }\
    } else {\
        partials3[deltaPartials + 0] = 1.0;/* unrolled to work around Apple OpenCL bug*/\
        partials3[deltaPartials + 1] = 1.0;\
        partials3[deltaPartials + 2] = 1.0;\
        partials3[deltaPartials + 3] = 1.0;\
    }

#define SUM_STATES_STATES_4_SCALE_CPU()\
    KW_GLOBAL_VAR REAL* KW_RESTRICT sMatrix1 = matrices1 + deltaMatrix;\
    KW_GLOBAL_VAR REAL* KW_RESTRICT sMatrix2 = matrices2 + deltaMatrix;\
    int state1 = states1[pattern];\
    int state2 = states2[pattern];\
    REAL oneOverScaling = 1.0/scalingFactors[pattern];\
    if (state1 < PADDED_STATE_COUNT && state2 < PADDED_STATE_COUNT) {\
        for(int i = 0; i < PADDED_STATE_COUNT; i++) {\
            partials3[deltaPartials + i] = sMatrix1[state1 * 4 + i] * sMatrix2[state2 * 4 + i]\
                                           * oneOverScaling;\
        }\
    } else if (state1 < PADDED_STATE_COUNT) {\
        for(int i = 0; i < PADDED_STATE_COUNT; i++) {\
            partials3[deltaPartials + i] = sMatrix1[state1 * 4 + i] * oneOverScaling;\
        }\
    } else if (state2 < PADDED_STATE_COUNT) {\
        for(int i = 0; i < PADDED_STATE_COUNT; i++) {\
            partials3[deltaPartials + i] = sMatrix2[state2 * 4 + i] * oneOverScaling;\
        }\
    } else {\
        for(int i = 0; i < PADDED_STATE_COUNT; i++) {\
            partials3[deltaPartials + i] = 1.0 * oneOverScaling;\
        }\
    }

#define SUM_PARTIALS_SINGLE_4_CPU()\
    REAL sum1[PADDED_STATE_COUNT];\
    KW_GLOBAL_VAR REAL* KW_RESTRICT sMatrix1 = matrices1 + deltaMatrix;\
    KW_GLOBAL_VAR REAL* KW_RESTRICT sPartials1 = partials1 + deltaPartials;\
    KW_GLOBAL_VAR REAL* KW_RESTRICT sPartials2 = partials2 + deltaPartials;\
    for(int i = 0; i < PADDED_STATE_COUNT; i++) {\
        sum1[i] = sMatrix1[0 * PADDED_STATE_COUNT + i] * sPartials1[0];\
    }\
    for (int s = 1; s < PADDED_STATE_COUNT; s++) {\
        for(int i = 0; i < PADDED_STATE_COUNT; i++) {\
            FMA(sMatrix1[s * PADDED_STATE_COUNT + i],  sPartials1[s], sum1[i]);\
        }\
    }

#define SUM_STATES_SINGLE_4_CPU()\
    REAL sum1[PADDED_STATE_COUNT];\
    KW_GLOBAL_VAR REAL* KW_RESTRICT sMatrix1 = matrices1 + deltaMatrix;\
    KW_GLOBAL_VAR REAL* KW_RESTRICT sPartials2 = partials2 + deltaPartials;\
    int state1 = dChildStates[pattern];\
    if (state1 < PADDED_STATE_COUNT) {\
        for(int i = 0; i < PADDED_STATE_COUNT; i++) {\
            sum1[i] = sMatrix1[state1 * PADDED_STATE_COUNT + i];\
        }\
    } else {\
        for(int i = 0; i < PADDED_STATE_COUNT; i++) {\
            sum1[i] = 1.0;\
        }\
    }

#define SUM_PARTIALS_DERIV_4_CPU()\
    REAL sum1[PADDED_STATE_COUNT];\
    REAL sumFirstDeriv[PADDED_STATE_COUNT], sumSecondDeriv[PADDED_STATE_COUNT];\
    KW_GLOBAL_VAR REAL* KW_RESTRICT sMatrix1 = matrices1 + deltaMatrix;\
    KW_GLOBAL_VAR REAL* KW_RESTRICT sMatrixFirstDeriv = dFirstDerivMatrix + deltaMatrix;\
    KW_GLOBAL_VAR REAL* KW_RESTRICT sMatrixSecondDeriv = dSecondDerivMatrix + deltaMatrix;\
    KW_GLOBAL_VAR REAL* KW_RESTRICT sPartials1 = partials1 + deltaPartials;\
    KW_GLOBAL_VAR REAL* KW_RESTRICT sPartials2 = partials2 + deltaPartials;\
    for(int i = 0; i < PADDED_STATE_COUNT; i++) {\
        sum1[i]           = sMatrix1[          0 * PADDED_STATE_COUNT + i] * sPartials1[0];\
        sumFirstDeriv[i]  = sMatrixFirstDeriv[ 0 * PADDED_STATE_COUNT + i] * sPartials1[0];\
        sumSecondDeriv[i] = sMatrixSecondDeriv[0 * PADDED_STATE_COUNT + i] * sPartials1[0];\
    }\
    for (int s = 1; s < PADDED_STATE_COUNT; s++) {\
        for(int i = 0; i < PADDED_STATE_COUNT; i++) {\
            FMA(sMatrix1[          s * PADDED_STATE_COUNT + i],  sPartials1[s], sum1[i]);\
            FMA(sMatrixFirstDeriv[ s * PADDED_STATE_COUNT + i],  sPartials1[s], sumFirstDeriv[i]);\
            FMA(sMatrixSecondDeriv[s * PADDED_STATE_COUNT + i],  sPartials1[s], sumSecondDeriv[i]);\
        }\
    }

#define SUM_STATES_DERIV_4_CPU()\
    REAL sum1[PADDED_STATE_COUNT];\
    REAL sumFirstDeriv[PADDED_STATE_COUNT], sumSecondDeriv[PADDED_STATE_COUNT];\
    KW_GLOBAL_VAR REAL* KW_RESTRICT sMatrix1 = matrices1 + deltaMatrix;\
    KW_GLOBAL_VAR REAL* KW_RESTRICT sMatrixFirstDeriv = dFirstDerivMatrix + deltaMatrix;\
    KW_GLOBAL_VAR REAL* KW_RESTRICT sMatrixSecondDeriv = dSecondDerivMatrix + deltaMatrix;\
    KW_GLOBAL_VAR REAL* KW_RESTRICT sPartials2 = partials2 + deltaPartials;\
    int state1 = dChildStates[pattern];\
    if (state1 < PADDED_STATE_COUNT) {\
        for(int i = 0; i < PADDED_STATE_COUNT; i++) {\
            sum1[i] =           sMatrix1[          state1 * PADDED_STATE_COUNT + i];\
            sumFirstDeriv[i]  = sMatrixFirstDeriv[ state1 * PADDED_STATE_COUNT + i];\
            sumSecondDeriv[i] = sMatrixSecondDeriv[state1 * PADDED_STATE_COUNT + i];\
        }\
    } else {\
        for(int i = 0; i < PADDED_STATE_COUNT; i++) {\
            sum1[i] = 1.0;\
            sumFirstDeriv[i] = 0.0;\
            sumSecondDeriv[i] = 0.0;\
        }\
    }

#define DETERMINE_SCALING_INDICES_4_CPU()\
    int patIdx = KW_LOCAL_ID_0;\
    int pattern = KW_GROUP_ID_0 * PATTERN_BLOCK_SIZE + patIdx;\
    int deltaPartialsByState = pattern * PADDED_STATE_COUNT;

#define DETERMINE_SCALING_INDICES_4_PARTITION_CPU()\
    int patIdx = KW_LOCAL_ID_0;\
    int pattern = startPattern + KW_GROUP_ID_0 * PATTERN_BLOCK_SIZE + patIdx;\
    int deltaPartialsByState = pattern * PADDED_STATE_COUNT;

#define FIND_MAX_PARTIALS_4_CPU()\
    REAL maxPartial = 0;\
    for(int m = 0; m < matrixCount; m++) {\
        int deltaPartialsByMatrix = m * PADDED_STATE_COUNT * PATTERN_BLOCK_SIZE * KW_NUM_GROUPS_0;\
        int deltaPartials = deltaPartialsByMatrix + deltaPartialsByState;\
        for(int i = 0; i < PADDED_STATE_COUNT; i++) {\
            REAL iPartial = allPartials[deltaPartials + i];\
            if (iPartial > maxPartial)\
                maxPartial = iPartial;\
        }\
    }

#define FIND_MAX_PARTIALS_4_PARTITION_CPU()\
    REAL maxPartial = 0;\
    for(int m = 0; m < matrixCount; m++) {\
        int deltaPartialsByMatrix = m * PADDED_STATE_COUNT * totalPatterns;\
        int deltaPartials = deltaPartialsByMatrix + deltaPartialsByState;\
        for(int i = 0; i < PADDED_STATE_COUNT; i++) {\
            REAL iPartial = allPartials[deltaPartials + i];\
            if (iPartial > maxPartial)\
                maxPartial = iPartial;\
        }\
    }

#define SCALE_PARTIALS_4_CPU()\
    for(int m = 0; m < matrixCount; m++) {\
        int deltaPartialsByMatrix = m * PADDED_STATE_COUNT * PATTERN_BLOCK_SIZE * KW_NUM_GROUPS_0;\
        int deltaPartials = deltaPartialsByMatrix + deltaPartialsByState;\
        for(int i = 0; i < PADDED_STATE_COUNT; i++) {\
            allPartials[deltaPartials + i] /= maxPartial;\
        }\
    }

#define SCALE_PARTIALS_4_PARTITION_CPU()\
    for(int m = 0; m < matrixCount; m++) {\
        int deltaPartialsByMatrix = m * PADDED_STATE_COUNT * totalPatterns;\
        int deltaPartials = deltaPartialsByMatrix + deltaPartialsByState;\
        for(int i = 0; i < PADDED_STATE_COUNT; i++) {\
            allPartials[deltaPartials + i] /= maxPartial;\
        }\
    }

#define DETERMINE_INTEGRATE_INDICES_4_CPU()\
    int pat = KW_LOCAL_ID_0;\
    int pattern = KW_GROUP_ID_0 * LIKE_PATTERN_BLOCK_SIZE + pat;\
    int u = pattern * PADDED_STATE_COUNT;\
    int delta = patternCount * PADDED_STATE_COUNT;

#define DETERMINE_INTEGRATE_INDICES_4_PARTITION_CPU()\
    int opIndexPtr = KW_GROUP_ID_0 * 6;\
    int startPattern = dPtrOffsets[opIndexPtr    ];\
    int endPattern   = dPtrOffsets[opIndexPtr + 1];\
          KW_GLOBAL_VAR REAL* KW_RESTRICT dRootPartials =  dRootPartialsOrigin + dPtrOffsets[opIndexPtr + 2];\
    const KW_GLOBAL_VAR REAL* KW_RESTRICT dWeights      =  dWeightsOrigin + dPtrOffsets[opIndexPtr + 3];\
    const KW_GLOBAL_VAR REAL* KW_RESTRICT dFrequencies  =  dFrequenciesOrigin + dPtrOffsets[opIndexPtr + 4];\
    int pat = KW_LOCAL_ID_0;\
    int pattern = startPattern + pat;\
    int u = pattern * PADDED_STATE_COUNT;\
    int delta = patternCount * PADDED_STATE_COUNT;

#define INTEGRATE_PARTIALS_4_CPU()\
    REAL sum[PADDED_STATE_COUNT];\
    REAL sumTotal = 0;\
    for(int i = 0; i < PADDED_STATE_COUNT; i++) {\
        sum[i] = dRootPartials[i + u] * dWeights[0];\
    }\
    for(int r = 1; r < matrixCount; r++) {\
        for(int i = 0; i < PADDED_STATE_COUNT; i++) {\
            FMA(dRootPartials[i + u + delta * r],  dWeights[r], sum[i]);\
        }\
    }\
    for(int i = 0; i < PADDED_STATE_COUNT; i++) {\
        sum[i] *= dFrequencies[i];\
        sumTotal += sum[i];\
    }

#define INTEGRATE_PARTIALS_DERIV_4_CPU()\
    int pat = KW_LOCAL_ID_0;\
    int pattern = KW_GROUP_ID_0 * LIKE_PATTERN_BLOCK_SIZE + pat;\
    int u = pattern * PADDED_STATE_COUNT;\
    int delta = patternCount * PADDED_STATE_COUNT;\
    REAL sum[PADDED_STATE_COUNT];\
    REAL sumD1[PADDED_STATE_COUNT], sumD2[PADDED_STATE_COUNT];\
    REAL sumTotal = 0, sumTotalD1 = 0, sumTotalD2 = 0;\
    REAL tmpLogLike, tmpFirstDeriv;\
    for(int i = 0; i < PADDED_STATE_COUNT; i++) {\
        sum[i]   = dRootPartials[   i + u] * dWeights[0];\
        sumD1[i] = dRootFirstDeriv[ i + u] * dWeights[0];\
        sumD2[i] = dRootSecondDeriv[i + u] * dWeights[0];\
    }\
    for(int r = 1; r < matrixCount; r++) {\
        for(int i = 0; i < PADDED_STATE_COUNT; i++) {\
            FMA(dRootPartials[   i + u + delta * r],  dWeights[r], sum[i]);\
            FMA(dRootFirstDeriv[ i + u + delta * r],  dWeights[r], sumD1[i]);\
            FMA(dRootSecondDeriv[i + u + delta * r],  dWeights[r], sumD2[i]);\
        }\
    }\
    for(int i = 0; i < PADDED_STATE_COUNT; i++) {\
        sum[i]     *= dFrequencies[i];\
        sumD1[i]   *= dFrequencies[i];\
        sumD2[i]   *= dFrequencies[i];\
        sumTotal   += sum[i];\
        sumTotalD1 += sumD1[i];\
        sumTotalD2 += sumD2[i];\
    }

///////////////////////////////////////////////////////////////////////////////
// kernel macros GPU

// Do not use | (instead of +) for any term involving PATTERN_BLOCK_SIZE
// as this should be adjustable
#define DETERMINE_INDICES_4_GPU()\
    int tx = KW_LOCAL_ID_0;\
    int state = tx & 0x3;\
    int pat = tx >> 2;\
    int patIdx = KW_LOCAL_ID_1;\
    int matrix = KW_GROUP_ID_1;\
    int pattern = __umul24(KW_GROUP_ID_0, PATTERN_BLOCK_SIZE * 4) + multBy4(patIdx) + pat;\
    int deltaPartialsByState = multBy16(KW_GROUP_ID_0 * PATTERN_BLOCK_SIZE + patIdx);\
    int deltaPartialsByMatrix = __umul24(matrix, multBy4(endPattern));\
    int x2 = multBy16(matrix);\
    int u = tx + deltaPartialsByState + deltaPartialsByMatrix;

#define DETERMINE_INDICES_4_MULTI_1_GPU()\
    int opIndexPtr = (gridStartOp + KW_GROUP_ID_0) * 8;\
    int startPat   = ptrOffsets[opIndexPtr    ];\
    int endPattern = ptrOffsets[opIndexPtr + 1];\
    int tx = KW_LOCAL_ID_0;\
    int state = tx & 0x3;\
    int pat = tx >> 2;\
    int patIdx = KW_LOCAL_ID_1;\
    int matrix = KW_GROUP_ID_1;\
    int pattern = startPat + multBy4(patIdx) + pat;\
    int deltaPartialsByState = multBy4(startPat) + multBy16(patIdx);\
    int deltaPartialsByMatrix = __umul24(matrix, multBy4(totalPatterns));\
    int x2 = multBy16(matrix);\
    int u = tx + deltaPartialsByState + deltaPartialsByMatrix;

#define DETERMINE_INDICES_4_MULTI_2_GPU()\
          KW_GLOBAL_VAR REAL* KW_RESTRICT partials3 =  partials + ptrOffsets[opIndexPtr + 4];\
    const KW_GLOBAL_VAR REAL* KW_RESTRICT matrix1   =  matrices + ptrOffsets[opIndexPtr + 5];\
    const KW_GLOBAL_VAR REAL* KW_RESTRICT matrix2   =  matrices + ptrOffsets[opIndexPtr + 6];

#define DETERMINE_INDICES_4_PART_GPU()\
    int tx = KW_LOCAL_ID_0;\
    int state = tx & 0x3;\
    int pat = tx >> 2;\
    int patIdx = KW_LOCAL_ID_1;\
    int matrix = KW_GROUP_ID_1;\
    int deltaPartialsByMatrix = __umul24(matrix, multBy4(totalPatterns));\
    int x2 = multBy16(matrix);\
    int pattern = startPattern + __umul24(KW_GROUP_ID_0, PATTERN_BLOCK_SIZE * 4) + multBy4(patIdx) + pat;\
    int deltaPartialsByState = multBy4(startPattern) + multBy16(KW_GROUP_ID_0 * PATTERN_BLOCK_SIZE + patIdx);\
    int u = tx + deltaPartialsByState + deltaPartialsByMatrix;

#define DETERMINE_INDICES_4_EDGEPART_1_GPU()\
    int opIndexPtr = KW_GROUP_ID_0 * 5;\
    int startPat   = ptrOffsets[opIndexPtr    ];\
    int endPattern = ptrOffsets[opIndexPtr + 1];\
    int tx = KW_LOCAL_ID_0;\
    int state = tx & 0x3;\
    int pat = tx >> 2;\
    int patIdx = KW_LOCAL_ID_1;\
    int matrix = KW_GROUP_ID_1;\
    int pattern = startPat + multBy4(patIdx) + pat;\
    int deltaPartialsByState = multBy4(startPat) + multBy16(patIdx);\
    int deltaPartialsByMatrix = __umul24(matrix, multBy4(totalPatterns));\
    int x2 = multBy16(matrix);\
    int u = tx + deltaPartialsByState + deltaPartialsByMatrix;

#define LOAD_PARTIALS_PARTIALS_4_GPU()\
    int y = deltaPartialsByState + deltaPartialsByMatrix;\
    KW_LOCAL_MEM REAL sPartials1[PATTERN_BLOCK_SIZE * 4 * 4];\
    KW_LOCAL_MEM REAL sPartials2[PATTERN_BLOCK_SIZE * 4 * 4];\
    /* copy PADDED_STATE_COUNT * PATTERN_BLOCK_SIZE lengthed partials*/\
    if (pattern < endPattern) {\
        sPartials1[multBy16(patIdx) | tx] = partials1[y | tx]; /*All coalesced memory*/\
        sPartials2[multBy16(patIdx) | tx] = partials2[y | tx];\
    } else {\
        sPartials1[multBy16(patIdx) | tx] = 0;\
        sPartials2[multBy16(patIdx) | tx] = 0;\
    }

#define LOAD_PARTIALS_PARTIALS_4_MULTI_PART_GPU()\
    KW_LOCAL_MEM REAL sPartials1[PATTERN_BLOCK_SIZE * 4 * 4];\
    KW_LOCAL_MEM REAL sPartials2[PATTERN_BLOCK_SIZE * 4 * 4];\
    /* copy PADDED_STATE_COUNT * PATTERN_BLOCK_SIZE lengthed partials*/\
    if (pattern < endPattern) {\
        sPartials1[multBy16(patIdx) | tx] = partials1[u]; /*All coalesced memory*/\
        sPartials2[multBy16(patIdx) | tx] = partials2[u];\
    } else {\
        sPartials1[multBy16(patIdx) | tx] = 0;\
        sPartials2[multBy16(patIdx) | tx] = 0;\
    }

#define LOAD_PARTIALS_SINGLE_4_GPU()\
    int y = deltaPartialsByState + deltaPartialsByMatrix;\
    KW_LOCAL_MEM REAL sPartials2[PATTERN_BLOCK_SIZE * 4 * 4];\
    if (pattern < endPattern) {\
        sPartials2[multBy16(patIdx) | tx] = partials2[y | tx];\
    } else {\
        sPartials2[multBy16(patIdx) | tx] = 0;\
    }

#define LOAD_PARTIALS_SINGLE_4_MULTI_PART_GPU()\
    KW_LOCAL_MEM REAL sPartials2[PATTERN_BLOCK_SIZE * 4 * 4];\
    if (pattern < endPattern) {\
        sPartials2[multBy16(patIdx) | tx] = partials2[u];\
    } else {\
        sPartials2[multBy16(patIdx) | tx] = 0;\
    }

#define LOAD_SCALING_4_GPU()\
    KW_LOCAL_MEM REAL fixedScalingFactors[PATTERN_BLOCK_SIZE * 4];\
    if (patIdx < 4) { /* need to load 4*PATTERN_BLOCK_SIZE factors for this block*/\
        fixedScalingFactors[patIdx * PATTERN_BLOCK_SIZE + tx] = \
            scalingFactors[KW_GROUP_ID_0 * PATTERN_BLOCK_SIZE * 4 + patIdx * PATTERN_BLOCK_SIZE + tx];\
    }

#define LOAD_SCALING_4_MULTI_GPU()\
    KW_LOCAL_MEM REAL fixedScalingFactors[PATTERN_BLOCK_SIZE * 4];\
    if (patIdx < 4) { /* need to load 4*PATTERN_BLOCK_SIZE factors for this block*/\
        fixedScalingFactors[patIdx * PATTERN_BLOCK_SIZE + tx] = \
            scalingFactors[startPat + patIdx * PATTERN_BLOCK_SIZE + tx];\
    }

#define LOAD_SCALING_4_PART_GPU()\
    KW_LOCAL_MEM REAL fixedScalingFactors[PATTERN_BLOCK_SIZE * 4];\
    if (patIdx < 4) { /* need to load 4*PATTERN_BLOCK_SIZE factors for this block*/\
        fixedScalingFactors[patIdx * PATTERN_BLOCK_SIZE + tx] = \
            scalingFactors[startPattern + KW_GROUP_ID_0 * PATTERN_BLOCK_SIZE * 4 + patIdx * PATTERN_BLOCK_SIZE + tx];\
    }


#define LOAD_MATRIX_4_GPU()\
    const KW_GLOBAL_VAR REAL* KW_RESTRICT matrix1 = matrices1 + x2; /*Points to *this* matrix*/\
    const KW_GLOBAL_VAR REAL* KW_RESTRICT matrix2 = matrices2 + x2;\
    KW_LOCAL_MEM REAL sMatrix1[16]; /*Load values into shared memory*/\
    KW_LOCAL_MEM REAL sMatrix2[16];\
    if (patIdx == 0 ) {\
        sMatrix1[tx] = matrix1[tx]; /*All coalesced memory reads*/\
        sMatrix2[tx] = matrix2[tx];\
    }\
    KW_LOCAL_FENCE;

#define LOAD_MATRIX_4_MULTI_GPU()\
    matrix1 += x2;\
    matrix2 += x2;\
    KW_LOCAL_MEM REAL sMatrix1[16];\
    KW_LOCAL_MEM REAL sMatrix2[16];\
    if (patIdx == 0 ) {\
        sMatrix1[tx] = matrix1[tx];\
        sMatrix2[tx] = matrix2[tx];\
    }\
    KW_LOCAL_FENCE;

#define LOAD_MATRIX_SINGLE_4_GPU()\
    KW_GLOBAL_VAR REAL* KW_RESTRICT matrix1 = matrices1 + x2;\
    KW_LOCAL_MEM REAL sMatrix1[16];\
    if (patIdx == 0 ) {\
        sMatrix1[tx] = matrix1[tx];\
    }\
    KW_LOCAL_FENCE;

#define LOAD_MATRIX_DERIV_4_GPU()\
    KW_GLOBAL_VAR REAL* KW_RESTRICT matrix1 = matrices1 + x2;\
    KW_GLOBAL_VAR REAL* KW_RESTRICT matrixFirstDeriv = dFirstDerivMatrix + x2;\
    KW_GLOBAL_VAR REAL* KW_RESTRICT matrixSecondDeriv = dSecondDerivMatrix + x2;\
    KW_LOCAL_MEM REAL sMatrix1[16];\
    KW_LOCAL_MEM REAL sMatrixFirstDeriv[16];\
    KW_LOCAL_MEM REAL sMatrixSecondDeriv[16];\
    if (patIdx == 0) {\
        sMatrix1[tx] = matrix1[tx];\
        sMatrixFirstDeriv[tx] = matrixFirstDeriv[tx];\
        sMatrixSecondDeriv[tx] = matrixSecondDeriv[tx];\
    }\
    KW_LOCAL_FENCE;

#define SUM_PARTIALS_PARTIALS_4_GPU()\
    REAL sum1, sum2;\
    int i = pat;\
    int patIdx16pat4 = multBy16(patIdx) | (tx & 0xC);\
    sum1 = sMatrix1[multBy4(i) | state] * sPartials1[patIdx16pat4 | i];\
    sum2 = sMatrix2[multBy4(i) | state] * sPartials2[patIdx16pat4 | i];\
    i = (i + 1) & 0x3;\
    FMA(   sMatrix1[multBy4(i) | state],  sPartials1[patIdx16pat4 | i], sum1);\
    FMA(   sMatrix2[multBy4(i) | state],  sPartials2[patIdx16pat4 | i], sum2);\
    i = (i + 1) & 0x3;\
    FMA(   sMatrix1[multBy4(i) | state],  sPartials1[patIdx16pat4 | i], sum1);\
    FMA(   sMatrix2[multBy4(i) | state],  sPartials2[patIdx16pat4 | i], sum2);\
    i = (i + 1) & 0x3;\
    FMA(   sMatrix1[multBy4(i) | state],  sPartials1[patIdx16pat4 | i], sum1);\
    FMA(   sMatrix2[multBy4(i) | state],  sPartials2[patIdx16pat4 | i], sum2);

#define SUM_STATES_PARTIALS_4_GPU()\
    REAL sum1 = 1, sum2;\
    int state1 = states1[pattern];\
    if (state1 < PADDED_STATE_COUNT)\
        sum1 = sMatrix1[state1 * 4 + state];\
    int i = pat;\
    int patIdx16pat4 = multBy16(patIdx) | (tx & 0xC);\
    sum2  = sMatrix2[multBy4(i) | state] * sPartials2[patIdx16pat4 | i];\
    i = (i + 1) & 0x3;\
    FMA(    sMatrix2[multBy4(i) | state],  sPartials2[patIdx16pat4 | i], sum2);\
    i = (i + 1) & 0x3;\
    FMA(    sMatrix2[multBy4(i) | state],  sPartials2[patIdx16pat4 | i], sum2);\
    i = (i + 1) & 0x3;\
    FMA(    sMatrix2[multBy4(i) | state],  sPartials2[patIdx16pat4 | i], sum2);

#define SUM_STATES_STATES_4_GPU()\
    int state1 = states1[pattern];\
    int state2 = states2[pattern];\
    if (state1 < PADDED_STATE_COUNT && state2 < PADDED_STATE_COUNT) {\
        partials3[u] = sMatrix1[state1 * 4 + state] * sMatrix2[state2 * 4 + state];\
    } else if (state1 < PADDED_STATE_COUNT) {\
        partials3[u] = sMatrix1[state1 * 4 + state];\
    } else if (state2 < PADDED_STATE_COUNT) {\
        partials3[u] = sMatrix2[state2 * 4 + state];\
    } else {\
        partials3[u] = 1.0;\
    }

#define SUM_STATES_STATES_4_SCALE_GPU()\
    int state1 = states1[pattern];\
    int state2 = states2[pattern];\
    if (state1 < PADDED_STATE_COUNT && state2 < PADDED_STATE_COUNT) {\
        partials3[u] = sMatrix1[state1 * 4 + state] * sMatrix2[state2 * 4 + state]\
                       / fixedScalingFactors[patIdx * 4 + pat];\
    } else if (state1 < PADDED_STATE_COUNT) {\
        partials3[u] = sMatrix1[state1 * 4 + state] / fixedScalingFactors[patIdx * 4 + pat];\
    } else if (state2 < PADDED_STATE_COUNT) {\
        partials3[u] = sMatrix2[state2 * 4 + state] / fixedScalingFactors[patIdx * 4 + pat];\
    } else {\
        partials3[u] = 1.0 / fixedScalingFactors[patIdx * 4 + pat];\
    }

#define SUM_PARTIALS_SINGLE_4_GPU()\
    REAL sum1;\
    int i = pat;\
    int patIdx16pat4 = multBy16(patIdx) | (tx & 0xC);\
    sum1  = sMatrix1[multBy4(i) | state] * sPartials1[patIdx16pat4 | i];\
    i = (i + 1) & 0x3;\
    FMA(    sMatrix1[multBy4(i) | state],  sPartials1[patIdx16pat4 | i], sum1);\
    i = (i + 1) & 0x3;\
    FMA(    sMatrix1[multBy4(i) | state],  sPartials1[patIdx16pat4 | i], sum1);\
    i = (i + 1) & 0x3;\
    FMA(    sMatrix1[multBy4(i) | state],  sPartials1[patIdx16pat4 | i], sum1);

#define SUM_STATES_SINGLE_4_GPU()\
    REAL sum1;\
    int state1 = dChildStates[pattern];\
    if (state1 < PADDED_STATE_COUNT)\
        sum1 = sMatrix1[state1 * 4 + state];\
    else\
        sum1 = 1.0;

#define SUM_PARTIALS_DERIV_4_GPU()\
    REAL sum1, sumFirstDeriv, sumSecondDeriv;\
    int i = pat;\
    int patIdx16pat4 = multBy16(patIdx) | (tx & 0xC);\
    sum1           = sMatrix1[          multBy4(i) | state] * sPartials1[patIdx16pat4 | i];\
    sumFirstDeriv  = sMatrixFirstDeriv[ multBy4(i) | state] * sPartials1[patIdx16pat4 | i];\
    sumSecondDeriv = sMatrixSecondDeriv[multBy4(i) | state] * sPartials1[patIdx16pat4 | i];\
    i = (i + 1) & 0x3;\
    FMA(sMatrix1[          multBy4(i) | state], sPartials1[patIdx16pat4 | i], sum1);\
    FMA(sMatrixFirstDeriv[ multBy4(i) | state], sPartials1[patIdx16pat4 | i], sumFirstDeriv);\
    FMA(sMatrixSecondDeriv[multBy4(i) | state], sPartials1[patIdx16pat4 | i], sumSecondDeriv);\
    i = (i + 1) & 0x3;\
    FMA(sMatrix1[          multBy4(i) | state], sPartials1[patIdx16pat4 | i], sum1);\
    FMA(sMatrixFirstDeriv[ multBy4(i) | state], sPartials1[patIdx16pat4 | i], sumFirstDeriv);\
    FMA(sMatrixSecondDeriv[multBy4(i) | state], sPartials1[patIdx16pat4 | i], sumSecondDeriv);\
    i = (i + 1) & 0x3;\
    FMA(sMatrix1[          multBy4(i) | state], sPartials1[patIdx16pat4 | i], sum1);\
    FMA(sMatrixFirstDeriv[ multBy4(i) | state], sPartials1[patIdx16pat4 | i], sumFirstDeriv);\
    FMA(sMatrixSecondDeriv[multBy4(i) | state], sPartials1[patIdx16pat4 | i], sumSecondDeriv);

#define SUM_STATES_DERIV_4_GPU()\
    REAL sum1, sumFirstDeriv, sumSecondDeriv;\
    int state1 = dChildStates[pattern];\
    if (state1 < PADDED_STATE_COUNT) {\
        sum1 = sMatrix1[state1 * 4 + state];\
        sumFirstDeriv = sMatrixFirstDeriv[state1 * 4 + state];\
        sumSecondDeriv = sMatrixSecondDeriv[state1 * 4 + state];\
    } else {\
        sum1 = 1.0;\
        sumFirstDeriv = 0.0;\
        sumSecondDeriv = 0.0;\
    }

#define DETERMINE_SCALING_INDICES_4_GPU()\
    int tx = KW_LOCAL_ID_0;\
    int state = tx & 0x3;\
    int pat = tx >> 2;\
    int patIdx = KW_GROUP_ID_0;\
    int pattern = (patIdx << 2) + pat;\
    int matrix = KW_LOCAL_ID_1;\
    /* TODO: Assumes matrixCount < MATRIX_BLOCK_SIZE*/\
    /* Patterns are always padded, so no reading/writing past end possible*/\
    /* Find start of patternBlock for thread-block*/\
    int partialsOffset = (matrix * KW_NUM_GROUPS_0 + patIdx) << 4; /* 16;*/

#define DETERMINE_SCALING_INDICES_4_PARTITION_GPU()\
    int tx = KW_LOCAL_ID_0;\
    int state = tx & 0x3;\
    int pat = tx >> 2;\
    int patIdx = KW_GROUP_ID_0;\
    int pattern = startPattern + (patIdx << 2) + pat;\
    int matrix = KW_LOCAL_ID_1;\
    int partialsOffset = ((startPattern + matrix * totalPatterns) << 2) + (patIdx << 4); /* 16;*/

#define FIND_MAX_PARTIALS_STATE_4_DECLARE_GPU()\
    KW_LOCAL_MEM REAL partials[MATRIX_BLOCK_SIZE][16]; /* 4 patterns at a time*/\
    KW_LOCAL_MEM REAL storedPartials[MATRIX_BLOCK_SIZE][16];\
    KW_LOCAL_MEM REAL matrixMax[4];

#define FIND_MAX_PARTIALS_STATE_4_GPU()\
    if (matrix < matrixCount)\
        partials[matrix][tx] = allPartials[partialsOffset + tx];          \
    storedPartials[matrix][tx] = partials[matrix][tx];\
    KW_LOCAL_FENCE;\
    /* Unrolled parallel max-reduction*/\
    if (state < 2) {\
        REAL compare1 = partials[matrix][tx];\
        REAL compare2 = partials[matrix][tx + 2];\
        if (compare2 > compare1)\
            partials[matrix][tx] = compare2;\
    }\
    KW_LOCAL_FENCE;\
    if (state < 1) {\
        REAL compare1 = partials[matrix][tx];\
        REAL compare2 = partials[matrix][tx + 1];\
        if (compare2 > compare1)\
            partials[matrix][tx] = compare2;\
    }\
    KW_LOCAL_FENCE;

#define FIND_MAX_PARTIALS_MATRIX_4_GPU()\
    matrixMax[pat] = 0;\
    for(int m = 0; m < matrixCount; m++) {\
        if (partials[m][tx] > matrixMax[pat])\
            matrixMax[pat] = partials[m][tx];\
    }

#define SCALE_PARTIALS_4_GPU()\
    KW_LOCAL_FENCE;\
    if (matrix < matrixCount)\
        allPartials[partialsOffset + tx] = storedPartials[matrix][tx] / matrixMax[pat];

#define DETERMINE_INTEGRATE_INDICES_4_GPU()\
    int state = KW_LOCAL_ID_0;\
    int pat = KW_LOCAL_ID_1;\
    int pattern = KW_GROUP_ID_0 * LIKE_PATTERN_BLOCK_SIZE + KW_LOCAL_ID_1;\
    int u = state + pattern * PADDED_STATE_COUNT;\
    int delta = patternCount * PADDED_STATE_COUNT;

#define DETERMINE_INTEGRATE_INDICES_4_PARTITION_GPU()\
    int opIndexPtr = KW_GROUP_ID_0 * 6;\
    int startPattern = dPtrOffsets[opIndexPtr    ];\
    int endPattern   = dPtrOffsets[opIndexPtr + 1];\
          KW_GLOBAL_VAR REAL* KW_RESTRICT dRootPartials =  dRootPartialsOrigin + dPtrOffsets[opIndexPtr + 2];\
    const KW_GLOBAL_VAR REAL* KW_RESTRICT dWeights      =  dWeightsOrigin      + dPtrOffsets[opIndexPtr + 3];\
    const KW_GLOBAL_VAR REAL* KW_RESTRICT dFrequencies  =  dFrequenciesOrigin  + dPtrOffsets[opIndexPtr + 4];\
    int state = KW_LOCAL_ID_0;\
    int pat = KW_LOCAL_ID_1;\
    int pattern = startPattern + KW_LOCAL_ID_1;\
    int u = state + pattern * PADDED_STATE_COUNT;\
    int delta = patternCount * PADDED_STATE_COUNT;

#define INTEGRATE_PARTIALS_4_GPU()\
    KW_LOCAL_MEM REAL stateFreq[4];\
    /* TODO: Currently assumes MATRIX_BLOCK_SIZE >= matrixCount */\
    KW_LOCAL_MEM REAL matrixProp[MATRIX_BLOCK_SIZE];\
    KW_LOCAL_MEM REAL sum[LIKE_PATTERN_BLOCK_SIZE][4];\
    /* Load shared memory */\
    if (pat == 0) {\
        stateFreq[state] = dFrequencies[state];\
    }\
    sum[pat][state] = 0;\
    /* TODO: Assumes matrixCount < LIKE_PATTERN_BLOCK_SIZE * 4 */\
    if (pat * 4 + state < matrixCount) {\
        matrixProp[pat * 4 + state] = dWeights[pat * 4 + state];\
    }\
    KW_LOCAL_FENCE;\
    for(int r = 0; r < matrixCount; r++) {\
        FMA(dRootPartials[u + delta * r], matrixProp[r], sum[pat][state]);\
    }\
    sum[pat][state] *= stateFreq[state];\
    KW_LOCAL_FENCE;\
    if (state < 2)\
        sum[pat][state] += sum[pat][state + 2];\
    KW_LOCAL_FENCE;\
    if (state < 1) {\
        sum[pat][state] += sum[pat][state + 1];\
    }

#define INTEGRATE_PARTIALS_DERIV_4_GPU()\
    int state = KW_LOCAL_ID_0;\
    int pat = KW_LOCAL_ID_1;\
    int pattern = KW_GROUP_ID_0 * LIKE_PATTERN_BLOCK_SIZE + KW_LOCAL_ID_1;\
    REAL tmpLogLike, tmpFirstDeriv;\
    KW_LOCAL_MEM REAL stateFreq[4];\
    /* TODO: Currently assumes MATRIX_BLOCK_SIZE >= matrixCount */\
    KW_LOCAL_MEM REAL matrixProp[MATRIX_BLOCK_SIZE];\
    KW_LOCAL_MEM REAL sum[LIKE_PATTERN_BLOCK_SIZE][4];\
    KW_LOCAL_MEM REAL sumD1[LIKE_PATTERN_BLOCK_SIZE][4];\
    KW_LOCAL_MEM REAL sumD2[LIKE_PATTERN_BLOCK_SIZE][4];\
    /* Load shared memory */\
    if (pat == 0) {\
        stateFreq[state] = dFrequencies[state];\
    }\
    sum[pat][state]   = 0;\
    sumD1[pat][state] = 0;\
    sumD2[pat][state] = 0;\
    /* TODO: Assumes matrixCount < LIKE_PATTERN_BLOCK_SIZE * 4 */\
    if (pat * 4 + state < matrixCount) {\
        matrixProp[pat * 4 + state] = dWeights[pat * 4 + state];\
    }\
    KW_LOCAL_FENCE;\
    int u = state + pattern * PADDED_STATE_COUNT;\
    int delta = patternCount * PADDED_STATE_COUNT;\
    for(int r = 0; r < matrixCount; r++) {\
        FMA(dRootPartials[u + delta * r]   , matrixProp[r], sum[pat][state]);\
        FMA(dRootFirstDeriv[u + delta * r] , matrixProp[r], sumD1[pat][state]);\
        FMA(dRootSecondDeriv[u + delta * r], matrixProp[r], sumD2[pat][state]);\
    }\
    sum[pat][state]   *= stateFreq[state];\
    sumD1[pat][state] *= stateFreq[state];\
    sumD2[pat][state] *= stateFreq[state];\
    KW_LOCAL_FENCE;\
    if (state < 2) {\
        sum[pat][state]   += sum[pat][state + 2];\
        sumD1[pat][state] += sumD1[pat][state + 2];\
        sumD2[pat][state] += sumD2[pat][state + 2];\
    }\
    KW_LOCAL_FENCE;\
    if (state < 1) {\
        sum[pat][state]   += sum[pat][state + 1];\
        sumD1[pat][state] += sumD1[pat][state + 1];\
        sumD2[pat][state] += sumD2[pat][state + 1];\
    }

///////////////////////////////////////////////////////////////////////////////


KW_GLOBAL_KERNEL void kernelPartialsPartialsNoScaleMulti(KW_GLOBAL_VAR REAL* KW_RESTRICT partials,
                                                         const KW_GLOBAL_VAR REAL* KW_RESTRICT matrices,
                                                         const KW_GLOBAL_VAR unsigned int* KW_RESTRICT ptrOffsets,
                                                         int gridStartOp,
                                                         int totalPatterns) {

#ifdef FW_OPENCL_CPU // CPU/MIC implementation
    DETERMINE_INDICES_4_MULTI_1_CPU();
    if (pattern < endPattern) {
        DETERMINE_INDICES_4_MULTI_2_CPU();
        const KW_GLOBAL_VAR REAL* KW_RESTRICT partials1 =  partials + ptrOffsets[opIndexPtr + 2];
        const KW_GLOBAL_VAR REAL* KW_RESTRICT partials2 =  partials + ptrOffsets[opIndexPtr + 3];
        DETERMINE_INDICES_4_MULTI_3_CPU();
        SUM_PARTIALS_PARTIALS_4_CPU();
        for(int i = 0; i < PADDED_STATE_COUNT; i++) {
            partials3[deltaPartials + i] = sum1[i] * sum2[i];
        }
    }
#else // GPU implementation
    DETERMINE_INDICES_4_MULTI_1_GPU();
    const KW_GLOBAL_VAR REAL* KW_RESTRICT partials1 =  partials + ptrOffsets[opIndexPtr + 2];
    const KW_GLOBAL_VAR REAL* KW_RESTRICT partials2 =  partials + ptrOffsets[opIndexPtr + 3];
    DETERMINE_INDICES_4_MULTI_2_GPU();
    LOAD_PARTIALS_PARTIALS_4_MULTI_PART_GPU();
    LOAD_MATRIX_4_MULTI_GPU();
    if (pattern < endPattern) { // Remove padded threads!
        SUM_PARTIALS_PARTIALS_4_GPU();
        partials3[u] = sum1 * sum2;
    }
#endif // FW_OPENCL_CPU
}

//KW_GLOBAL_KERNEL void kernelPartialsPartialsEdgeFirstDerivatives(KW_GLOBAL_VAR REAL* KW_RESTRICT out,
//                                                                 KW_GLOBAL_VAR REAL* KW_RESTRICT partials1,
//                                                                 KW_GLOBAL_VAR REAL* KW_RESTRICT partials2,
//                                                                 KW_GLOBAL_VAR REAL* KW_RESTRICT matrices1,
//                                                                 KW_GLOBAL_VAR REAL* KW_RESTRICT weights,
//                                                                 int endPattern, int rateCategories) {
//#ifdef FW_OPENCL_CPU // CPU/MIC implementation
//    // Not implemented
//#else // GPU implementation
////    DETERMINE_INDICES_4_GPU();
////    LOAD_PARTIALS_PARTIALS_4_GPU();
////
////    const KW_GLOBAL_VAR REAL* KW_RESTRICT matrix1 = matrices1 + x2; /*Points to *this* matrix*/
////    const KW_GLOBAL_VAR REAL* KW_RESTRICT matrix2 = matrices2 + x2;
////    KW_LOCAL_MEM REAL sMatrix1[16]; /*Load values into shared memory*/
////    KW_LOCAL_MEM REAL sMatrix2[16];
////    if (patIdx == 0 ) {
////        sMatrix1[multBy4(state) | pat] = matrix1[tx]; /* Should write transpose into sMatrix1 */
////        sMatrix2[tx] = matrix2[tx];
////    }
////    KW_LOCAL_FENCE;
////
////    KW_LOCAL_MEM REAL sProduct[PATTERN_BLOCK_SIZE * 4 * 4];
////    if (pattern < endPattern) { // Remove padded threads!
////        REAL sum2;
////        int i = pat;
////        int patIdx16pat4 = multBy16(patIdx) | (tx & 0xC);
////
////        sum2 = sMatrix2[multBy4(i) | state] * sPartials2[patIdx16pat4 | i];
////        i = (i + 1) & 0x3;
////        FMA(   sMatrix2[multBy4(i) | state],  sPartials2[patIdx16pat4 | i], sum2);
////        i = (i + 1) & 0x3;
////        FMA(   sMatrix2[multBy4(i) | state],  sPartials2[patIdx16pat4 | i], sum2);
////        i = (i + 1) & 0x3;
////        FMA(   sMatrix2[multBy4(i) | state],  sPartials2[patIdx16pat4 | i], sum2);
////
////        sProduct[multBy16(patIdx) | tx] = sPartials1[multBy16(patIdx) | tx] * sum2;
////        //partials3[u] = sProduct[multBy16(patIdx) | tx]; /* These appear correct */
////    }
////
////    KW_LOCAL_FENCE;
////
////    if (pattern < endPattern) {
////        REAL sum1;
////        int i = pat;
////        int patIdx16pat4 = multBy16(patIdx) | (tx & 0xC);
////
////        sum1 = sMatrix1[multBy4(i) | state] * sProduct[patIdx16pat4 | i];
////        i = (i + 1) & 0x3;
////        FMA(   sMatrix1[multBy4(i) | state],  sProduct[patIdx16pat4 | i], sum1);
////        i = (i + 1) & 0x3;
////        FMA(   sMatrix1[multBy4(i) | state],  sProduct[patIdx16pat4 | i], sum1);
////        i = (i + 1) & 0x3;
////        FMA(   sMatrix1[multBy4(i) | state],  sProduct[patIdx16pat4 | i], sum1);
////
////        partials3[u] = sum1;
////    }
//#endif // FW_OPENCL_CPU
//}

KW_GLOBAL_KERNEL void kernelPartialsPartialsNoScalePartition(KW_GLOBAL_VAR REAL* KW_RESTRICT partials1,
                                                    KW_GLOBAL_VAR REAL* KW_RESTRICT partials2,
                                                    KW_GLOBAL_VAR REAL* KW_RESTRICT partials3,
                                                    KW_GLOBAL_VAR REAL* KW_RESTRICT matrices1,
                                                    KW_GLOBAL_VAR REAL* KW_RESTRICT matrices2,
                                                    int startPattern,
                                                    int endPattern,
                                                    int totalPatterns) {
#ifdef FW_OPENCL_CPU // CPU/MIC implementation
    DETERMINE_INDICES_4_PART_1_CPU();
    if (pattern < endPattern) {
        DETERMINE_INDICES_4_PART_2_CPU();
        SUM_PARTIALS_PARTIALS_4_CPU();
        for(int i = 0; i < PADDED_STATE_COUNT; i++) {
            partials3[deltaPartials + i] = sum1[i] * sum2[i];
        }
    }
#else // GPU implementation
    DETERMINE_INDICES_4_PART_GPU();
    LOAD_PARTIALS_PARTIALS_4_MULTI_PART_GPU();
    LOAD_MATRIX_4_GPU();
    if (pattern < endPattern) { // Remove padded threads!
        SUM_PARTIALS_PARTIALS_4_GPU();
        partials3[u] = sum1 * sum2;
    }
#endif // FW_OPENCL_CPU
}

KW_GLOBAL_KERNEL void kernelPartialsPartialsNoScale(KW_GLOBAL_VAR REAL* KW_RESTRICT partials1,
                                                    KW_GLOBAL_VAR REAL* KW_RESTRICT partials2,
                                                    KW_GLOBAL_VAR REAL* KW_RESTRICT partials3,
                                                    KW_GLOBAL_VAR REAL* KW_RESTRICT matrices1,
                                                    KW_GLOBAL_VAR REAL* KW_RESTRICT matrices2,
                                                    int endPattern) {
#ifdef FW_OPENCL_CPU // CPU/MIC implementation
    DETERMINE_INDICES_4_CPU();
    SUM_PARTIALS_PARTIALS_4_CPU();
    for(int i = 0; i < PADDED_STATE_COUNT; i++) {
        partials3[deltaPartials + i] = sum1[i] * sum2[i];
    }
#else // GPU implementation
    DETERMINE_INDICES_4_GPU();
    LOAD_PARTIALS_PARTIALS_4_GPU();
    LOAD_MATRIX_4_GPU();
    if (pattern < endPattern) { // Remove padded threads!
        SUM_PARTIALS_PARTIALS_4_GPU();
        partials3[u] = sum1 * sum2;
    }
#endif // FW_OPENCL_CPU

#ifdef KERNEL_PRINT_ENABLED
    printf("matrix = %d, pat = %d for tx = %d and state = %d :  u = %d\n",
           matrix, pattern, tx, state, u);
#endif
}

KW_GLOBAL_KERNEL void kernelPartialsPartialsFixedScaleMulti(
                                             KW_GLOBAL_VAR REAL*         KW_RESTRICT partials,
                                       const KW_GLOBAL_VAR REAL*         KW_RESTRICT matrices,
                                       const KW_GLOBAL_VAR REAL*         KW_RESTRICT scaleFactors,
                                       const KW_GLOBAL_VAR unsigned int* KW_RESTRICT ptrOffsets,
                                                                                 int gridStartOp,
                                                                                 int totalPatterns) {

#ifdef FW_OPENCL_CPU // CPU/MIC implementation
    DETERMINE_INDICES_4_MULTI_1_CPU();
    if (pattern < endPattern) {
        DETERMINE_INDICES_4_MULTI_2_CPU();
        const KW_GLOBAL_VAR REAL* KW_RESTRICT partials1 =  partials + ptrOffsets[opIndexPtr + 2];
        const KW_GLOBAL_VAR REAL* KW_RESTRICT partials2 =  partials + ptrOffsets[opIndexPtr + 3];
        DETERMINE_INDICES_4_MULTI_3_CPU();
        const KW_GLOBAL_VAR REAL* KW_RESTRICT scalingFactors = scaleFactors + ptrOffsets[opIndexPtr + 7];
        SUM_PARTIALS_PARTIALS_4_CPU();
        REAL oneOverScaling = 1.0/scalingFactors[pattern];
        for(int i = 0; i < PADDED_STATE_COUNT; i++) {
            partials3[deltaPartials + i] = sum1[i] * sum2[i] * oneOverScaling;
        }
    }
#else // GPU implementation
    DETERMINE_INDICES_4_MULTI_1_GPU();
    const KW_GLOBAL_VAR REAL* KW_RESTRICT partials1 =  partials + ptrOffsets[opIndexPtr + 2];
    const KW_GLOBAL_VAR REAL* KW_RESTRICT partials2 =  partials + ptrOffsets[opIndexPtr + 3];
    DETERMINE_INDICES_4_MULTI_2_GPU();
    const KW_GLOBAL_VAR REAL* KW_RESTRICT scalingFactors = scaleFactors + ptrOffsets[opIndexPtr + 7];
    LOAD_PARTIALS_PARTIALS_4_MULTI_PART_GPU();
    LOAD_SCALING_4_MULTI_GPU();
    LOAD_MATRIX_4_MULTI_GPU();
    if (pattern < endPattern) { // Remove padded threads!
        SUM_PARTIALS_PARTIALS_4_GPU();
        partials3[u] = sum1 * sum2 / fixedScalingFactors[patIdx * 4 + pat];
    }
#endif // FW_OPENCL_CPU
}


KW_GLOBAL_KERNEL void kernelPartialsPartialsFixedScalePartition(
                                        KW_GLOBAL_VAR REAL* KW_RESTRICT partials1,
                                        KW_GLOBAL_VAR REAL* KW_RESTRICT partials2,
                                        KW_GLOBAL_VAR REAL* KW_RESTRICT partials3,
                                        KW_GLOBAL_VAR REAL* KW_RESTRICT matrices1,
                                        KW_GLOBAL_VAR REAL* KW_RESTRICT matrices2,
                                        KW_GLOBAL_VAR REAL* KW_RESTRICT scalingFactors,
                                                                    int startPattern,
                                                                    int endPattern,
                                                                    int totalPatterns) {
#ifdef FW_OPENCL_CPU // CPU/MIC implementation
    DETERMINE_INDICES_4_PART_1_CPU();
    if (pattern < endPattern) {
        DETERMINE_INDICES_4_PART_2_CPU();
        SUM_PARTIALS_PARTIALS_4_CPU();
        REAL oneOverScaling = 1.0/scalingFactors[pattern];
        for(int i = 0; i < PADDED_STATE_COUNT; i++) {
            partials3[deltaPartials + i] = sum1[i] * sum2[i] * oneOverScaling;
        }
    }
#else // GPU implementation
    DETERMINE_INDICES_4_PART_GPU();
    LOAD_PARTIALS_PARTIALS_4_MULTI_PART_GPU();
    LOAD_SCALING_4_PART_GPU();
    LOAD_MATRIX_4_GPU();
    if (pattern < endPattern) { // Remove padded threads!
        SUM_PARTIALS_PARTIALS_4_GPU();
        partials3[u] = sum1 * sum2 / fixedScalingFactors[patIdx * 4 + pat];
    }
#endif // FW_OPENCL_CPU
}

KW_GLOBAL_KERNEL void kernelPartialsPartialsFixedScale(KW_GLOBAL_VAR REAL* KW_RESTRICT partials1,
                                                       KW_GLOBAL_VAR REAL* KW_RESTRICT partials2,
                                                       KW_GLOBAL_VAR REAL* KW_RESTRICT partials3,
                                                       KW_GLOBAL_VAR REAL* KW_RESTRICT matrices1,
                                                       KW_GLOBAL_VAR REAL* KW_RESTRICT matrices2,
                                                       KW_GLOBAL_VAR REAL* KW_RESTRICT scalingFactors,
                                                       int endPattern) {
#ifdef FW_OPENCL_CPU // CPU/MIC implementation
    DETERMINE_INDICES_4_CPU()
    SUM_PARTIALS_PARTIALS_4_CPU();
    REAL oneOverScaling = 1.0/scalingFactors[pattern];
    for(int i = 0; i < PADDED_STATE_COUNT; i++) {
        partials3[deltaPartials + i] = sum1[i] * sum2[i] * oneOverScaling;
    }
#else // GPU implementation
    DETERMINE_INDICES_4_GPU();
    LOAD_PARTIALS_PARTIALS_4_GPU();
    LOAD_SCALING_4_GPU();
    LOAD_MATRIX_4_GPU();
    if (pattern < endPattern) { // Remove padded threads!
        SUM_PARTIALS_PARTIALS_4_GPU();
        partials3[u] = sum1 * sum2 / fixedScalingFactors[patIdx * 4 + pat];
    }
#endif // FW_OPENCL_CPU
}

KW_GLOBAL_KERNEL void kernelStatesPartialsNoScaleMulti(KW_GLOBAL_VAR int* KW_RESTRICT states,
                                                       KW_GLOBAL_VAR REAL* KW_RESTRICT partials,
                                                       const KW_GLOBAL_VAR REAL* KW_RESTRICT matrices,
                                                       const KW_GLOBAL_VAR unsigned int* KW_RESTRICT ptrOffsets,
                                                       int gridStartOp,
                                                       int totalPatterns) {

#ifdef FW_OPENCL_CPU // CPU/MIC implementation
    DETERMINE_INDICES_4_MULTI_1_CPU();
    if (pattern < endPattern) {
        DETERMINE_INDICES_4_MULTI_2_CPU();
        const KW_GLOBAL_VAR int*  KW_RESTRICT states1   =  states   + ptrOffsets[opIndexPtr + 2];
        const KW_GLOBAL_VAR REAL* KW_RESTRICT partials2 =  partials + ptrOffsets[opIndexPtr + 3];
        DETERMINE_INDICES_4_MULTI_3_CPU();
        SUM_STATES_PARTIALS_4_CPU();
        for(int i = 0; i < PADDED_STATE_COUNT; i++) {
            partials3[deltaPartials + i] = sum1[i] * sum2[i];
        }
    }
#else // GPU implementation
    DETERMINE_INDICES_4_MULTI_1_GPU();
    const KW_GLOBAL_VAR int*  KW_RESTRICT states1   =  states   + ptrOffsets[opIndexPtr + 2];
    const KW_GLOBAL_VAR REAL* KW_RESTRICT partials2 =  partials + ptrOffsets[opIndexPtr + 3];
    DETERMINE_INDICES_4_MULTI_2_GPU();
    LOAD_PARTIALS_SINGLE_4_MULTI_PART_GPU();
    LOAD_MATRIX_4_MULTI_GPU();
    if (pattern < endPattern) { // Remove padded threads!
        SUM_STATES_PARTIALS_4_GPU();
        partials3[u] = sum1 * sum2;
    }
#endif // FW_OPENCL_CPU
}

KW_GLOBAL_KERNEL void kernelStatesPartialsNoScalePartition(KW_GLOBAL_VAR int*  KW_RESTRICT states1,
                                                           KW_GLOBAL_VAR REAL* KW_RESTRICT partials2,
                                                           KW_GLOBAL_VAR REAL* KW_RESTRICT partials3,
                                                           KW_GLOBAL_VAR REAL* KW_RESTRICT matrices1,
                                                           KW_GLOBAL_VAR REAL* KW_RESTRICT matrices2,
                                                                                  int startPattern,
                                                                                  int endPattern,
                                                                                  int totalPatterns) {

#ifdef FW_OPENCL_CPU // CPU/MIC implementation
    DETERMINE_INDICES_4_PART_1_CPU();
    if (pattern < endPattern) {
        DETERMINE_INDICES_4_PART_2_CPU();
        SUM_STATES_PARTIALS_4_CPU();
        for(int i = 0; i < PADDED_STATE_COUNT; i++) {
            partials3[deltaPartials + i] = sum1[i] * sum2[i];
        }
    }
#else // GPU implementation
    DETERMINE_INDICES_4_PART_GPU();
    LOAD_PARTIALS_SINGLE_4_MULTI_PART_GPU();
    LOAD_MATRIX_4_GPU();
    if (pattern < endPattern) { // Remove padded threads!
        SUM_STATES_PARTIALS_4_GPU();
        partials3[u] = sum1 * sum2;
    }
#endif // FW_OPENCL_CPU
}

KW_GLOBAL_KERNEL void kernelStatesPartialsNoScale(KW_GLOBAL_VAR int* KW_RESTRICT states1,
                                                  KW_GLOBAL_VAR REAL* KW_RESTRICT partials2,
                                                  KW_GLOBAL_VAR REAL* KW_RESTRICT partials3,
                                                  KW_GLOBAL_VAR REAL* KW_RESTRICT matrices1,
                                                  KW_GLOBAL_VAR REAL* KW_RESTRICT matrices2,
                                                  int endPattern) {
#ifdef FW_OPENCL_CPU // CPU/MIC implementation
    DETERMINE_INDICES_4_CPU()
    SUM_STATES_PARTIALS_4_CPU();
    for(int i = 0; i < PADDED_STATE_COUNT; i++) {
        partials3[deltaPartials + i] = sum1[i] * sum2[i];
    }
#else // GPU implementation
    DETERMINE_INDICES_4_GPU();
    LOAD_PARTIALS_SINGLE_4_GPU();
    LOAD_MATRIX_4_GPU();
    if (pattern < endPattern) { // Remove padded threads!
        SUM_STATES_PARTIALS_4_GPU();
        partials3[u] = sum1 * sum2;
    }
#endif // FW_OPENCL_CPU
}

KW_GLOBAL_KERNEL void kernelStatesPartialsFixedScaleMulti(
                                               KW_GLOBAL_VAR int*          KW_RESTRICT states,
                                               KW_GLOBAL_VAR REAL*         KW_RESTRICT partials,
                                         const KW_GLOBAL_VAR REAL*         KW_RESTRICT matrices,
                                         const KW_GLOBAL_VAR REAL*         KW_RESTRICT scaleFactors,
                                         const KW_GLOBAL_VAR unsigned int* KW_RESTRICT ptrOffsets,
                                                                                   int gridStartOp,
                                                                                   int totalPatterns) {

#ifdef FW_OPENCL_CPU // CPU/MIC implementation
    DETERMINE_INDICES_4_MULTI_1_CPU();
    if (pattern < endPattern) {
        DETERMINE_INDICES_4_MULTI_2_CPU();
        const KW_GLOBAL_VAR int*  KW_RESTRICT states1   =  states   + ptrOffsets[opIndexPtr + 2];
        const KW_GLOBAL_VAR REAL* KW_RESTRICT partials2 =  partials + ptrOffsets[opIndexPtr + 3];
        DETERMINE_INDICES_4_MULTI_3_CPU();
        const KW_GLOBAL_VAR REAL* KW_RESTRICT scalingFactors = scaleFactors + ptrOffsets[opIndexPtr + 7];
        SUM_STATES_PARTIALS_4_CPU();
        REAL oneOverScaling = 1.0/scalingFactors[pattern];
        for(int i = 0; i < PADDED_STATE_COUNT; i++) {
            partials3[deltaPartials + i] = sum1[i] * sum2[i] * oneOverScaling;
        }
    }
#else // GPU implementation
    DETERMINE_INDICES_4_MULTI_1_GPU();
    const KW_GLOBAL_VAR int*  KW_RESTRICT states1   =  states   + ptrOffsets[opIndexPtr + 2];
    const KW_GLOBAL_VAR REAL* KW_RESTRICT partials2 =  partials + ptrOffsets[opIndexPtr + 3];
    DETERMINE_INDICES_4_MULTI_2_GPU();
    const KW_GLOBAL_VAR REAL* KW_RESTRICT scalingFactors = scaleFactors + ptrOffsets[opIndexPtr + 7];
    LOAD_PARTIALS_SINGLE_4_MULTI_PART_GPU();
    LOAD_SCALING_4_MULTI_GPU();
    LOAD_MATRIX_4_MULTI_GPU();
    if (pattern < endPattern) { // Remove padded threads!
        SUM_STATES_PARTIALS_4_GPU();
        partials3[u] = sum1 * sum2 / fixedScalingFactors[patIdx * 4 + pat];
    }
#endif // FW_OPENCL_CPU
}

KW_GLOBAL_KERNEL void kernelStatesPartialsFixedScalePartition(
                                                    KW_GLOBAL_VAR int*  KW_RESTRICT states1,
                                                    KW_GLOBAL_VAR REAL* KW_RESTRICT partials2,
                                                    KW_GLOBAL_VAR REAL* KW_RESTRICT partials3,
                                                    KW_GLOBAL_VAR REAL* KW_RESTRICT matrices1,
                                                    KW_GLOBAL_VAR REAL* KW_RESTRICT matrices2,
                                                    KW_GLOBAL_VAR REAL* KW_RESTRICT scalingFactors,
                                                                                int startPattern,
                                                                                int endPattern,
                                                                                int totalPatterns) {

#ifdef FW_OPENCL_CPU // CPU/MIC implementation
    DETERMINE_INDICES_4_PART_1_CPU();
    if (pattern < endPattern) {
        DETERMINE_INDICES_4_PART_2_CPU();
        SUM_STATES_PARTIALS_4_CPU();
        REAL oneOverScaling = 1.0/scalingFactors[pattern];
        for(int i = 0; i < PADDED_STATE_COUNT; i++) {
            partials3[deltaPartials + i] = sum1[i] * sum2[i] * oneOverScaling;
        }
    }
#else // GPU implementation
    DETERMINE_INDICES_4_PART_GPU();
    LOAD_PARTIALS_SINGLE_4_MULTI_PART_GPU();
    LOAD_SCALING_4_PART_GPU();
    LOAD_MATRIX_4_GPU();
    if (pattern < endPattern) { // Remove padded threads!
        SUM_STATES_PARTIALS_4_GPU();
        partials3[u] = sum1 * sum2 / fixedScalingFactors[patIdx * 4 + pat];
    }
#endif // FW_OPENCL_CPU
}


KW_GLOBAL_KERNEL void kernelStatesPartialsFixedScale(KW_GLOBAL_VAR int* KW_RESTRICT states1,
                                                     KW_GLOBAL_VAR REAL* KW_RESTRICT partials2,
                                                     KW_GLOBAL_VAR REAL* KW_RESTRICT partials3,
                                                     KW_GLOBAL_VAR REAL* KW_RESTRICT matrices1,
                                                     KW_GLOBAL_VAR REAL* KW_RESTRICT matrices2,
                                                     KW_GLOBAL_VAR REAL* KW_RESTRICT scalingFactors,
                                                     int endPattern) {

#ifdef FW_OPENCL_CPU // CPU/MIC implementation
    DETERMINE_INDICES_4_CPU()
    SUM_STATES_PARTIALS_4_CPU();
    REAL oneOverScaling = 1.0/scalingFactors[pattern];
    for(int i = 0; i < PADDED_STATE_COUNT; i++) {
        partials3[deltaPartials + i] = sum1[i] * sum2[i] * oneOverScaling;
    }
#else // GPU implementation
    DETERMINE_INDICES_4_GPU();
    LOAD_PARTIALS_SINGLE_4_GPU();
    LOAD_SCALING_4_GPU();
    LOAD_MATRIX_4_GPU();
    if (pattern < endPattern) { // Remove padded threads!
        SUM_STATES_PARTIALS_4_GPU();
        partials3[u] = sum1 * sum2 / fixedScalingFactors[patIdx * 4 + pat];
    }
#endif // FW_OPENCL_CPU
}

KW_GLOBAL_KERNEL void kernelStatesStatesNoScaleMulti(KW_GLOBAL_VAR int* KW_RESTRICT states,
                                                     KW_GLOBAL_VAR REAL* KW_RESTRICT partials,
                                                     const KW_GLOBAL_VAR REAL* KW_RESTRICT matrices,
                                                     const KW_GLOBAL_VAR unsigned int* KW_RESTRICT ptrOffsets,
                                                     int gridStartOp,
                                                     int totalPatterns) {

#ifdef FW_OPENCL_CPU // CPU/MIC implementation
    DETERMINE_INDICES_4_MULTI_1_CPU();
    if (pattern < endPattern) {
        DETERMINE_INDICES_4_MULTI_2_CPU();
        const KW_GLOBAL_VAR int*  KW_RESTRICT states1   =  states   + ptrOffsets[opIndexPtr + 2];
        const KW_GLOBAL_VAR int*  KW_RESTRICT states2   =  states   + ptrOffsets[opIndexPtr + 3];
        DETERMINE_INDICES_4_MULTI_3_CPU();
        SUM_STATES_STATES_4_CPU();
    }
#else // GPU implementation
    DETERMINE_INDICES_4_MULTI_1_GPU()
    const KW_GLOBAL_VAR int*  KW_RESTRICT states1   =  states   + ptrOffsets[opIndexPtr + 2];
    const KW_GLOBAL_VAR int*  KW_RESTRICT states2   =  states   + ptrOffsets[opIndexPtr + 3];
    DETERMINE_INDICES_4_MULTI_2_GPU();
    LOAD_MATRIX_4_MULTI_GPU();
    if (pattern < endPattern) { // Remove padded threads!
        SUM_STATES_STATES_4_GPU();
    }
#endif // FW_OPENCL_CPU
}

KW_GLOBAL_KERNEL void kernelStatesStatesNoScalePartition(KW_GLOBAL_VAR int*  KW_RESTRICT states1,
                                                         KW_GLOBAL_VAR int*  KW_RESTRICT states2,
                                                         KW_GLOBAL_VAR REAL* KW_RESTRICT partials3,
                                                         KW_GLOBAL_VAR REAL* KW_RESTRICT matrices1,
                                                         KW_GLOBAL_VAR REAL* KW_RESTRICT matrices2,
                                                                                  int startPattern,
                                                                                  int endPattern,
                                                                                  int totalPatterns) {

#ifdef FW_OPENCL_CPU // CPU/MIC implementation
    DETERMINE_INDICES_4_PART_1_CPU();
    if (pattern < endPattern) {
        DETERMINE_INDICES_4_PART_2_CPU();
        SUM_STATES_STATES_4_CPU();
    }
#else // GPU implementation
    DETERMINE_INDICES_4_PART_GPU();
    LOAD_MATRIX_4_GPU();
    if (pattern < endPattern) { // Remove padded threads!
        SUM_STATES_STATES_4_GPU();
    }
#endif // FW_OPENCL_CPU
}

KW_GLOBAL_KERNEL void kernelStatesStatesNoScale(KW_GLOBAL_VAR int* KW_RESTRICT states1,
                                                KW_GLOBAL_VAR int* KW_RESTRICT states2,
                                                KW_GLOBAL_VAR REAL* KW_RESTRICT partials3,
                                                KW_GLOBAL_VAR REAL* KW_RESTRICT matrices1,
                                                KW_GLOBAL_VAR REAL* KW_RESTRICT matrices2,
                                                int endPattern) {
#ifdef FW_OPENCL_CPU // CPU/MIC implementation
    DETERMINE_INDICES_4_CPU();
    SUM_STATES_STATES_4_CPU();
#else // GPU implementation
    DETERMINE_INDICES_4_GPU();
    LOAD_MATRIX_4_GPU();
    if (pattern < endPattern) {
        SUM_STATES_STATES_4_GPU();
    }
#endif // FW_OPENCL_CPU
}

KW_GLOBAL_KERNEL void kernelStatesStatesFixedScaleMulti(
                                             KW_GLOBAL_VAR int*          KW_RESTRICT states,
                                             KW_GLOBAL_VAR REAL*         KW_RESTRICT partials,
                                       const KW_GLOBAL_VAR REAL*         KW_RESTRICT matrices,
                                       const KW_GLOBAL_VAR REAL*         KW_RESTRICT scaleFactors,
                                       const KW_GLOBAL_VAR unsigned int* KW_RESTRICT ptrOffsets,
                                                                                 int gridStartOp,
                                                                                 int totalPatterns) {

#ifdef FW_OPENCL_CPU // CPU/MIC implementation
    DETERMINE_INDICES_4_MULTI_1_CPU();
    if (pattern < endPattern) {
        DETERMINE_INDICES_4_MULTI_2_CPU();
        const KW_GLOBAL_VAR int*  KW_RESTRICT states1   =  states   + ptrOffsets[opIndexPtr + 2];
        const KW_GLOBAL_VAR int*  KW_RESTRICT states2   =  states   + ptrOffsets[opIndexPtr + 3];
        DETERMINE_INDICES_4_MULTI_3_CPU();
        const KW_GLOBAL_VAR REAL* KW_RESTRICT scalingFactors = scaleFactors + ptrOffsets[opIndexPtr + 7];
        SUM_STATES_STATES_4_SCALE_CPU();
    }
#else // GPU implementation
    DETERMINE_INDICES_4_MULTI_1_GPU();
    const KW_GLOBAL_VAR int*  KW_RESTRICT states1   =  states   + ptrOffsets[opIndexPtr + 2];
    const KW_GLOBAL_VAR int*  KW_RESTRICT states2   =  states   + ptrOffsets[opIndexPtr + 3];
    DETERMINE_INDICES_4_MULTI_2_GPU();
    const KW_GLOBAL_VAR REAL* KW_RESTRICT scalingFactors = scaleFactors + ptrOffsets[opIndexPtr + 7];
    LOAD_SCALING_4_MULTI_GPU();
    LOAD_MATRIX_4_MULTI_GPU();
    if (pattern < endPattern) { // Remove padded threads!
        SUM_STATES_STATES_4_SCALE_GPU();
    }
#endif // FW_OPENCL_CPU
}

KW_GLOBAL_KERNEL void kernelStatesStatesFixedScalePartition(
                                                     KW_GLOBAL_VAR int*  KW_RESTRICT states1,
                                                     KW_GLOBAL_VAR int*  KW_RESTRICT states2,
                                                     KW_GLOBAL_VAR REAL* KW_RESTRICT partials3,
                                                     KW_GLOBAL_VAR REAL* KW_RESTRICT matrices1,
                                                     KW_GLOBAL_VAR REAL* KW_RESTRICT matrices2,
                                                     KW_GLOBAL_VAR REAL* KW_RESTRICT scalingFactors,
                                                                                 int startPattern,
                                                                                 int endPattern,
                                                                                 int totalPatterns) {

#ifdef FW_OPENCL_CPU // CPU/MIC implementation
    DETERMINE_INDICES_4_PART_1_CPU();
    if (pattern < endPattern) {
        DETERMINE_INDICES_4_PART_2_CPU();
        SUM_STATES_STATES_4_SCALE_CPU();
    }
#else // GPU implementation
    DETERMINE_INDICES_4_PART_GPU();
    LOAD_SCALING_4_PART_GPU();
    LOAD_MATRIX_4_GPU();
    if (pattern < endPattern) { // Remove padded threads!
        SUM_STATES_STATES_4_SCALE_GPU();
    }
#endif // FW_OPENCL_CPU
}

KW_GLOBAL_KERNEL void kernelStatesStatesFixedScale(KW_GLOBAL_VAR int* KW_RESTRICT states1,
                                                   KW_GLOBAL_VAR int* KW_RESTRICT states2,
                                                   KW_GLOBAL_VAR REAL* KW_RESTRICT partials3,
                                                   KW_GLOBAL_VAR REAL* KW_RESTRICT matrices1,
                                                   KW_GLOBAL_VAR REAL* KW_RESTRICT matrices2,
                                                   KW_GLOBAL_VAR REAL* KW_RESTRICT scalingFactors,
                                                   int endPattern) {
#ifdef FW_OPENCL_CPU // CPU/MIC implementation
    DETERMINE_INDICES_4_CPU()
    SUM_STATES_STATES_4_SCALE_CPU();
#else // GPU implementation
    DETERMINE_INDICES_4_GPU();
    LOAD_SCALING_4_GPU();
    LOAD_MATRIX_4_GPU();
    if (pattern < endPattern) {
        SUM_STATES_STATES_4_SCALE_GPU();
    }
#endif // FW_OPENCL_CPU
}

// Find a scaling factor for each pattern
KW_GLOBAL_KERNEL void kernelPartialsDynamicScaling(KW_GLOBAL_VAR REAL* KW_RESTRICT allPartials,
                                                   KW_GLOBAL_VAR REAL* KW_RESTRICT scalingFactors,
                                                   int matrixCount) {
#ifdef FW_OPENCL_CPU // CPU/MIC implementation
    DETERMINE_SCALING_INDICES_4_CPU();
    FIND_MAX_PARTIALS_4_CPU();
    if (maxPartial == 0)
        maxPartial = 1.0;
    scalingFactors[pattern] = maxPartial;
    SCALE_PARTIALS_4_CPU();
#else // GPU implementation
    DETERMINE_SCALING_INDICES_4_GPU();
    FIND_MAX_PARTIALS_STATE_4_DECLARE_GPU();
    FIND_MAX_PARTIALS_STATE_4_GPU();
    // Could also parallel-reduce here.
    if (state == 0 && matrix == 0) {
        FIND_MAX_PARTIALS_MATRIX_4_GPU();
        if (matrixMax[pat] == 0)
            matrixMax[pat] = 1.0;
        scalingFactors[pattern] = matrixMax[pat]; // TODO: Are these incoherent writes?
    }
    SCALE_PARTIALS_4_GPU();
#endif // FW_OPENCL_CPU
}

KW_GLOBAL_KERNEL void kernelPartialsDynamicScalingByPartition(
                                                   KW_GLOBAL_VAR REAL* KW_RESTRICT allPartials,
                                                   KW_GLOBAL_VAR REAL* KW_RESTRICT scalingFactors,
                                                   int matrixCount,
                                                   int startPattern,
                                                   int endPattern,
                                                   int totalPatterns) {
#ifdef FW_OPENCL_CPU // CPU/MIC implementation
    DETERMINE_SCALING_INDICES_4_PARTITION_CPU();
    if (pattern < endPattern) {
        FIND_MAX_PARTIALS_4_PARTITION_CPU();
        if (maxPartial == 0)
            maxPartial = 1.0;
        scalingFactors[pattern] = maxPartial;
        SCALE_PARTIALS_4_PARTITION_CPU();
    }
#else // GPU implementation
    DETERMINE_SCALING_INDICES_4_PARTITION_GPU();
    FIND_MAX_PARTIALS_STATE_4_DECLARE_GPU();
    if (pattern < endPattern) {
        FIND_MAX_PARTIALS_STATE_4_GPU();
        if (state == 0 && matrix == 0) {
            FIND_MAX_PARTIALS_MATRIX_4_GPU();
            if (matrixMax[pat] == 0)
                matrixMax[pat] = 1.0;
            scalingFactors[pattern] = matrixMax[pat];
        }
        SCALE_PARTIALS_4_GPU();
    }
#endif // FW_OPENCL_CPU

// if (KW_LOCAL_ID_1 == 0 && KW_LOCAL_ID_0 == 0)
//     printf("pattern = %d, endPattern %d\n", pattern, endPattern);

}

KW_GLOBAL_KERNEL void kernelPartialsDynamicScalingScalersLog(KW_GLOBAL_VAR REAL* KW_RESTRICT allPartials,
                                                             KW_GLOBAL_VAR REAL* KW_RESTRICT scalingFactors,
                                                             int matrixCount) {
#ifdef FW_OPENCL_CPU // CPU/MIC implementation
    DETERMINE_SCALING_INDICES_4_CPU();
    FIND_MAX_PARTIALS_4_CPU();
    if (maxPartial == 0) {
        maxPartial = 1.0;
        scalingFactors[pattern] = 0.0;
    } else {
        scalingFactors[pattern] = log(maxPartial);
    }
    SCALE_PARTIALS_4_CPU();
#else // GPU implementation
    DETERMINE_SCALING_INDICES_4_GPU();
    FIND_MAX_PARTIALS_STATE_4_DECLARE_GPU();
    FIND_MAX_PARTIALS_STATE_4_GPU();
    if (state == 0 && matrix == 0) {
        FIND_MAX_PARTIALS_MATRIX_4_GPU();
        if (matrixMax[pat] == 0) {
            matrixMax[pat] = 1.0;
            scalingFactors[pattern] = 0.0;
        } else {
            scalingFactors[pattern] = log(matrixMax[pat]);
        }
    }
    SCALE_PARTIALS_4_GPU();
#endif // FW_OPENCL_CPU
}

KW_GLOBAL_KERNEL void kernelPartialsDynamicScalingScalersLogByPartition(
                                                   KW_GLOBAL_VAR REAL* KW_RESTRICT allPartials,
                                                   KW_GLOBAL_VAR REAL* KW_RESTRICT scalingFactors,
                                                   int matrixCount,
                                                   int startPattern,
                                                   int endPattern,
                                                   int totalPatterns) {
#ifdef FW_OPENCL_CPU // CPU/MIC implementation
    DETERMINE_SCALING_INDICES_4_PARTITION_CPU();
    if (pattern < endPattern) {
        FIND_MAX_PARTIALS_4_PARTITION_CPU();
        if (maxPartial == 0) {
            maxPartial = 1.0;
            scalingFactors[pattern] = 0.0;
        } else {
            scalingFactors[pattern] = log(maxPartial);
        }
        SCALE_PARTIALS_4_PARTITION_CPU();
    }
#else // GPU implementation
    DETERMINE_SCALING_INDICES_4_PARTITION_GPU();
    FIND_MAX_PARTIALS_STATE_4_DECLARE_GPU();
    if (pattern < endPattern) {
        FIND_MAX_PARTIALS_STATE_4_GPU();
        if (state == 0 && matrix == 0) {
            FIND_MAX_PARTIALS_MATRIX_4_GPU();
            if (matrixMax[pat] == 0) {
                matrixMax[pat] = 1.0;
                scalingFactors[pattern] = 0.0;
            } else {
                scalingFactors[pattern] = log(matrixMax[pat]);
            }
        }
        SCALE_PARTIALS_4_GPU();
    }
#endif // FW_OPENCL_CPU
}

// Find a scaling factor for each pattern and accumulate into buffer
KW_GLOBAL_KERNEL void kernelPartialsDynamicScalingAccumulate(KW_GLOBAL_VAR REAL* KW_RESTRICT allPartials,
                                                             KW_GLOBAL_VAR REAL* KW_RESTRICT scalingFactors,
                                                             KW_GLOBAL_VAR REAL* KW_RESTRICT cumulativeScaling,
                                                             int matrixCount) {
#ifdef FW_OPENCL_CPU // CPU/MIC implementation
    DETERMINE_SCALING_INDICES_4_CPU();
    FIND_MAX_PARTIALS_4_CPU();
    if (maxPartial == 0)
        maxPartial = 1.0;
    scalingFactors[pattern] = maxPartial;
    cumulativeScaling[pattern] += log(maxPartial);
    SCALE_PARTIALS_4_CPU();
#else // GPU implementation
    DETERMINE_SCALING_INDICES_4_GPU();
    FIND_MAX_PARTIALS_STATE_4_DECLARE_GPU();
    FIND_MAX_PARTIALS_STATE_4_GPU();
    if (state == 0 && matrix == 0) {
        FIND_MAX_PARTIALS_MATRIX_4_GPU();
        if (matrixMax[pat] == 0)
            matrixMax[pat] = 1.0;
        scalingFactors[pattern] = matrixMax[pat];
        #ifdef CUDA
            atomicAdd(&cumulativeScaling[pattern], log(matrixMax[pat]));
        #else
            cumulativeScaling[pattern] += log(matrixMax[pat]);
        #endif
    }
    SCALE_PARTIALS_4_GPU();
#endif // FW_OPENCL_CPU
}

KW_GLOBAL_KERNEL void kernelPartialsDynamicScalingAccumulateByPartition(
                                                   KW_GLOBAL_VAR REAL* KW_RESTRICT allPartials,
                                                   KW_GLOBAL_VAR REAL* KW_RESTRICT scalingFactors,
                                                   KW_GLOBAL_VAR REAL* KW_RESTRICT cumulativeScaling,
                                                   int matrixCount,
                                                   int startPattern,
                                                   int endPattern,
                                                   int totalPatterns) {
#ifdef FW_OPENCL_CPU // CPU/MIC implementation
    DETERMINE_SCALING_INDICES_4_PARTITION_CPU();
    if (pattern < endPattern) {
        FIND_MAX_PARTIALS_4_PARTITION_CPU();
        if (maxPartial == 0)
            maxPartial = 1.0;
        scalingFactors[pattern] = maxPartial;
        cumulativeScaling[pattern] += log(maxPartial);
        SCALE_PARTIALS_4_PARTITION_CPU();
    }
#else // GPU implementation
    DETERMINE_SCALING_INDICES_4_PARTITION_GPU();
    FIND_MAX_PARTIALS_STATE_4_DECLARE_GPU();
    if (pattern < endPattern) {
        FIND_MAX_PARTIALS_STATE_4_GPU();
        if (state == 0 && matrix == 0) {
            FIND_MAX_PARTIALS_MATRIX_4_GPU();
            if (matrixMax[pat] == 0)
                matrixMax[pat] = 1.0;
            scalingFactors[pattern] = matrixMax[pat];
            #ifdef CUDA
                atomicAdd(&cumulativeScaling[pattern], log(matrixMax[pat]));
            #else
                cumulativeScaling[pattern] += log(matrixMax[pat]);
            #endif
        }
        SCALE_PARTIALS_4_GPU();
    }
#endif // FW_OPENCL_CPU
}


KW_GLOBAL_KERNEL void kernelPartialsDynamicScalingAccumulateScalersLog(KW_GLOBAL_VAR REAL* KW_RESTRICT allPartials,
                                                                       KW_GLOBAL_VAR REAL* KW_RESTRICT scalingFactors,
                                                                       KW_GLOBAL_VAR REAL* KW_RESTRICT cumulativeScaling,
                                                                       int matrixCount) {
#ifdef FW_OPENCL_CPU // CPU/MIC implementation
    DETERMINE_SCALING_INDICES_4_CPU();
    FIND_MAX_PARTIALS_4_CPU();
    if (maxPartial == 0) {
        maxPartial = 1.0;
        scalingFactors[pattern] = 0.0;
    } else {
        REAL logMax = log(maxPartial);
        scalingFactors[pattern] = logMax;
        cumulativeScaling[pattern] += logMax;
    }
    SCALE_PARTIALS_4_CPU();
#else // GPU implementation
    DETERMINE_SCALING_INDICES_4_GPU();
    FIND_MAX_PARTIALS_STATE_4_DECLARE_GPU();
    FIND_MAX_PARTIALS_STATE_4_GPU();
    if (state == 0 && matrix == 0) {
        FIND_MAX_PARTIALS_MATRIX_4_GPU();
        if (matrixMax[pat] == 0) {
            matrixMax[pat] = 1.0;
            scalingFactors[pattern] = 0.0;
        } else {
            REAL logMax = log(matrixMax[pat]);
            scalingFactors[pattern] = logMax;
            #ifdef CUDA
                atomicAdd(&cumulativeScaling[pattern], logMax);
            #else
                cumulativeScaling[pattern] += logMax;
            #endif
        }
    }
    SCALE_PARTIALS_4_GPU();
#endif // FW_OPENCL_CPU
}

KW_GLOBAL_KERNEL void kernelPartialsDynamicScalingAccumulateScalersLogByPartition(
                                                   KW_GLOBAL_VAR REAL* KW_RESTRICT allPartials,
                                                   KW_GLOBAL_VAR REAL* KW_RESTRICT scalingFactors,
                                                   KW_GLOBAL_VAR REAL* KW_RESTRICT cumulativeScaling,
                                                   int matrixCount,
                                                   int startPattern,
                                                   int endPattern,
                                                   int totalPatterns) {
#ifdef FW_OPENCL_CPU // CPU/MIC implementation
    DETERMINE_SCALING_INDICES_4_PARTITION_CPU();
    if (pattern < endPattern) {
        FIND_MAX_PARTIALS_4_PARTITION_CPU();
        if (maxPartial == 0) {
            maxPartial = 1.0;
            scalingFactors[pattern] = 0.0;
        } else {
            REAL logMax = log(maxPartial);
            scalingFactors[pattern] = logMax;
            cumulativeScaling[pattern] += logMax;
        }
        SCALE_PARTIALS_4_PARTITION_CPU();
    }
#else // GPU implementation
    DETERMINE_SCALING_INDICES_4_PARTITION_GPU();
    FIND_MAX_PARTIALS_STATE_4_DECLARE_GPU();
    if (pattern < endPattern) {
        FIND_MAX_PARTIALS_STATE_4_GPU();
        if (state == 0 && matrix == 0) {
            FIND_MAX_PARTIALS_MATRIX_4_GPU();
            if (matrixMax[pat] == 0) {
                matrixMax[pat] = 1.0;
                scalingFactors[pattern] = 0.0;
            } else {
                REAL logMax = log(matrixMax[pat]);
                scalingFactors[pattern] = logMax;
                #ifdef CUDA
                    atomicAdd(&cumulativeScaling[pattern], logMax);
                #else
                    cumulativeScaling[pattern] += logMax;
                #endif
            }
        }
        SCALE_PARTIALS_4_GPU();
    }
#endif // FW_OPENCL_CPU
}

KW_GLOBAL_KERNEL void kernelIntegrateLikelihoods(KW_GLOBAL_VAR REAL* KW_RESTRICT dResult,
                                                 KW_GLOBAL_VAR REAL* KW_RESTRICT dRootPartials,
                                                 KW_GLOBAL_VAR REAL* KW_RESTRICT dWeights,
                                                 KW_GLOBAL_VAR REAL* KW_RESTRICT dFrequencies,
                                                 int matrixCount,
                                                 int patternCount) {
#ifdef FW_OPENCL_CPU // CPU/MIC implementation
    DETERMINE_INTEGRATE_INDICES_4_CPU();
    INTEGRATE_PARTIALS_4_CPU();
    dResult[pattern] = log(sumTotal);
#else // GPU implementation
    DETERMINE_INTEGRATE_INDICES_4_GPU();
    INTEGRATE_PARTIALS_4_GPU();
    if (state == 0)
        dResult[pattern] = log(sum[pat][state]);
#endif // FW_OPENCL_CPU
}

KW_GLOBAL_KERNEL void kernelIntegrateLikelihoodsPartition(
                                        KW_GLOBAL_VAR REAL*         KW_RESTRICT dResult,
                                        KW_GLOBAL_VAR REAL*         KW_RESTRICT dRootPartialsOrigin,
                                        KW_GLOBAL_VAR REAL*         KW_RESTRICT dWeightsOrigin,
                                        KW_GLOBAL_VAR REAL*         KW_RESTRICT dFrequenciesOrigin,
                                        KW_GLOBAL_VAR unsigned int* KW_RESTRICT dPtrOffsets,
                                                      int                       matrixCount,
                                                      int                       patternCount) {
#ifdef FW_OPENCL_CPU // CPU/MIC implementation
    DETERMINE_INTEGRATE_INDICES_4_PARTITION_CPU();
    INTEGRATE_PARTIALS_4_CPU();
    if (pattern < endPattern) {
        dResult[pattern] = log(sumTotal);
    }
#else // GPU implementation
    DETERMINE_INTEGRATE_INDICES_4_PARTITION_GPU();
    INTEGRATE_PARTIALS_4_GPU();
    if (pattern < endPattern) {
        if (state == 0) {
            dResult[pattern] = log(sum[pat][state]);
        }
    }
#endif // FW_OPENCL_CPU
}

KW_GLOBAL_KERNEL void kernelIntegrateLikelihoodsFixedScale(KW_GLOBAL_VAR REAL* KW_RESTRICT dResult,
                                                           KW_GLOBAL_VAR REAL* KW_RESTRICT dRootPartials,
                                                           KW_GLOBAL_VAR REAL* KW_RESTRICT dWeights,
                                                           KW_GLOBAL_VAR REAL* KW_RESTRICT dFrequencies,
                                                           KW_GLOBAL_VAR REAL* KW_RESTRICT dRootScalingFactors,
                                                           int matrixCount,
                                                           int patternCount) {
#ifdef FW_OPENCL_CPU // CPU/MIC implementation
    DETERMINE_INTEGRATE_INDICES_4_CPU();
    INTEGRATE_PARTIALS_4_CPU();
    dResult[pattern] = log(sumTotal) + dRootScalingFactors[pattern];
#else // GPU implementation
    DETERMINE_INTEGRATE_INDICES_4_GPU();
    INTEGRATE_PARTIALS_4_GPU();
    if (state == 0)
        dResult[pattern] = log(sum[pat][state]) + dRootScalingFactors[pattern];
#endif // FW_OPENCL_CPU
}

KW_GLOBAL_KERNEL void kernelIntegrateLikelihoodsFixedScalePartition(
                                        KW_GLOBAL_VAR REAL*         KW_RESTRICT dResult,
                                        KW_GLOBAL_VAR REAL*         KW_RESTRICT dRootPartialsOrigin,
                                        KW_GLOBAL_VAR REAL*         KW_RESTRICT dWeightsOrigin,
                                        KW_GLOBAL_VAR REAL*         KW_RESTRICT dFrequenciesOrigin,
                                        KW_GLOBAL_VAR REAL* KW_RESTRICT dRootScalingFactorsOrigin,
                                        KW_GLOBAL_VAR unsigned int* KW_RESTRICT dPtrOffsets,
                                                      int                       matrixCount,
                                                      int                       patternCount) {
#ifdef FW_OPENCL_CPU // CPU/MIC implementation
    DETERMINE_INTEGRATE_INDICES_4_PARTITION_CPU();
    const KW_GLOBAL_VAR REAL* KW_RESTRICT dRootScalingFactors  =  dRootScalingFactorsOrigin + dPtrOffsets[opIndexPtr + 5];
    INTEGRATE_PARTIALS_4_CPU();
    if (pattern < endPattern) {
        dResult[pattern] = log(sumTotal) + dRootScalingFactors[pattern];
    }
#else // GPU implementation
    DETERMINE_INTEGRATE_INDICES_4_PARTITION_GPU();
    const KW_GLOBAL_VAR REAL* KW_RESTRICT dRootScalingFactors  =  dRootScalingFactorsOrigin + dPtrOffsets[opIndexPtr + 5];
    INTEGRATE_PARTIALS_4_GPU();
    if (pattern < endPattern) {
        if (state == 0) {
            dResult[pattern] = log(sum[pat][state]) + dRootScalingFactors[pattern];
        }
    }
#endif // FW_OPENCL_CPU
}

KW_GLOBAL_KERNEL void kernelIntegrateLikelihoodsMulti(KW_GLOBAL_VAR REAL* KW_RESTRICT dResult,
                                                      KW_GLOBAL_VAR REAL* KW_RESTRICT dRootPartials,
                                                      KW_GLOBAL_VAR REAL* KW_RESTRICT dWeights,
                                                      KW_GLOBAL_VAR REAL* KW_RESTRICT dFrequencies,
                                                      int matrixCount,
                                                      int patternCount,
                                                      int takeLog) {
#ifdef FW_OPENCL_CPU // CPU/MIC implementation
    DETERMINE_INTEGRATE_INDICES_4_CPU();
    INTEGRATE_PARTIALS_4_CPU();
    if (takeLog == 0)
        dResult[pattern] = sumTotal;
    else if (takeLog == 1)
        dResult[pattern] = log(dResult[pattern] + sumTotal);
    else
        dResult[pattern] += sumTotal;
#else // GPU implementation
    DETERMINE_INTEGRATE_INDICES_4_GPU();
    INTEGRATE_PARTIALS_4_GPU();
    if (state == 0) {
        if (takeLog == 0)
            dResult[pattern] = sum[pat][state];
        else if (takeLog == 1)
            dResult[pattern] = log(dResult[pattern] + sum[pat][state]);
        else
            dResult[pattern] += sum[pat][state];
    }
#endif // FW_OPENCL_CPU
}

KW_GLOBAL_KERNEL void kernelIntegrateLikelihoodsFixedScaleMulti(KW_GLOBAL_VAR REAL* KW_RESTRICT dResult,
                                                                KW_GLOBAL_VAR REAL* KW_RESTRICT dRootPartials,
                                                                KW_GLOBAL_VAR REAL* KW_RESTRICT dWeights,
                                                                KW_GLOBAL_VAR REAL* KW_RESTRICT dFrequencies,
                                                                KW_GLOBAL_VAR REAL* KW_RESTRICT dScalingFactors,
                                                                KW_GLOBAL_VAR unsigned int* KW_RESTRICT dPtrQueue,
                                                                KW_GLOBAL_VAR REAL* KW_RESTRICT dMaxScalingFactors,
                                                                KW_GLOBAL_VAR unsigned int* KW_RESTRICT dIndexMaxScalingFactors,
                                                                int matrixCount,
                                                                int patternCount,
                                                                int subsetCount,
                                                                int subsetIndex) {
#ifdef FW_OPENCL_CPU // CPU/MIC implementation
    DETERMINE_INTEGRATE_INDICES_4_CPU();
    INTEGRATE_PARTIALS_4_CPU();
    REAL cumulativeScalingFactor = (dScalingFactors + dPtrQueue[subsetIndex])[pattern];
    if (subsetIndex == 0) {
        int indexMaxScalingFactor = 0;
        REAL maxScalingFactor = cumulativeScalingFactor;
        for (int j = 1; j < subsetCount; j++) {
            REAL tmpScalingFactor = (dScalingFactors + dPtrQueue[j])[pattern];
            if (tmpScalingFactor > maxScalingFactor) {
                indexMaxScalingFactor = j;
                maxScalingFactor = tmpScalingFactor;
            }
        }
        dIndexMaxScalingFactors[pattern] = indexMaxScalingFactor;
        dMaxScalingFactors[pattern] = maxScalingFactor;
        if (indexMaxScalingFactor != 0)
            sumTotal *= exp((REAL)(cumulativeScalingFactor - maxScalingFactor));
        dResult[pattern] = sumTotal;
    } else {
        if (subsetIndex != dIndexMaxScalingFactors[pattern])
            sumTotal *= exp((REAL)(cumulativeScalingFactor - dMaxScalingFactors[pattern]));
        if (subsetIndex == subsetCount - 1)
            dResult[pattern] = (log(dResult[pattern] + sumTotal) + dMaxScalingFactors[pattern]);
        else
            dResult[pattern] += sumTotal;
    }
#else // GPU implementation
    DETERMINE_INTEGRATE_INDICES_4_GPU();
    INTEGRATE_PARTIALS_4_GPU();
    REAL cumulativeScalingFactor = (dScalingFactors + dPtrQueue[subsetIndex])[pattern];
    if (subsetIndex == 0) {
        int indexMaxScalingFactor = 0;
        REAL maxScalingFactor = cumulativeScalingFactor;
        for (int j = 1; j < subsetCount; j++) {
            REAL tmpScalingFactor = (dScalingFactors + dPtrQueue[j])[pattern];
            if (tmpScalingFactor > maxScalingFactor) {
                indexMaxScalingFactor = j;
                maxScalingFactor = tmpScalingFactor;
            }
        }
        dIndexMaxScalingFactors[pattern] = indexMaxScalingFactor;
        dMaxScalingFactors[pattern] = maxScalingFactor;
        if (indexMaxScalingFactor != 0)
            sum[pat][state] *= exp((REAL)(cumulativeScalingFactor - maxScalingFactor));
        if (state == 0)
            dResult[pattern] = sum[pat][state];
        KW_LOCAL_FENCE;
    } else {
        if (subsetIndex != dIndexMaxScalingFactors[pattern])
            sum[pat][state] *= exp((REAL)(cumulativeScalingFactor - dMaxScalingFactors[pattern]));
        if (state == 0) {
            if (subsetIndex == subsetCount - 1)
                dResult[pattern] = (log(dResult[pattern] + sum[pat][state]) + dMaxScalingFactors[pattern]);
            else
                dResult[pattern] += sum[pat][state];
        }
    }
#endif // FW_OPENCL_CPU
}

////////////////////////////////////////////////////////////////////////////////////////////////
// edge and deriv kernels

KW_GLOBAL_KERNEL void kernelPartialsPartialsEdgeLikelihoods(KW_GLOBAL_VAR REAL* KW_RESTRICT dPartialsTmp,
                                                            KW_GLOBAL_VAR REAL* KW_RESTRICT partials1,
                                                            KW_GLOBAL_VAR REAL* KW_RESTRICT partials2,
                                                            KW_GLOBAL_VAR REAL* KW_RESTRICT matrices1,
                                                            int endPattern) {
#ifdef FW_OPENCL_CPU // CPU/MIC implementation
    DETERMINE_INDICES_4_CPU();
    SUM_PARTIALS_SINGLE_4_CPU();
    for(int i = 0; i < PADDED_STATE_COUNT; i++) {
        dPartialsTmp[deltaPartials + i] = sum1[i] * sPartials2[i];
    }
#else // GPU implementation
    DETERMINE_INDICES_4_GPU();
    LOAD_PARTIALS_PARTIALS_4_GPU();
    LOAD_MATRIX_SINGLE_4_GPU();
    if (pattern < endPattern) {
        SUM_PARTIALS_SINGLE_4_GPU();
        dPartialsTmp[u] = sum1 * sPartials2[patIdx16pat4 | state];
    }
#endif // FW_OPENCL_CPU
}

KW_GLOBAL_KERNEL void kernelPartialsPartialsEdgeLikelihoodsByPartition(
                                    KW_GLOBAL_VAR REAL* KW_RESTRICT         dPartialsTmp,
                              const KW_GLOBAL_VAR REAL* KW_RESTRICT         partials,
                                    KW_GLOBAL_VAR REAL* KW_RESTRICT         matrices,
                              const KW_GLOBAL_VAR unsigned int* KW_RESTRICT ptrOffsets,
                                                  int                       totalPatterns) {
#ifdef FW_OPENCL_CPU // CPU/MIC implementation
    DETERMINE_INDICES_4_EDGEPART_1_CPU();
    if (pattern < endPattern) {
        DETERMINE_INDICES_4_MULTI_2_CPU();
        const KW_GLOBAL_VAR REAL* KW_RESTRICT partials1 =  partials + ptrOffsets[opIndexPtr + 2];
        const KW_GLOBAL_VAR REAL* KW_RESTRICT partials2 =  partials + ptrOffsets[opIndexPtr + 3];
        const KW_GLOBAL_VAR REAL* KW_RESTRICT matrices1 =  matrices + ptrOffsets[opIndexPtr + 4];
        SUM_PARTIALS_SINGLE_4_CPU();
        for(int i = 0; i < PADDED_STATE_COUNT; i++) {
            dPartialsTmp[deltaPartials + i] = sum1[i] * sPartials2[i];
        }
    }

#else // GPU implementation
    DETERMINE_INDICES_4_EDGEPART_1_GPU();
    const KW_GLOBAL_VAR REAL* KW_RESTRICT partials1 =  partials + ptrOffsets[opIndexPtr + 2];
    const KW_GLOBAL_VAR REAL* KW_RESTRICT partials2 =  partials + ptrOffsets[opIndexPtr + 3];
          KW_GLOBAL_VAR REAL* KW_RESTRICT matrices1 =  matrices + ptrOffsets[opIndexPtr + 4];
    LOAD_PARTIALS_PARTIALS_4_MULTI_PART_GPU();
    LOAD_MATRIX_SINGLE_4_GPU();
    if (pattern < endPattern) {
        SUM_PARTIALS_SINGLE_4_GPU();
        dPartialsTmp[u] = sum1 * sPartials2[patIdx16pat4 | state];
    }
#endif // FW_OPENCL_CPU
}


KW_GLOBAL_KERNEL void kernelPartialsPartialsEdgeLikelihoodsSecondDeriv(KW_GLOBAL_VAR REAL* KW_RESTRICT dPartialsTmp,
                                                                       KW_GLOBAL_VAR REAL* KW_RESTRICT dFirstDerivTmp,
                                                                       KW_GLOBAL_VAR REAL* KW_RESTRICT dSecondDerivTmp,
                                                                       KW_GLOBAL_VAR REAL* KW_RESTRICT partials1,
                                                                       KW_GLOBAL_VAR REAL* KW_RESTRICT partials2,
                                                                       KW_GLOBAL_VAR REAL* KW_RESTRICT matrices1,
                                                                       KW_GLOBAL_VAR REAL* KW_RESTRICT dFirstDerivMatrix,
                                                                       KW_GLOBAL_VAR REAL* KW_RESTRICT dSecondDerivMatrix,
                                                                       int endPattern) {
#ifdef FW_OPENCL_CPU // CPU/MIC implementation
    DETERMINE_INDICES_4_CPU();
    SUM_PARTIALS_DERIV_4_CPU();
    for(int i = 0; i < PADDED_STATE_COUNT; i++) {
        dPartialsTmp[deltaPartials + i]    = sum1[i]           * sPartials2[i];
        dFirstDerivTmp[deltaPartials + i]  = sumFirstDeriv[i]  * sPartials2[i];
        dSecondDerivTmp[deltaPartials + i] = sumSecondDeriv[i] * sPartials2[i];
    }
#else // GPU implementation
    DETERMINE_INDICES_4_GPU();
    LOAD_PARTIALS_PARTIALS_4_GPU()
    LOAD_MATRIX_DERIV_4_GPU();
    if (pattern < endPattern) {
        SUM_PARTIALS_DERIV_4_GPU();
        dPartialsTmp[u]    = sum1           * sPartials2[patIdx16pat4 | state];
        dFirstDerivTmp[u]  = sumFirstDeriv  * sPartials2[patIdx16pat4 | state];
        dSecondDerivTmp[u] = sumSecondDeriv * sPartials2[patIdx16pat4 | state];
    }
#endif // FW_OPENCL_CPU
}

KW_GLOBAL_KERNEL void kernelStatesPartialsEdgeLikelihoods(KW_GLOBAL_VAR REAL* KW_RESTRICT dPartialsTmp,
                                                          KW_GLOBAL_VAR REAL* KW_RESTRICT partials2,
                                                          KW_GLOBAL_VAR int* KW_RESTRICT dChildStates,
                                                          KW_GLOBAL_VAR REAL* KW_RESTRICT matrices1,
                                                          int endPattern) {
#ifdef FW_OPENCL_CPU // CPU/MIC implementation
    DETERMINE_INDICES_4_CPU();
    SUM_STATES_SINGLE_4_CPU();
    for(int i = 0; i < PADDED_STATE_COUNT; i++) {
        dPartialsTmp[deltaPartials + i] = sum1[i] * sPartials2[i];
    }
#else // GPU implementation
    DETERMINE_INDICES_4_GPU();
    LOAD_PARTIALS_SINGLE_4_GPU();
    LOAD_MATRIX_SINGLE_4_GPU();
    if (pattern < endPattern) {
        SUM_STATES_SINGLE_4_GPU();
        int patIdx16pat4 = multBy16(patIdx) | (tx & 0xC);
        dPartialsTmp[u] = sum1 * sPartials2[patIdx16pat4 | state];
    }
#endif // FW_OPENCL_CPU
}

KW_GLOBAL_KERNEL void kernelStatesPartialsEdgeLikelihoodsByPartition(
                                    KW_GLOBAL_VAR REAL*         KW_RESTRICT dPartialsTmp,
                              const KW_GLOBAL_VAR REAL*         KW_RESTRICT partials,
                              const KW_GLOBAL_VAR int*          KW_RESTRICT states,
                                    KW_GLOBAL_VAR REAL*         KW_RESTRICT matrices,
                              const KW_GLOBAL_VAR unsigned int* KW_RESTRICT ptrOffsets,
                                                  int                       totalPatterns) {
#ifdef FW_OPENCL_CPU // CPU/MIC implementation
    DETERMINE_INDICES_4_EDGEPART_1_CPU();
    if (pattern < endPattern) {
        DETERMINE_INDICES_4_MULTI_2_CPU();
        const KW_GLOBAL_VAR REAL* KW_RESTRICT partials2 = partials + ptrOffsets[opIndexPtr + 2];
        const KW_GLOBAL_VAR int*  KW_RESTRICT dChildStates = states + ptrOffsets[opIndexPtr + 3];
        const KW_GLOBAL_VAR REAL* KW_RESTRICT matrices1 = matrices + ptrOffsets[opIndexPtr + 4];
        SUM_STATES_SINGLE_4_CPU();
        for(int i = 0; i < PADDED_STATE_COUNT; i++) {
            dPartialsTmp[deltaPartials + i] = sum1[i] * sPartials2[i];
        }
    }

#else // GPU implementation
    DETERMINE_INDICES_4_EDGEPART_1_GPU();
    const KW_GLOBAL_VAR REAL* KW_RESTRICT partials2 =  partials + ptrOffsets[opIndexPtr + 2];
    const KW_GLOBAL_VAR int*  KW_RESTRICT dChildStates = states + ptrOffsets[opIndexPtr + 3];
          KW_GLOBAL_VAR REAL* KW_RESTRICT matrices1 =  matrices + ptrOffsets[opIndexPtr + 4];
    LOAD_PARTIALS_SINGLE_4_MULTI_PART_GPU();
    LOAD_MATRIX_SINGLE_4_GPU();
    if (pattern < endPattern) {
        SUM_STATES_SINGLE_4_GPU();
        int patIdx16pat4 = multBy16(patIdx) | (tx & 0xC);
        dPartialsTmp[u] = sum1 * sPartials2[patIdx16pat4 | state];
    }
#endif // FW_OPENCL_CPU
}

KW_GLOBAL_KERNEL void kernelStatesPartialsEdgeLikelihoodsSecondDeriv(KW_GLOBAL_VAR REAL* KW_RESTRICT dPartialsTmp,
                                                                     KW_GLOBAL_VAR REAL* KW_RESTRICT dFirstDerivTmp,
                                                                     KW_GLOBAL_VAR REAL* KW_RESTRICT dSecondDerivTmp,
                                                                     KW_GLOBAL_VAR REAL* KW_RESTRICT partials2,
                                                                     KW_GLOBAL_VAR int* KW_RESTRICT dChildStates,
                                                                     KW_GLOBAL_VAR REAL* KW_RESTRICT matrices1,
                                                                     KW_GLOBAL_VAR REAL* KW_RESTRICT dFirstDerivMatrix,
                                                                     KW_GLOBAL_VAR REAL* KW_RESTRICT dSecondDerivMatrix,
                                                                     int endPattern) {
#ifdef FW_OPENCL_CPU // CPU/MIC implementation
    DETERMINE_INDICES_4_CPU();
    SUM_STATES_DERIV_4_CPU();
    for(int i = 0; i < PADDED_STATE_COUNT; i++) {
        dPartialsTmp[deltaPartials + i]    = sum1[i]           * sPartials2[i];
        dFirstDerivTmp[deltaPartials + i]  = sumFirstDeriv[i]  * sPartials2[i];
        dSecondDerivTmp[deltaPartials + i] = sumSecondDeriv[i] * sPartials2[i];
    }
#else // GPU implementation
    DETERMINE_INDICES_4_GPU();
    LOAD_PARTIALS_SINGLE_4_GPU();
    LOAD_MATRIX_DERIV_4_GPU();
    if (pattern < endPattern) { // Remove padded threads!
        SUM_STATES_DERIV_4_GPU();
        int patIdx16pat4 = multBy16(patIdx) | (tx & 0xC);
        dPartialsTmp[u]    = sum1           * sPartials2[patIdx16pat4 | state];
        dFirstDerivTmp[u]  = sumFirstDeriv  * sPartials2[patIdx16pat4 | state];
        dSecondDerivTmp[u] = sumSecondDeriv * sPartials2[patIdx16pat4 | state];
    }
#endif // FW_OPENCL_CPU
}

KW_GLOBAL_KERNEL void kernelIntegrateLikelihoodsSecondDeriv(KW_GLOBAL_VAR REAL* KW_RESTRICT dResult,
                                                            KW_GLOBAL_VAR REAL* KW_RESTRICT dFirstDerivResult,
                                                            KW_GLOBAL_VAR REAL* KW_RESTRICT dSecondDerivResult,
                                                            KW_GLOBAL_VAR REAL* KW_RESTRICT dRootPartials,
                                                            KW_GLOBAL_VAR REAL* KW_RESTRICT dRootFirstDeriv,
                                                            KW_GLOBAL_VAR REAL* KW_RESTRICT dRootSecondDeriv,
                                                            KW_GLOBAL_VAR REAL* KW_RESTRICT dWeights,
                                                            KW_GLOBAL_VAR REAL* KW_RESTRICT dFrequencies,
                                                            int matrixCount,
                                                            int patternCount) {
#ifdef FW_OPENCL_CPU // CPU/MIC implementation
    INTEGRATE_PARTIALS_DERIV_4_CPU();
    tmpLogLike = sumTotal;
    dResult[pattern] = log(tmpLogLike);
    tmpFirstDeriv = sumTotalD1 / tmpLogLike;
    dFirstDerivResult[pattern] = tmpFirstDeriv;
    dSecondDerivResult[pattern] = (sumTotalD2 / tmpLogLike - tmpFirstDeriv * tmpFirstDeriv);
#else // GPU implementation
    INTEGRATE_PARTIALS_DERIV_4_GPU();
    if (state == 0) {
        tmpLogLike = sum[pat][state];
        dResult[pattern] = log(tmpLogLike);
        tmpFirstDeriv = sumD1[pat][state] / tmpLogLike;
        dFirstDerivResult[pattern] = tmpFirstDeriv;
        dSecondDerivResult[pattern] = (sumD2[pat][state] / tmpLogLike - tmpFirstDeriv * tmpFirstDeriv);
    }
#endif // FW_OPENCL_CPU
}

KW_GLOBAL_KERNEL void kernelIntegrateLikelihoodsFixedScaleSecondDeriv(KW_GLOBAL_VAR REAL* KW_RESTRICT dResult,
                                                                      KW_GLOBAL_VAR REAL* KW_RESTRICT dFirstDerivResult,
                                                                      KW_GLOBAL_VAR REAL* KW_RESTRICT dSecondDerivResult,
                                                                      KW_GLOBAL_VAR REAL* KW_RESTRICT dRootPartials,
                                                                      KW_GLOBAL_VAR REAL* KW_RESTRICT dRootFirstDeriv,
                                                                      KW_GLOBAL_VAR REAL* KW_RESTRICT dRootSecondDeriv,
                                                                      KW_GLOBAL_VAR REAL* KW_RESTRICT dWeights,
                                                                      KW_GLOBAL_VAR REAL* KW_RESTRICT dFrequencies,
                                                                      KW_GLOBAL_VAR REAL* KW_RESTRICT dRootScalingFactors,
                                                                      int matrixCount,
                                                                      int patternCount) {
#ifdef FW_OPENCL_CPU // CPU/MIC implementation
    INTEGRATE_PARTIALS_DERIV_4_CPU();
    tmpLogLike = sumTotal;
    dResult[pattern] = log(tmpLogLike) + dRootScalingFactors[pattern];
    tmpFirstDeriv = sumTotalD1 / tmpLogLike;
    dFirstDerivResult[pattern] = tmpFirstDeriv;
    dSecondDerivResult[pattern] = (sumTotalD2 / tmpLogLike - tmpFirstDeriv * tmpFirstDeriv);
#else // GPU implementation
    INTEGRATE_PARTIALS_DERIV_4_GPU();
    if (state == 0) {
        tmpLogLike = sum[pat][state];
        dResult[pattern] = log(tmpLogLike) + dRootScalingFactors[pattern];
        tmpFirstDeriv = sumD1[pat][state] / tmpLogLike;
        dFirstDerivResult[pattern] = tmpFirstDeriv;
        dSecondDerivResult[pattern] = (sumD2[pat][state] / tmpLogLike - tmpFirstDeriv * tmpFirstDeriv);
    }
#endif // FW_OPENCL_CPU
}

////////////////////////////////////////////////////////////////////////////////////////////////
// scaling experiments kernels

KW_GLOBAL_KERNEL void kernelPartialsPartialsCheckScale(KW_GLOBAL_VAR REAL* partials1,
                                                                  KW_GLOBAL_VAR REAL* partials2,
                                                                  KW_GLOBAL_VAR REAL* partials3,
                                                                  KW_GLOBAL_VAR REAL* matrices1,
                                                                  KW_GLOBAL_VAR REAL* matrices2,
                                                                  KW_GLOBAL_VAR int* dRescalingTrigger,
                                                                  int endPattern) {
        REAL sum1;
        REAL sum2;
        int i;

        DETERMINE_INDICES_4_GPU();

        int patIdx16pat4 = multBy16(patIdx) | (tx & 0xC);
        int y = deltaPartialsByState + deltaPartialsByMatrix;

        KW_GLOBAL_VAR REAL* matrix1 = matrices1 + x2; // Points to *this* matrix
        KW_GLOBAL_VAR REAL* matrix2 = matrices2 + x2;

    #ifdef KERNEL_PRINT_ENABLED
        printf("matrix = %d, pat = %d for tx = %d and state = %d :  u = %d\n", matrix, pattern, tx,
               state, u);
    #endif

        // Load values into shared memory
        KW_LOCAL_MEM REAL sMatrix1[16];
        KW_LOCAL_MEM REAL sMatrix2[16];

        KW_LOCAL_MEM REAL sPartials1[PATTERN_BLOCK_SIZE * 4 * 4];
        KW_LOCAL_MEM REAL sPartials2[PATTERN_BLOCK_SIZE * 4 * 4];

        // copy PADDED_STATE_COUNT * PATTERN_BLOCK_SIZE lengthed partials
        if (pattern < endPattern) {
            sPartials1[multBy16(patIdx) | tx] = partials1[y | tx]; // All coalesced memory reads
            sPartials2[multBy16(patIdx) | tx] = partials2[y | tx];
        } else {
            sPartials1[multBy16(patIdx) | tx] = 0;
            sPartials2[multBy16(patIdx) | tx] = 0;
        }

        if (patIdx == 0 ) {
            sMatrix1[tx] = matrix1[tx]; // All coalesced memory reads
            sMatrix2[tx] = matrix2[tx];
        }

        KW_LOCAL_FENCE;

        if (pattern < endPattern) { // Remove padded threads!

            i = pat;
            sum1  = sMatrix1[multBy4(i) | state] * sPartials1[patIdx16pat4 | i];
            sum2  = sMatrix2[multBy4(i) | state] * sPartials2[patIdx16pat4 | i];

            i = (i + 1) & 0x3;
            sum1 += sMatrix1[multBy4(i) | state] * sPartials1[patIdx16pat4 | i];
            sum2 += sMatrix2[multBy4(i) | state] * sPartials2[patIdx16pat4 | i];

            i = (i + 1) & 0x3;
            sum1 += sMatrix1[multBy4(i) | state] * sPartials1[patIdx16pat4 | i];
            sum2 += sMatrix2[multBy4(i) | state] * sPartials2[patIdx16pat4 | i];

            i = (i + 1) & 0x3;
            sum1 += sMatrix1[multBy4(i) | state] * sPartials1[patIdx16pat4 | i];
            sum2 += sMatrix2[multBy4(i) | state] * sPartials2[patIdx16pat4 | i];

            REAL tmpPartial = sum1 * sum2;

            partials3[u] = tmpPartial;

            if (tmpPartial < SCALING_THRESHOLD_LOWER || tmpPartial > SCALING_THRESHOLD_UPPER)
                *dRescalingTrigger = 1;

//            union {float f; long l;} fl;
//            fl.f = sum1 * sum2;;
//
//          partials3[u] = fl.f;
//
//            int expTmp  = ((fl.l >> 23) & 0x000000ff) - 0x7e;
//
//            if (abs(expTmp) > SCALING_EXPONENT_THRESHOLD)
//                *dRescalingTrigger = 1;
        }

    }

KW_GLOBAL_KERNEL void kernelPartialsPartialsFixedCheckScale(KW_GLOBAL_VAR REAL* partials1,
                                                      KW_GLOBAL_VAR REAL* partials2,
                                                      KW_GLOBAL_VAR REAL* partials3,
                                                      KW_GLOBAL_VAR REAL* matrices1,
                                                      KW_GLOBAL_VAR REAL* matrices2,
                                                      KW_GLOBAL_VAR REAL* scalingFactors,
                                                      KW_GLOBAL_VAR int* dRescalingTrigger,
                                                      int endPattern) {
    REAL sum1;
    REAL sum2;
    int i;

    DETERMINE_INDICES_4_GPU();
    int y = deltaPartialsByState + deltaPartialsByMatrix;
    KW_GLOBAL_VAR REAL* matrix1 = matrices1 + x2; // Points to *this* matrix
    KW_GLOBAL_VAR REAL* matrix2 = matrices2 + x2;

#ifdef KERNEL_PRINT_ENABLED
    printf("matrix = %d, pat = %d for tx = %d and state = %d :  u = %d\n", matrix, pattern, tx,
           state, u);
#endif

    // Load values into shared memory
    KW_LOCAL_MEM REAL sMatrix1[16];
    KW_LOCAL_MEM REAL sMatrix2[16];

    KW_LOCAL_MEM REAL sPartials1[PATTERN_BLOCK_SIZE * 4 * 4];
    KW_LOCAL_MEM REAL sPartials2[PATTERN_BLOCK_SIZE * 4 * 4];

    KW_LOCAL_MEM REAL fixedScalingFactors[PATTERN_BLOCK_SIZE * 4];

    // copy PADDED_STATE_COUNT*PATTERN_BLOCK_SIZE lengthed partials
    if (pattern < endPattern) {
        sPartials1[patIdx * 16 + tx] = partials1[y + tx]; // All coalesced memory reads
        sPartials2[patIdx * 16 + tx] = partials2[y + tx];
    } else {
        sPartials1[patIdx * 16 + tx] = 0;
        sPartials2[patIdx * 16 + tx] = 0;
    }

    if (patIdx < 4) // need to load 4*PATTERN_BLOCK_SIZE factors for this block
        fixedScalingFactors[patIdx * PATTERN_BLOCK_SIZE + tx] =
            scalingFactors[KW_GROUP_ID_0 * PATTERN_BLOCK_SIZE * 4 + patIdx * PATTERN_BLOCK_SIZE + tx];

    if (patIdx == 0 ) {
        sMatrix1[tx] = matrix1[tx]; // All coalesced memory reads
        sMatrix2[tx] = matrix2[tx];
    }

    KW_LOCAL_FENCE;

    if (pattern < endPattern) { // Remove padded threads!

        i = pat;
        sum1  = sMatrix1[i * 4 + state] * sPartials1[patIdx * 16 + pat * 4 + i];
        sum2  = sMatrix2[i * 4 + state] * sPartials2[patIdx * 16 + pat * 4 + i];

        i = (i + 1) & 0x3;
        sum1 += sMatrix1[i * 4 + state] * sPartials1[patIdx * 16 + pat * 4 + i];
        sum2 += sMatrix2[i * 4 + state] * sPartials2[patIdx * 16 + pat * 4 + i];

        i = (i + 1) & 0x3;
        sum1 += sMatrix1[i * 4 + state] * sPartials1[patIdx * 16 + pat * 4 + i];
        sum2 += sMatrix2[i * 4 + state] * sPartials2[patIdx * 16 + pat * 4 + i];

        i = (i + 1) & 0x3;
        sum1 += sMatrix1[i * 4 + state] * sPartials1[patIdx * 16 + pat * 4 + i];
        sum2 += sMatrix2[i * 4 + state] * sPartials2[patIdx * 16 + pat * 4 + i];

        REAL tmpPartial = sum1 * sum2 * fixedScalingFactors[patIdx * 4 + pat];

        partials3[u] = tmpPartial;

        if (tmpPartial < SCALING_THRESHOLD_LOWER || tmpPartial > SCALING_THRESHOLD_UPPER)
            *dRescalingTrigger = 1;

    }

}

KW_GLOBAL_KERNEL void kernelPartialsPartialsAutoScale(KW_GLOBAL_VAR REAL* partials1,
                                                KW_GLOBAL_VAR REAL* partials2,
                                                KW_GLOBAL_VAR REAL* partials3,
                                                KW_GLOBAL_VAR REAL* matrices1,
                                                KW_GLOBAL_VAR REAL* matrices2,
                                                KW_GLOBAL_VAR signed char* scalingFactors,
                                                int endPattern) {
    REAL sum1;
    REAL sum2;
    int i;

    DETERMINE_INDICES_4_GPU();

    int patIdx16pat4 = multBy16(patIdx) | (tx & 0xC);
    int y = deltaPartialsByState + deltaPartialsByMatrix;
    int myIdx = multBy16(patIdx) + tx; // threadId in block

    KW_GLOBAL_VAR REAL* matrix1 = matrices1 + x2; // Points to *this* matrix
    KW_GLOBAL_VAR REAL* matrix2 = matrices2 + x2;

#ifdef KERNEL_PRINT_ENABLED
    printf("matrix = %d, pat = %d for tx = %d and state = %d :  u = %d\n", matrix, pattern, tx,
           state, u);
#endif

    // Load values into shared memory
    KW_LOCAL_MEM REAL sMatrix1[16];
    KW_LOCAL_MEM REAL sMatrix2[16];

    KW_LOCAL_MEM REAL sPartials1[PATTERN_BLOCK_SIZE * 4 * 4];
    KW_LOCAL_MEM REAL sPartials2[PATTERN_BLOCK_SIZE * 4 * 4];

    // copy PADDED_STATE_COUNT * PATTERN_BLOCK_SIZE lengthed partials
    if (pattern < endPattern) {
        sPartials1[multBy16(patIdx) | tx] = partials1[y | tx]; // All coalesced memory reads
        sPartials2[multBy16(patIdx) | tx] = partials2[y | tx];
    } else {
        sPartials1[multBy16(patIdx) | tx] = 0;
        sPartials2[multBy16(patIdx) | tx] = 0;
    }

    if (patIdx == 0 ) {
        sMatrix1[tx] = matrix1[tx]; // All coalesced memory reads
        sMatrix2[tx] = matrix2[tx];
    }

    KW_LOCAL_FENCE;

    i = pat;
    sum1  = sMatrix1[multBy4(i) | state] * sPartials1[patIdx16pat4 | i];
    sum2  = sMatrix2[multBy4(i) | state] * sPartials2[patIdx16pat4 | i];

    i = (i + 1) & 0x3;
    sum1 += sMatrix1[multBy4(i) | state] * sPartials1[patIdx16pat4 | i];
    sum2 += sMatrix2[multBy4(i) | state] * sPartials2[patIdx16pat4 | i];

    i = (i + 1) & 0x3;
    sum1 += sMatrix1[multBy4(i) | state] * sPartials1[patIdx16pat4 | i];
    sum2 += sMatrix2[multBy4(i) | state] * sPartials2[patIdx16pat4 | i];

    i = (i + 1) & 0x3;
    sum1 += sMatrix1[multBy4(i) | state] * sPartials1[patIdx16pat4 | i];
    sum2 += sMatrix2[multBy4(i) | state] * sPartials2[patIdx16pat4 | i];

    REAL tmpPartial = sum1 * sum2;
    int expTmp;
    REAL sigTmp = frexp(tmpPartial, &expTmp);

    KW_LOCAL_FENCE;

    if (pattern < endPattern) {
        if (abs(expTmp) > SCALING_EXPONENT_THRESHOLD) {
            // now using sPartials2 to hold scaling trigger boolean
            sPartials2[patIdx16pat4] = 1;
        } else {
            partials3[u] = tmpPartial;
            sPartials2[patIdx16pat4] = 0;
            sPartials1[myIdx] = 0;
        }
    }

    KW_LOCAL_FENCE;

    int scalingActive = sPartials2[patIdx16pat4];

    if (scalingActive) {
        // now using sPartials1 to store max unscaled partials3
        sPartials1[myIdx] = tmpPartial;
    }

    KW_LOCAL_FENCE;

    // Unrolled parallel max-reduction
    if (scalingActive && state < 2) {
        REAL compare = sPartials1[myIdx + 2];
        if (compare >  sPartials1[myIdx])
            sPartials1[myIdx] = compare;
    }

    KW_LOCAL_FENCE;

    if (scalingActive && state < 1) {
        REAL maxPartial = sPartials1[myIdx + 1];
        if (maxPartial < sPartials1[myIdx])
            maxPartial = sPartials1[myIdx];
        int expMax;
        frexp(maxPartial, &expMax);
        sPartials1[myIdx] = expMax;
    }

    KW_LOCAL_FENCE;

    if (scalingActive)
        partials3[u] = ldexp(sigTmp, expTmp - sPartials1[patIdx16pat4]);

    if ((myIdx < PATTERN_BLOCK_SIZE * 4) && (myIdx + (KW_GROUP_ID_0 * PATTERN_BLOCK_SIZE * 4) < endPattern))
        scalingFactors[(KW_GROUP_ID_0 * PATTERN_BLOCK_SIZE * 4) + (matrix * endPattern) + myIdx] = sPartials1[multBy4(myIdx)];
}


KW_GLOBAL_KERNEL void kernelPartialsDynamicScalingAccumulateReciprocal(KW_GLOBAL_VAR REAL* allPartials,
                                                       KW_GLOBAL_VAR REAL* scalingFactors,
                                                       KW_GLOBAL_VAR REAL* cumulativeScaling,
                                                       int matrixCount) {
    int tx = KW_LOCAL_ID_0;

    int state = tx & 0x3;
    int pat = tx >> 2;

    int patIdx = KW_GROUP_ID_0;

    int pattern = (patIdx << 2) + pat;
    int matrix = KW_LOCAL_ID_1;
    // TODO: Assumes matrixCount < MATRIX_BLOCK_SIZ

    // Patterns are always padded, so no reading/writing past end possible
    // Find start of patternBlock for thread-block
    int partialsOffset = (matrix * KW_NUM_GROUPS_0 + patIdx) << 4; //* 16;

    KW_LOCAL_MEM REAL partials[MATRIX_BLOCK_SIZE][16]; // 4 patterns at a time
    KW_LOCAL_MEM REAL storedPartials[MATRIX_BLOCK_SIZE][16];

    KW_LOCAL_MEM REAL matrixMax[4];

    if (matrix < matrixCount)
        partials[matrix][tx] = allPartials[partialsOffset + tx];

    storedPartials[matrix][tx] = partials[matrix][tx];

    KW_LOCAL_FENCE;

    // Unrolled parallel max-reduction
    if (state < 2) {
        REAL compare1 = partials[matrix][tx];
        REAL compare2 = partials[matrix][tx + 2];
        if (compare2 > compare1)
            partials[matrix][tx] = compare2;
    }
    KW_LOCAL_FENCE;

    if (state < 1) {
        REAL compare1 = partials[matrix][tx];
        REAL compare2 = partials[matrix][tx + 1];
        if (compare2 > compare1)
            partials[matrix][tx] = compare2;
    }
    KW_LOCAL_FENCE;

    // Could also parallel-reduce here.
    if (state == 0 && matrix == 0) {
        matrixMax[pat] = 0;
        int m;
        for(m = 0; m < matrixCount; m++) {
            if (partials[m][tx] > matrixMax[pat])
                matrixMax[pat] = partials[m][tx];
        }

        if (matrixMax[pat] == 0)
            matrixMax[pat] = 1.0;

        scalingFactors[pattern] = 1/matrixMax[pat];
        cumulativeScaling[pattern] += log(matrixMax[pat]);
    }

    KW_LOCAL_FENCE;

    if (matrix < matrixCount)
        allPartials[partialsOffset + tx] = storedPartials[matrix][tx] / matrixMax[pat];

}

KW_GLOBAL_KERNEL void kernelPartialsDynamicScalingAccumulateDifference(KW_GLOBAL_VAR REAL* allPartials,
                                                                 KW_GLOBAL_VAR REAL* scalingFactors,
                                                                 KW_GLOBAL_VAR REAL* existingScalingFactors,
                                                                 KW_GLOBAL_VAR REAL* cumulativeScaling,
                                                                 int matrixCount) {
    int tx = KW_LOCAL_ID_0;

    int state = tx & 0x3;
    int pat = tx >> 2;

    int patIdx = KW_GROUP_ID_0;

    int pattern = (patIdx << 2) + pat;
    int matrix = KW_LOCAL_ID_1;
    // TODO: Assumes matrixCount < MATRIX_BLOCK_SIZ

    // Patterns are always padded, so no reading/writing past end possible
    // Find start of patternBlock for thread-block
    int partialsOffset = (matrix * KW_NUM_GROUPS_0 + patIdx) << 4; //* 16;

    KW_LOCAL_MEM REAL partials[MATRIX_BLOCK_SIZE][16]; // 4 patterns at a time
    KW_LOCAL_MEM REAL storedPartials[MATRIX_BLOCK_SIZE][16];

    KW_LOCAL_MEM REAL matrixMax[4];

    if (matrix < matrixCount)
        partials[matrix][tx] = allPartials[partialsOffset + tx];

    storedPartials[matrix][tx] = partials[matrix][tx];

    KW_LOCAL_FENCE;

    // Unrolled parallel max-reduction
    if (state < 2) {
        REAL compare1 = partials[matrix][tx];
        REAL compare2 = partials[matrix][tx + 2];
        if (compare2 > compare1)
            partials[matrix][tx] = compare2;
    }
    KW_LOCAL_FENCE;

    if (state < 1) {
        REAL compare1 = partials[matrix][tx];
        REAL compare2 = partials[matrix][tx + 1];
        if (compare2 > compare1)
            partials[matrix][tx] = compare2;
    }
    KW_LOCAL_FENCE;

    // Could also parallel-reduce here.
    if (state == 0 && matrix == 0) {
        matrixMax[pat] = 0;
        int m;
        for(m = 0; m < matrixCount; m++) {
            if (partials[m][tx] > matrixMax[pat])
                matrixMax[pat] = partials[m][tx];
        }

        if (matrixMax[pat] == 0)
            matrixMax[pat] = 1.0;

        REAL currentFactors = existingScalingFactors[pattern];
        scalingFactors[pattern] = 1/matrixMax[pat] * currentFactors;
        cumulativeScaling[pattern] += (log(matrixMax[pat]));
    }

    KW_LOCAL_FENCE;

    if (matrix < matrixCount)
        allPartials[partialsOffset + tx] = storedPartials[matrix][tx] / matrixMax[pat];

}

KW_GLOBAL_KERNEL void kernelIntegrateLikelihoodsAutoScaling(KW_GLOBAL_VAR REAL* dResult,
                                                     KW_GLOBAL_VAR REAL* dRootPartials,
                                                     KW_GLOBAL_VAR REAL* dWeights,
                                                     KW_GLOBAL_VAR REAL* dFrequencies,
                                                     KW_GLOBAL_VAR int* dRootScalingFactors,
                                                     int matrixCount,
                                                     int patternCount) {
     int state   = KW_LOCAL_ID_0;
    int pat = KW_LOCAL_ID_1;
    int pattern = KW_GROUP_ID_0 * LIKE_PATTERN_BLOCK_SIZE + KW_LOCAL_ID_1;

    KW_LOCAL_MEM REAL stateFreq[4];

    // TODO: Currently assumes MATRIX_BLOCK_SIZE >= matrixCount
    KW_LOCAL_MEM REAL matrixProp[MATRIX_BLOCK_SIZE];
    KW_LOCAL_MEM REAL sum[LIKE_PATTERN_BLOCK_SIZE][4];

    // Load shared memory

    if (pat == 0) {
        stateFreq[state] = dFrequencies[state];
    }

    sum[pat][state] = 0;

    // TODO: Assumes matrixCount < LIKE_PATTERN_BLOCK_SIZE * 4
    if (pat * 4 + state < matrixCount) {
        matrixProp[pat * 4 + state] = dWeights[pat * 4 + state];
    }

    KW_LOCAL_FENCE;

    int u = state + pattern * PADDED_STATE_COUNT;
    int delta = patternCount * PADDED_STATE_COUNT;

    short maxScaleFactor = dRootScalingFactors[pattern];
    for(int r = 1; r < matrixCount; r++) {
        int tmpFactor = dRootScalingFactors[pattern + (r * patternCount)];
        if (tmpFactor > maxScaleFactor)
            maxScaleFactor = tmpFactor;
    }

    for(int r = 0; r < matrixCount; r++) {
        int tmpFactor = dRootScalingFactors[pattern + (r * patternCount)];
        if (tmpFactor != maxScaleFactor) {
            // TODO: verify which of the two methods below is faster
            int expTmp;
            sum[pat][state] += ldexp(frexp(dRootPartials[u + delta * r], &expTmp), expTmp + (tmpFactor - maxScaleFactor)) * matrixProp[r];
//            sum[pat][state] += dRootPartials[u + delta * r] * pow(2.0, tmpFactor - maxScaleFactor) * matrixProp[r];
        } else {
            sum[pat][state] += dRootPartials[u + delta * r] * matrixProp[r];
        }
    }

    sum[pat][state] *= stateFreq[state];
    KW_LOCAL_FENCE;
    if (state < 2)
        sum[pat][state] += sum[pat][state + 2];
    KW_LOCAL_FENCE;
    if (state < 1) {
        sum[pat][state] += sum[pat][state + 1];
    }

    if (state == 0)
        dResult[pattern] = (log(sum[pat][state]) + (M_LN2 * maxScaleFactor));
}

#ifdef CUDA
    #include "kernels4Derivatives.cu"
#endif // CUDA

#ifdef CUDA
} // extern "C"
#endif //CUDA
