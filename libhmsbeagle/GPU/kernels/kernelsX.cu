#include "hip/hip_runtime.h"
/*
 *
 * Copyright 2009 Phylogenetic Likelihood Working Group
 *
 * This file is part of BEAGLE.
 *
 * Use of this source code is governed by an MIT-style
 * license that can be found in the LICENSE file or at
 * https://opensource.org/licenses/MIT.
 *
 * @author Marc Suchard
 * @author Daniel Ayres
 */

#ifdef CUDA
    #include "libhmsbeagle/GPU/GPUImplDefs.h"
    #include "libhmsbeagle/GPU/kernels/kernelsAll.cu" // This file includes the non-state-count specific kernels
    extern "C" {
#endif

///////////////////////////////////////////////////////////////////////////////
// kernel macros CPU

#define DETERMINE_INDICES_X_CPU()\
    int state = KW_LOCAL_ID_0;\
    int patIdx = get_global_id(1);\
    int pattern = __umul24(KW_GROUP_ID_0,PATTERN_BLOCK_SIZE) + patIdx;\
    int matrix = KW_GROUP_ID_2;\
    int patternCount = totalPatterns;\
    int deltaPartialsByState = pattern * PADDED_STATE_COUNT;\
    int deltaPartialsByMatrix = matrix * PADDED_STATE_COUNT * patternCount;\
    int deltaMatrix = matrix * PADDED_STATE_COUNT * PADDED_STATE_COUNT;\
    int u = state + deltaPartialsByState + deltaPartialsByMatrix;

#define SUM_PARTIALS_PARTIALS_X_CPU()\
    REAL sum1 = 0, sum2 = 0;\
    int deltaPartials = deltaPartialsByMatrix + deltaPartialsByState;\
    KW_GLOBAL_VAR REAL* KW_RESTRICT sMatrix1 = matrices1 + deltaMatrix;\
    KW_GLOBAL_VAR REAL* KW_RESTRICT sMatrix2 = matrices2 + deltaMatrix;\
    KW_GLOBAL_VAR REAL* KW_RESTRICT sPartials1 = partials1 + deltaPartials;\
    KW_GLOBAL_VAR REAL* KW_RESTRICT sPartials2 = partials2 + deltaPartials;\
    for(int i = 0; i < PADDED_STATE_COUNT; i++) {\
        FMA(sMatrix1[i * PADDED_STATE_COUNT + state],  sPartials1[i], sum1);\
        FMA(sMatrix2[i * PADDED_STATE_COUNT + state],  sPartials2[i], sum2);\
    }

#define SUM_STATES_PARTIALS_X_CPU()\
    REAL sum1 = 0, sum2 = 0;\
    int deltaPartials = deltaPartialsByMatrix + deltaPartialsByState;\
    KW_GLOBAL_VAR REAL* KW_RESTRICT sMatrix1 = matrices1 + deltaMatrix;\
    KW_GLOBAL_VAR REAL* KW_RESTRICT sMatrix2 = matrices2 + deltaMatrix;\
    KW_GLOBAL_VAR REAL* KW_RESTRICT sPartials2 = partials2 + deltaPartials;\
    int state1 = states1[pattern];\
    if (state1 < PADDED_STATE_COUNT)\
        sum1 = sMatrix1[state1 * PADDED_STATE_COUNT + state];\
    else\
        sum1 = 1.0;\
    for(int i = 0; i < PADDED_STATE_COUNT; i++) {\
        FMA(sMatrix2[i * PADDED_STATE_COUNT + state],  sPartials2[i], sum2);\
    }

#define FIND_MAX_PARTIALS_X_CPU()\
    int patIdx = KW_LOCAL_ID_0;\
    int pattern = KW_GROUP_ID_0 * PATTERN_BLOCK_SIZE + patIdx;\
    int deltaPartialsByState = pattern * PADDED_STATE_COUNT;\
    REAL max = 0;\
    for(int m = 0; m < matrixCount; m++) {\
        int deltaPartialsByMatrix = m * PADDED_STATE_COUNT * PATTERN_BLOCK_SIZE * KW_NUM_GROUPS_0;\
        int deltaPartials = deltaPartialsByMatrix + deltaPartialsByState;\
        for(int i = 0; i < PADDED_STATE_COUNT; i++) {\
            REAL iPartial = allPartials[deltaPartials + i];\
            if (iPartial > max)\
                max = iPartial;\
        }\
    }

#define SCALE_PARTIALS_X_CPU()\
    for(int m = 0; m < matrixCount; m++) {\
        int deltaPartialsByMatrix = m * PADDED_STATE_COUNT * PATTERN_BLOCK_SIZE * KW_NUM_GROUPS_0;\
        int deltaPartials = deltaPartialsByMatrix + deltaPartialsByState;\
        for(int i = 0; i < PADDED_STATE_COUNT; i++) {\
            allPartials[deltaPartials + i] /= max;\
        }\
    }

#define INTEGRATE_PARTIALS_X_CPU()\
    int pattern = KW_GROUP_ID_0;\
    int u = pattern * PADDED_STATE_COUNT;\
    int delta = patternCount * PADDED_STATE_COUNT;\
    REAL sumTotal = 0;\
    for (int i = 0; i < PADDED_STATE_COUNT; i++) {\
        REAL sumState = dRootPartials[i + u] * dWeights[0];\
        for(int r = 1; r < matrixCount; r++) {\
            FMA(dRootPartials[i + u + delta * r],  dWeights[r], sumState);\
        }\
        sumState *= dFrequencies[i];\
        sumTotal += sumState;\
    }

#define INTEGRATE_PARTIALS_DERIV_X_CPU()\
    int pattern = KW_GROUP_ID_0;\
    int u = pattern * PADDED_STATE_COUNT;\
    int delta = patternCount * PADDED_STATE_COUNT;\
    REAL sumTotal = 0, sumTotalD1 = 0, sumTotalD2 = 0;\
    REAL tmpLogLike, tmpFirstDeriv;\
    for (int i = 0; i < PADDED_STATE_COUNT; i++) {\
        REAL sumState = dRootPartials[   i + u] * dWeights[0];\
        REAL sumD1    = dRootFirstDeriv[ i + u] * dWeights[0];\
        REAL sumD2    = dRootSecondDeriv[i + u] * dWeights[0];\
        for(int r = 1; r < matrixCount; r++) {\
            FMA(dRootPartials[   i + u + delta * r],  dWeights[r], sumState);\
            FMA(dRootFirstDeriv[ i + u + delta * r],  dWeights[r], sumD1);\
            FMA(dRootSecondDeriv[i + u + delta * r],  dWeights[r], sumD2);\
        }\
        sumState   *= dFrequencies[i];\
        sumD1      *= dFrequencies[i];\
        sumD2      *= dFrequencies[i];\
        sumTotal   += sumState;\
        sumTotalD1 += sumD1;\
        sumTotalD2 += sumD2;\
    }

///////////////////////////////////////////////////////////////////////////////
// kernel macros GPU

#define DETERMINE_INDICES_X_GPU()\
    int state = KW_LOCAL_ID_0;\
    int patIdx = KW_LOCAL_ID_1;\
    int pattern = __umul24(KW_GROUP_ID_0,PATTERN_BLOCK_SIZE) + patIdx;\
    int matrix = KW_GROUP_ID_1;\
    int patternCount = totalPatterns;\
    int deltaPartialsByState = pattern * PADDED_STATE_COUNT;\
    int deltaPartialsByMatrix = matrix * PADDED_STATE_COUNT * patternCount;\
    int deltaMatrix = matrix * PADDED_STATE_COUNT * PADDED_STATE_COUNT;\
    int u = state + deltaPartialsByState + deltaPartialsByMatrix;

#define LOAD_SCALING_X_GPU()\
    KW_LOCAL_MEM REAL fixedScalingFactors[PATTERN_BLOCK_SIZE];\
    if (patIdx == 0 && state < PATTERN_BLOCK_SIZE ) {\
        /* TODO: If PATTERN_BLOCK_SIZE > PADDED_STATE_COUNT, there is a bug here */\
        fixedScalingFactors[state] = scalingFactors[KW_GROUP_ID_0 * PATTERN_BLOCK_SIZE + state];\
    }

#define SUM_PARTIALS_PARTIALS_X_GPU()\
    KW_GLOBAL_VAR REAL* KW_RESTRICT matrix1 = matrices1 + deltaMatrix; /* Points to *this* matrix */\
    KW_GLOBAL_VAR REAL* KW_RESTRICT matrix2 = matrices2 + deltaMatrix;\
    /* Load values into shared memory */\
    KW_LOCAL_MEM REAL sMatrix1[BLOCK_PEELING_SIZE][PADDED_STATE_COUNT];\
    KW_LOCAL_MEM REAL sMatrix2[BLOCK_PEELING_SIZE][PADDED_STATE_COUNT];\
    KW_LOCAL_MEM REAL sPartials1[PATTERN_BLOCK_SIZE][PADDED_STATE_COUNT];\
    KW_LOCAL_MEM REAL sPartials2[PATTERN_BLOCK_SIZE][PADDED_STATE_COUNT];\
    int y = deltaPartialsByState + deltaPartialsByMatrix;\
    /* copy PADDED_STATE_COUNT*PATTERN_BLOCK_SIZE lengthed partials */\
    /* These are all coherent global memory reads; checked in Profiler */\
    if (pattern < totalPatterns) {\
        sPartials1[patIdx][state] = partials1[y + state];\
        sPartials2[patIdx][state] = partials2[y + state];\
    } else {\
        sPartials1[patIdx][state] = 0;\
        sPartials2[patIdx][state] = 0;\
    }\
    REAL sum1 = 0, sum2 = 0;\
    for (int i = 0; i < PADDED_STATE_COUNT; i += BLOCK_PEELING_SIZE) {\
        /* load one row of matrices */\
        if (patIdx < BLOCK_PEELING_SIZE) {\
            /* These are all coherent global memory reads. */\
            sMatrix1[patIdx][state] = matrix1[patIdx * PADDED_STATE_COUNT + state];\
            sMatrix2[patIdx][state] = matrix2[patIdx * PADDED_STATE_COUNT + state];\
            /* sMatrix now filled with starting in state and ending in i */\
            matrix1 += BLOCK_PEELING_SIZE * PADDED_STATE_COUNT;\
            matrix2 += BLOCK_PEELING_SIZE * PADDED_STATE_COUNT;\
        }\
        KW_LOCAL_FENCE;\
        for(int j = 0; j < BLOCK_PEELING_SIZE; j++) {\
            FMA(sMatrix1[j][state],  sPartials1[patIdx][i + j], sum1);\
            FMA(sMatrix2[j][state],  sPartials2[patIdx][i + j], sum2);\
        }\
        KW_LOCAL_FENCE;\
    }

#define SUM_STATES_PARTIALS_X_GPU()\
    KW_LOCAL_MEM REAL sMatrix2[BLOCK_PEELING_SIZE][PADDED_STATE_COUNT];\
    KW_LOCAL_MEM REAL sPartials2[PATTERN_BLOCK_SIZE][PADDED_STATE_COUNT];\
    int y = deltaPartialsByState + deltaPartialsByMatrix;\
    if (pattern < totalPatterns) {\
        sPartials2[patIdx][state] = partials2[y + state];\
    } else {\
        sPartials2[patIdx][state] = 0;\
    }\
    KW_GLOBAL_VAR REAL* KW_RESTRICT matrix2 = matrices2 + deltaMatrix;\
    REAL sum1 = 0, sum2 = 0;\
    if (pattern < totalPatterns) {\
        int state1 = states1[pattern]; /* Coalesced; no need to share */\
        KW_GLOBAL_VAR REAL* KW_RESTRICT matrix1 = matrices1 + deltaMatrix + state1 * PADDED_STATE_COUNT;\
        if (state1 < PADDED_STATE_COUNT)\
            sum1 = matrix1[state];\
        else\
            sum1 = 1.0;\
    }\
    for (int i = 0; i < PADDED_STATE_COUNT; i += BLOCK_PEELING_SIZE) {\
        if (patIdx < BLOCK_PEELING_SIZE) {\
            sMatrix2[patIdx][state] = matrix2[patIdx * PADDED_STATE_COUNT + state];\
            matrix2 += BLOCK_PEELING_SIZE * PADDED_STATE_COUNT;\
        }\
        KW_LOCAL_FENCE;\
        for(int j = 0; j < BLOCK_PEELING_SIZE; j++) {\
            FMA(sMatrix2[j][state], sPartials2[patIdx][i + j], sum2);\
        }\
        KW_LOCAL_FENCE;\
    }

#define LOAD_PARTIALS_SCALING_X_GPU()\
    int state = KW_LOCAL_ID_0;\
    int matrix = KW_LOCAL_ID_1;\
    int pattern = KW_GROUP_ID_0;\
    int patternCount = KW_NUM_GROUPS_0;\
    int offsetPartials = matrix * patternCount * PADDED_STATE_COUNT\
                         + pattern * PADDED_STATE_COUNT + state;\
    /* TODO: Currently assumes MATRIX_BLOCK_SIZE > matrixCount; FIX!!! */\
    KW_LOCAL_MEM REAL partials[MATRIX_BLOCK_SIZE][PADDED_STATE_COUNT];\
    KW_LOCAL_MEM REAL storedPartials[MATRIX_BLOCK_SIZE][PADDED_STATE_COUNT];\
    KW_LOCAL_MEM REAL max;\
    if (matrix < matrixCount)\
        partials[matrix][state] = allPartials[offsetPartials];\
    else\
        partials[matrix][state] = 0;\
    storedPartials[matrix][state] = partials[matrix][state];\
    KW_LOCAL_FENCE;

#define FIND_MAX_PARTIALS_STATE_POWER_OF_TWO_X_GPU()\
    /* parallelized reduction, only works for powers-of-2 */\
    for (int i = PADDED_STATE_COUNT / 2; i > 0; i >>= 1) {\
        if (state < i) {\
            REAL compare1 = partials[matrix][state];\
            REAL compare2 = partials[matrix][state + i];\
            if (compare2 > compare1)\
                partials[matrix][state] = compare2;\
        }\
        KW_LOCAL_FENCE;\
    }

#define FIND_MAX_PARTIALS_STATE_X_GPU()\
    /* not power-of-2 */\
    for (int i = SMALLEST_POWER_OF_TWO / 2; i > 0; i >>= 1) {\
        if (state < i && state + i < PADDED_STATE_COUNT ) {\
            REAL compare1 = partials[matrix][state];\
            REAL compare2 = partials[matrix][state + i];\
            if (compare2 > compare1)\
               partials[matrix][state] = compare2;\
        }\
        KW_LOCAL_FENCE;\
    }

#define FIND_MAX_PARTIALS_MATRIX_X_GPU()\
    max = 0;\
    for(int m = 0; m < matrixCount; m++) {\
        if (partials[m][0] > max)\
            max = partials[m][0];\
    }

#define SCALE_PARTIALS_X_GPU()\
    KW_LOCAL_FENCE;\
    if (matrix < matrixCount)\
        allPartials[offsetPartials] = storedPartials[matrix][state] / max;

#define INTEGRATE_PARTIALS_X_GPU()\
    int state   = KW_LOCAL_ID_0;\
    int pattern = KW_GROUP_ID_0;\
    KW_LOCAL_MEM REAL stateFreq[PADDED_STATE_COUNT];\
    /* TODO: Currently assumes MATRIX_BLOCK_SIZE >> matrixCount */\
    KW_LOCAL_MEM REAL matrixProp[MATRIX_BLOCK_SIZE];\
    KW_LOCAL_MEM REAL sum[PADDED_STATE_COUNT];\
    /* Load shared memory */\
    stateFreq[state] = dFrequencies[state];\
    sum[state] = 0;\
    for(int matrixEdge = 0; matrixEdge < matrixCount; matrixEdge += PADDED_STATE_COUNT) {\
        int x = matrixEdge + state;\
        if (x < matrixCount)\
            matrixProp[x] = dWeights[x];\
    }\
    KW_LOCAL_FENCE;\
    int u = state + pattern * PADDED_STATE_COUNT;\
    int delta = patternCount * PADDED_STATE_COUNT;\
    for(int r = 0; r < matrixCount; r++) {\
        FMA(dRootPartials[u + delta * r], matrixProp[r], sum[state]);\
    }\
    sum[state] *= stateFreq[state];\
    KW_LOCAL_FENCE;

#define INTEGRATE_PARTIALS_DERIV_X_GPU()\
    int state   = KW_LOCAL_ID_0;\
    int pattern = KW_GROUP_ID_0;\
    REAL tmpLogLike, tmpFirstDeriv;\
    KW_LOCAL_MEM REAL stateFreq[PADDED_STATE_COUNT];\
    KW_LOCAL_MEM REAL matrixProp[MATRIX_BLOCK_SIZE];\
    KW_LOCAL_MEM REAL sum[PADDED_STATE_COUNT];\
    KW_LOCAL_MEM REAL sumD1[PADDED_STATE_COUNT];\
    KW_LOCAL_MEM REAL sumD2[PADDED_STATE_COUNT];\
    stateFreq[state] = dFrequencies[state];\
    sum[state]   = 0;\
    sumD1[state] = 0;\
    sumD2[state] = 0;\
    for(int matrixEdge = 0; matrixEdge < matrixCount; matrixEdge += PADDED_STATE_COUNT) {\
        int x = matrixEdge + state;\
        if (x < matrixCount)\
            matrixProp[x] = dWeights[x];\
    }\
    KW_LOCAL_FENCE;\
    int u = state + pattern * PADDED_STATE_COUNT;\
    int delta = patternCount * PADDED_STATE_COUNT;\
    for(int r = 0; r < matrixCount; r++) {\
        FMA(dRootPartials[   u + delta * r], matrixProp[r], sum[state]  );\
        FMA(dRootFirstDeriv[ u + delta * r], matrixProp[r], sumD1[state]);\
        FMA(dRootSecondDeriv[u + delta * r], matrixProp[r], sumD2[state]);\
    }\
    sum[state]   *= stateFreq[state];\
    sumD1[state] *= stateFreq[state];\
    sumD2[state] *= stateFreq[state];\
    KW_LOCAL_FENCE;

#define SUM_STATES_POWER_OF_TWO_X_GPU()\
    /* parallelized reduction, only works for powers-of-2 */\
    for (int i = PADDED_STATE_COUNT / 2; i > 0; i >>= 1) {\
        if (state < i) {\
            sum[state] += sum[state + i];\
        }\
        KW_LOCAL_FENCE;\
    }

#define SUM_STATES_X_GPU()\
    /* not power-of-2 */\
    for (int i = SMALLEST_POWER_OF_TWO / 2; i > 0; i >>= 1) {\
        if (state < i && state + i < PADDED_STATE_COUNT ) {\
            sum[state] += sum[state + i];\
        }\
        KW_LOCAL_FENCE;\
    }

#define SUM_STATES_DERIVS_POWER_OF_TWO_X_GPU()\
    for (int i = PADDED_STATE_COUNT / 2; i > 0; i >>= 1) {\
        if (state < i) {\
            sum[state]   += sum[state + i];\
            sumD1[state] += sumD1[state + i];\
            sumD2[state] += sumD2[state + i];\
        }\
        KW_LOCAL_FENCE;\
    }

#define SUM_STATES_DERIVS_X_GPU()\
    for (int i = SMALLEST_POWER_OF_TWO / 2; i > 0; i >>= 1) {\
        if (state < i && state + i < PADDED_STATE_COUNT ) {\
            sum[state]   += sum[state + i];\
            sumD1[state] += sumD1[state + i];\
            sumD2[state] += sumD2[state + i];\
        }\
        KW_LOCAL_FENCE;\
    }

///////////////////////////////////////////////////////////////////////////////

//KW_GLOBAL_KERNEL void kernelPartialsPartialsEdgeFirstDerivatives(KW_GLOBAL_VAR REAL* KW_RESTRICT partials1,
//                                                                 KW_GLOBAL_VAR REAL* KW_RESTRICT partials2,
//                                                                 KW_GLOBAL_VAR REAL* KW_RESTRICT matrices1,
//                                                                 int totalPatterns, int categoryCount) {
//#ifdef FW_OPENCL_CPU // CPU/MIC implementation
//    // Not implemented
//#else // GPU implementation
////    DETERMINE_INDICES_X_GPU();
////
////    KW_GLOBAL_VAR REAL* KW_RESTRICT matrix1 = matrices1 + deltaMatrix; /* Points to *this* matrix */
////    KW_GLOBAL_VAR REAL* KW_RESTRICT matrix2 = matrices2 + deltaMatrix;
////
////    /* Load values into shared memory */
////    KW_LOCAL_MEM REAL sMatrix1[BLOCK_PEELING_SIZE][PADDED_STATE_COUNT];
////    KW_LOCAL_MEM REAL sMatrix2[BLOCK_PEELING_SIZE][PADDED_STATE_COUNT];
////
////    KW_LOCAL_MEM REAL sPartials1[PATTERN_BLOCK_SIZE][PADDED_STATE_COUNT];
////    KW_LOCAL_MEM REAL sPartials2[PATTERN_BLOCK_SIZE][PADDED_STATE_COUNT];
////
////    int y = deltaPartialsByState + deltaPartialsByMatrix;
////
////    /* copy PADDED_STATE_COUNT*PATTERN_BLOCK_SIZE lengthed partials */
////    /* These are all coherent global memory reads; checked in Profiler */
////    if (pattern < totalPatterns) {
////        sPartials1[patIdx][state] = partials1[y + state];
////        sPartials2[patIdx][state] = partials2[y + state];
////    } else {
////        sPartials1[patIdx][state] = 0;
////        sPartials2[patIdx][state] = 0;
////    }
////
////    REAL sum2 = 0;
////    for (int i = 0; i < PADDED_STATE_COUNT; i += BLOCK_PEELING_SIZE) {
////        /* load one row of matrices */
////        if (patIdx < BLOCK_PEELING_SIZE) {
////            /* These are all coherent global memory reads. */
////            sMatrix2[patIdx][state] = matrix2[patIdx * PADDED_STATE_COUNT + state];
////            /* sMatrix now filled with starting in state and ending in i */
////            matrix2 += BLOCK_PEELING_SIZE * PADDED_STATE_COUNT;
////        }
////
////        KW_LOCAL_FENCE;
////
////        for(int j = 0; j < BLOCK_PEELING_SIZE; j++) {
////            FMA(sMatrix2[j][state],  sPartials2[patIdx][i + j], sum2);
////        }
////
////        KW_LOCAL_FENCE;
////    }
////
////    sPartials1[patIdx][state] *= sum2;
////
////    KW_LOCAL_FENCE; // TODO Remove?
////
////    REAL sum1 = 0;
////    for (int i = 0; i < PADDED_STATE_COUNT; i += BLOCK_PEELING_SIZE) {
////        /* load one row of matrices */
////        if (patIdx < BLOCK_PEELING_SIZE) {
////            /* These are all coherent global memory reads. */
////            sMatrix1[patIdx][state] = matrix1[patIdx * PADDED_STATE_COUNT + state];
////            /* sMatrix now filled with starting in state and ending in i */
////            matrix1 += BLOCK_PEELING_SIZE * PADDED_STATE_COUNT;
////        }
////
////        KW_LOCAL_FENCE;
////
////        for(int j = 0; j < BLOCK_PEELING_SIZE; j++) {
////            FMA(sMatrix1[j][state],  sPartials1[patIdx][i + j], sum1);
////        }
////
////        KW_LOCAL_FENCE;
////    }
////
////    if (pattern < totalPatterns) {
////        partials3[u] = sum1;
////    }
//
//#endif
//}

KW_GLOBAL_KERNEL void kernelPartialsPartialsNoScale(KW_GLOBAL_VAR REAL* KW_RESTRICT partials1,
                                                    KW_GLOBAL_VAR REAL* KW_RESTRICT partials2,
                                                    KW_GLOBAL_VAR REAL* KW_RESTRICT partials3,
                                                    KW_GLOBAL_VAR REAL* KW_RESTRICT matrices1,
                                                    KW_GLOBAL_VAR REAL* KW_RESTRICT matrices2,
                                                    int totalPatterns) {
#ifdef FW_OPENCL_CPU // CPU/MIC implementation
    DETERMINE_INDICES_X_CPU();
    SUM_PARTIALS_PARTIALS_X_CPU();
    partials3[u] = sum1 * sum2;
#else // GPU implementation
    DETERMINE_INDICES_X_GPU();
    SUM_PARTIALS_PARTIALS_X_GPU();
    if (pattern < totalPatterns)
        partials3[u] = sum1 * sum2;
#endif // FW_OPENCL_CPU
}

KW_GLOBAL_KERNEL void kernelPartialsPartialsFixedScale(KW_GLOBAL_VAR REAL* KW_RESTRICT partials1,
                                                       KW_GLOBAL_VAR REAL* KW_RESTRICT partials2,
                                                       KW_GLOBAL_VAR REAL* KW_RESTRICT partials3,
                                                       KW_GLOBAL_VAR REAL* KW_RESTRICT matrices1,
                                                       KW_GLOBAL_VAR REAL* KW_RESTRICT matrices2,
                                                       KW_GLOBAL_VAR REAL* KW_RESTRICT scalingFactors,
                                                       int totalPatterns) {
#ifdef FW_OPENCL_CPU // CPU/MIC implementation
    DETERMINE_INDICES_X_CPU();
    SUM_PARTIALS_PARTIALS_X_CPU();
    partials3[u] = sum1 * sum2 / scalingFactors[pattern];
#else // GPU implementation
    DETERMINE_INDICES_X_GPU();
    LOAD_SCALING_X_GPU();
    SUM_PARTIALS_PARTIALS_X_GPU();
    if (pattern < totalPatterns)
        partials3[u] = sum1 * sum2 / fixedScalingFactors[patIdx];
#endif // FW_OPENCL_CPU
}

KW_GLOBAL_KERNEL void kernelStatesPartialsNoScale(KW_GLOBAL_VAR int* KW_RESTRICT states1,
                                                  KW_GLOBAL_VAR REAL* KW_RESTRICT partials2,
                                                  KW_GLOBAL_VAR REAL* KW_RESTRICT partials3,
                                                  KW_GLOBAL_VAR REAL* KW_RESTRICT matrices1,
                                                  KW_GLOBAL_VAR REAL* KW_RESTRICT matrices2,
                                                  int totalPatterns) {
#ifdef FW_OPENCL_CPU // CPU/MIC implementation
    DETERMINE_INDICES_X_CPU();
    SUM_STATES_PARTIALS_X_CPU();
    partials3[u] = sum1 * sum2;
#else // GPU implementation
    DETERMINE_INDICES_X_GPU();
    SUM_STATES_PARTIALS_X_GPU();
    if (pattern < totalPatterns)
        partials3[u] = sum1 * sum2;
#endif // FW_OPENCL_CPU
}

KW_GLOBAL_KERNEL void kernelStatesPartialsFixedScale(KW_GLOBAL_VAR int* KW_RESTRICT states1,
                                                     KW_GLOBAL_VAR REAL* KW_RESTRICT partials2,
                                                     KW_GLOBAL_VAR REAL* KW_RESTRICT partials3,
                                                     KW_GLOBAL_VAR REAL* KW_RESTRICT matrices1,
                                                     KW_GLOBAL_VAR REAL* KW_RESTRICT matrices2,
                                                     KW_GLOBAL_VAR REAL* KW_RESTRICT scalingFactors,
                                                     int totalPatterns) {
#ifdef FW_OPENCL_CPU // CPU/MIC implementation
    DETERMINE_INDICES_X_CPU();
    SUM_STATES_PARTIALS_X_CPU();
    partials3[u] = sum1 * sum2 / scalingFactors[pattern];
#else // GPU implementation
    DETERMINE_INDICES_X_GPU();
    LOAD_SCALING_X_GPU();
    SUM_STATES_PARTIALS_X_GPU();
    if (pattern < totalPatterns)
        partials3[u] = sum1 * sum2 / fixedScalingFactors[patIdx];
#endif // FW_OPENCL_CPU
}

KW_GLOBAL_KERNEL void kernelStatesStatesNoScale(KW_GLOBAL_VAR int* KW_RESTRICT states1,
                                                KW_GLOBAL_VAR int* KW_RESTRICT states2,
                                                KW_GLOBAL_VAR REAL* KW_RESTRICT partials3,
                                                KW_GLOBAL_VAR REAL* KW_RESTRICT matrices1,
                                                KW_GLOBAL_VAR REAL* KW_RESTRICT matrices2,
                                                int totalPatterns) {
#ifdef FW_OPENCL_CPU // CPU/MIC implementation
    DETERMINE_INDICES_X_CPU();
    int state1 = states1[pattern];
    int state2 = states2[pattern];
    KW_GLOBAL_VAR REAL* KW_RESTRICT matrix1 = matrices1 + deltaMatrix + state1 * PADDED_STATE_COUNT;
    KW_GLOBAL_VAR REAL* KW_RESTRICT matrix2 = matrices2 + deltaMatrix + state2 * PADDED_STATE_COUNT;
    if (state1 < PADDED_STATE_COUNT && state2 < PADDED_STATE_COUNT) {
        partials3[u] = matrix1[state] * matrix2[state];
    } else if (state1 < PADDED_STATE_COUNT) {
        partials3[u] = matrix1[state];
    } else if (state2 < PADDED_STATE_COUNT) {
        partials3[u] = matrix2[state];
    } else {
        partials3[u] = 1.0;
    }
#else // GPU implementation
    DETERMINE_INDICES_X_GPU();
    int state1 = states1[pattern];
    int state2 = states2[pattern];
    KW_GLOBAL_VAR REAL* KW_RESTRICT matrix1 = matrices1 + deltaMatrix + state1 * PADDED_STATE_COUNT;
    KW_GLOBAL_VAR REAL* KW_RESTRICT matrix2 = matrices2 + deltaMatrix + state2 * PADDED_STATE_COUNT;
    if (pattern < totalPatterns) {
        if (state1 < PADDED_STATE_COUNT && state2 < PADDED_STATE_COUNT) {
            partials3[u] = matrix1[state] * matrix2[state];
        } else if (state1 < PADDED_STATE_COUNT) {
            partials3[u] = matrix1[state];
        } else if (state2 < PADDED_STATE_COUNT) {
            partials3[u] = matrix2[state];
        } else {
            partials3[u] = 1.0;
        }
    }
#endif // FW_OPENCL_CPU
}

KW_GLOBAL_KERNEL void kernelStatesStatesFixedScale(KW_GLOBAL_VAR int* KW_RESTRICT states1,
                                                   KW_GLOBAL_VAR int* KW_RESTRICT states2,
                                                   KW_GLOBAL_VAR REAL* KW_RESTRICT partials3,
                                                   KW_GLOBAL_VAR REAL* KW_RESTRICT matrices1,
                                                   KW_GLOBAL_VAR REAL* KW_RESTRICT matrices2,
                                                   KW_GLOBAL_VAR REAL* KW_RESTRICT scalingFactors,
                                                   int totalPatterns) {
#ifdef FW_OPENCL_CPU // CPU/MIC implementation
    DETERMINE_INDICES_X_CPU();
    int state1 = states1[pattern];
    int state2 = states2[pattern];
    KW_GLOBAL_VAR REAL* KW_RESTRICT matrix1 = matrices1 + deltaMatrix + state1 * PADDED_STATE_COUNT;
    KW_GLOBAL_VAR REAL* KW_RESTRICT matrix2 = matrices2 + deltaMatrix + state2 * PADDED_STATE_COUNT;
    if (state1 < PADDED_STATE_COUNT && state2 < PADDED_STATE_COUNT) {
        partials3[u] = matrix1[state] * matrix2[state] / scalingFactors[pattern];
    } else if (state1 < PADDED_STATE_COUNT) {
        partials3[u] = matrix1[state] / scalingFactors[pattern];
    } else if (state2 < PADDED_STATE_COUNT) {
        partials3[u] = matrix2[state] / scalingFactors[pattern];
    } else {
        partials3[u] = 1.0 / scalingFactors[pattern];
    }
#else // GPU implementation
    DETERMINE_INDICES_X_GPU();
    int state1 = states1[pattern];
    int state2 = states2[pattern];
    KW_GLOBAL_VAR REAL* KW_RESTRICT matrix1 = matrices1 + deltaMatrix + state1 * PADDED_STATE_COUNT;
    KW_GLOBAL_VAR REAL* KW_RESTRICT matrix2 = matrices2 + deltaMatrix + state2 * PADDED_STATE_COUNT;
    LOAD_SCALING_X_GPU();
    KW_LOCAL_FENCE;
    if (pattern < totalPatterns) {
        if (state1 < PADDED_STATE_COUNT && state2 < PADDED_STATE_COUNT) {
            partials3[u] = matrix1[state] * matrix2[state] / fixedScalingFactors[patIdx];
        } else if (state1 < PADDED_STATE_COUNT) {
            partials3[u] = matrix1[state] / fixedScalingFactors[patIdx];
        } else if (state2 < PADDED_STATE_COUNT) {
            partials3[u] = matrix2[state] / fixedScalingFactors[patIdx];
        } else {
            partials3[u] = 1.0 / fixedScalingFactors[patIdx];
        }
    }
#endif // FW_OPENCL_CPU
}

// Find a scaling factor for each pattern
KW_GLOBAL_KERNEL void kernelPartialsDynamicScaling(KW_GLOBAL_VAR REAL* KW_RESTRICT allPartials,
                                                   KW_GLOBAL_VAR REAL* KW_RESTRICT scalingFactors,
                                                   int matrixCount) {
#ifdef FW_OPENCL_CPU // CPU/MIC implementation
    FIND_MAX_PARTIALS_X_CPU();
    if (max == 0)
        max = 1.0;
    scalingFactors[pattern] = max;
    SCALE_PARTIALS_X_CPU();
#else // GPU implementation
    LOAD_PARTIALS_SCALING_X_GPU();
#ifdef IS_POWER_OF_TWO
    FIND_MAX_PARTIALS_STATE_POWER_OF_TWO_X_GPU();
#else // not power-of-2
    FIND_MAX_PARTIALS_STATE_X_GPU();
#endif // IS_POWER_OF_TWO
    if (state == 0 && matrix == 0) {
        FIND_MAX_PARTIALS_MATRIX_X_GPU();
        if (max == 0)
        	max = 1.0;
        scalingFactors[pattern] = max; // TODO: These are incoherent memory writes!!!
    }
    SCALE_PARTIALS_X_GPU();
#endif // FW_OPENCL_CPU
}

KW_GLOBAL_KERNEL void kernelPartialsDynamicScalingScalersLog(KW_GLOBAL_VAR REAL* KW_RESTRICT allPartials,
                                                             KW_GLOBAL_VAR REAL* KW_RESTRICT scalingFactors,
                                                             int matrixCount) {
#ifdef FW_OPENCL_CPU // CPU/MIC implementation
    FIND_MAX_PARTIALS_X_CPU();
    if (max == 0) {
        max = 1.0;
        scalingFactors[pattern] = 0.0;
    } else {
        scalingFactors[pattern] = log(max);
    }
    SCALE_PARTIALS_X_CPU();
#else // GPU implementation
    LOAD_PARTIALS_SCALING_X_GPU();
#ifdef IS_POWER_OF_TWO
    FIND_MAX_PARTIALS_STATE_POWER_OF_TWO_X_GPU();
#else // not power-of-2
    FIND_MAX_PARTIALS_STATE_X_GPU();
#endif // IS_POWER_OF_TWO
    if (state == 0 && matrix == 0) {
        FIND_MAX_PARTIALS_MATRIX_X_GPU();
        if (max == 0) {
            max = 1.0;
            scalingFactors[pattern] = 0.0;
        } else {
            scalingFactors[pattern] = log(max);
        }
    }
    SCALE_PARTIALS_X_GPU();
#endif // FW_OPENCL_CPU
}



// Find a scaling factor for each pattern and accumulate into buffer
KW_GLOBAL_KERNEL void kernelPartialsDynamicScalingAccumulate(KW_GLOBAL_VAR REAL* KW_RESTRICT allPartials,
                                                             KW_GLOBAL_VAR REAL* KW_RESTRICT scalingFactors,
                                                             KW_GLOBAL_VAR REAL* KW_RESTRICT cumulativeScaling,
                                                             int matrixCount) {
#ifdef FW_OPENCL_CPU // CPU/MIC implementation
    FIND_MAX_PARTIALS_X_CPU();
    if (max == 0)
        max = 1.0;
    scalingFactors[pattern] = max;
    cumulativeScaling[pattern] += log(max);
    SCALE_PARTIALS_X_CPU();
#else // GPU implementation
    LOAD_PARTIALS_SCALING_X_GPU();
#ifdef IS_POWER_OF_TWO
    FIND_MAX_PARTIALS_STATE_POWER_OF_TWO_X_GPU();
#else // not power-of-2
    FIND_MAX_PARTIALS_STATE_X_GPU();
#endif // IS_POWER_OF_TWO
    if (state == 0 && matrix == 0) {
        FIND_MAX_PARTIALS_MATRIX_X_GPU();
        if (max == 0)
            max = 1.0;
        scalingFactors[pattern] = max;
        #ifdef CUDA
            atomicAdd(&cumulativeScaling[pattern], log(max));
        #else
            cumulativeScaling[pattern] += log(max);
        #endif
    }
    SCALE_PARTIALS_X_GPU();
#endif // FW_OPENCL_CPU
}

KW_GLOBAL_KERNEL void kernelPartialsDynamicScalingAccumulateScalersLog(KW_GLOBAL_VAR REAL* KW_RESTRICT allPartials,
                                                                       KW_GLOBAL_VAR REAL* KW_RESTRICT scalingFactors,
                                                                       KW_GLOBAL_VAR REAL* KW_RESTRICT cumulativeScaling,
                                                                       int matrixCount) {
#ifdef FW_OPENCL_CPU // CPU/MIC implementation
    FIND_MAX_PARTIALS_X_CPU();
    if (max == 0) {
        max = 1.0;
        scalingFactors[pattern] = 0.0;
    } else {
        REAL logMax = log(max);
        scalingFactors[pattern] = logMax;
        cumulativeScaling[pattern] += logMax;
    }
    SCALE_PARTIALS_X_CPU();
#else // GPU implementation
    LOAD_PARTIALS_SCALING_X_GPU();
#ifdef IS_POWER_OF_TWO
    FIND_MAX_PARTIALS_STATE_POWER_OF_TWO_X_GPU();
#else // not power-of-2
    FIND_MAX_PARTIALS_STATE_X_GPU();
#endif // IS_POWER_OF_TWO
    if (state == 0 && matrix == 0) {
        FIND_MAX_PARTIALS_MATRIX_X_GPU();
        if (max == 0) {
            max = 1.0;
            scalingFactors[pattern] = 0.0;
        } else {
            REAL logMax = log(max);
            scalingFactors[pattern] = logMax;
            #ifdef CUDA
                atomicAdd(&cumulativeScaling[pattern], logMax);
            #else
                cumulativeScaling[pattern] += logMax;
            #endif
        }
    }
    SCALE_PARTIALS_X_GPU();
#endif // FW_OPENCL_CPU
}

KW_GLOBAL_KERNEL void kernelIntegrateLikelihoods(KW_GLOBAL_VAR REAL* KW_RESTRICT dResult,
                                                 KW_GLOBAL_VAR REAL* KW_RESTRICT dRootPartials,
                                                 KW_GLOBAL_VAR REAL* KW_RESTRICT dWeights,
                                                 KW_GLOBAL_VAR REAL* KW_RESTRICT dFrequencies,
                                                 int matrixCount,
                                                 int patternCount) {
#ifdef FW_OPENCL_CPU // CPU/MIC implementation
    INTEGRATE_PARTIALS_X_CPU();
    dResult[pattern] = log(sumTotal);
#else // GPU implementation
    INTEGRATE_PARTIALS_X_GPU();
    #ifdef IS_POWER_OF_TWO
        SUM_STATES_POWER_OF_TWO_X_GPU();
    #else // not power-of-2
        SUM_STATES_X_GPU();
    #endif // IS_POWER_OF_TWO
    if (state == 0)
        dResult[pattern] = log(sum[state]);
#endif // FW_OPENCL_CPU
}

KW_GLOBAL_KERNEL void kernelIntegrateLikelihoodsFixedScale(KW_GLOBAL_VAR REAL* KW_RESTRICT dResult,
                                                           KW_GLOBAL_VAR REAL* KW_RESTRICT dRootPartials,
                                                           KW_GLOBAL_VAR REAL* KW_RESTRICT dWeights,
                                                           KW_GLOBAL_VAR REAL* KW_RESTRICT dFrequencies,
                                                           KW_GLOBAL_VAR REAL* KW_RESTRICT dRootScalingFactors,
                                                           int matrixCount,
                                                           int patternCount) {
#ifdef FW_OPENCL_CPU // CPU/MIC implementation
    INTEGRATE_PARTIALS_X_CPU();
    dResult[pattern] = log(sumTotal) + dRootScalingFactors[pattern];
#else // GPU implementation
    INTEGRATE_PARTIALS_X_GPU();
    #ifdef IS_POWER_OF_TWO
        SUM_STATES_POWER_OF_TWO_X_GPU();
    #else // not power-of-2
        SUM_STATES_X_GPU();
    #endif // IS_POWER_OF_TWO
    if (state == 0)
        dResult[pattern] = log(sum[state]) + dRootScalingFactors[pattern];
#endif // FW_OPENCL_CPU
}

KW_GLOBAL_KERNEL void kernelIntegrateLikelihoodsMulti(KW_GLOBAL_VAR REAL* KW_RESTRICT dResult,
                                                      KW_GLOBAL_VAR REAL* KW_RESTRICT dRootPartials,
                                                      KW_GLOBAL_VAR REAL* KW_RESTRICT dWeights,
                                                      KW_GLOBAL_VAR REAL* KW_RESTRICT dFrequencies,
                                                      int matrixCount,
                                                      int patternCount,
                                                      int takeLog) {
#ifdef FW_OPENCL_CPU // CPU/MIC implementation
    INTEGRATE_PARTIALS_X_CPU();
    if (takeLog == 0)
        dResult[pattern] = sumTotal;
    else if (takeLog == 1)
        dResult[pattern] = log(dResult[pattern] + sumTotal);
    else
        dResult[pattern] += sumTotal;
#else // GPU implementation
    INTEGRATE_PARTIALS_X_GPU();
    #ifdef IS_POWER_OF_TWO
        SUM_STATES_POWER_OF_TWO_X_GPU();
    #else // not power-of-2
        SUM_STATES_X_GPU();
    #endif // IS_POWER_OF_TWO
    if (state == 0) {
        if (takeLog == 0)
            dResult[pattern] = sum[state];
        else if (takeLog == 1)
            dResult[pattern] = log(dResult[pattern] + sum[state]);
        else
            dResult[pattern] += sum[state];
    }
#endif // FW_OPENCL_CPU
}

KW_GLOBAL_KERNEL void kernelIntegrateLikelihoodsFixedScaleMulti(KW_GLOBAL_VAR REAL* KW_RESTRICT dResult,
											                    KW_GLOBAL_VAR REAL* KW_RESTRICT dRootPartials,
                                                                KW_GLOBAL_VAR REAL* KW_RESTRICT dWeights,
                                                                KW_GLOBAL_VAR REAL* KW_RESTRICT dFrequencies,
                                                                KW_GLOBAL_VAR REAL* KW_RESTRICT dScalingFactors,
											                    KW_GLOBAL_VAR unsigned int* KW_RESTRICT dPtrQueue,
											                    KW_GLOBAL_VAR REAL* KW_RESTRICT dMaxScalingFactors,
											                    KW_GLOBAL_VAR unsigned int* KW_RESTRICT dIndexMaxScalingFactors,
                                                                int matrixCount,
                                                                int patternCount,
											                    int subsetCount,
											                    int subsetIndex) {
#ifdef FW_OPENCL_CPU // CPU/MIC implementation
    INTEGRATE_PARTIALS_X_CPU();
    REAL cumulativeScalingFactor = (dScalingFactors + dPtrQueue[subsetIndex])[pattern];
    if (subsetIndex == 0) {
        int indexMaxScalingFactor = 0;
        REAL maxScalingFactor = cumulativeScalingFactor;
        for (int j = 1; j < subsetCount; j++) {
            REAL tmpScalingFactor = (dScalingFactors + dPtrQueue[j])[pattern];
            if (tmpScalingFactor > maxScalingFactor) {
                indexMaxScalingFactor = j;
                maxScalingFactor = tmpScalingFactor;
            }
        }
        dIndexMaxScalingFactors[pattern] = indexMaxScalingFactor;
        dMaxScalingFactors[pattern] = maxScalingFactor;
        if (indexMaxScalingFactor != 0)
            sumTotal *= exp((REAL)(cumulativeScalingFactor - maxScalingFactor));
        dResult[pattern] = sumTotal;
    } else {
        if (subsetIndex != dIndexMaxScalingFactors[pattern])
            sumTotal *= exp((REAL)(cumulativeScalingFactor - dMaxScalingFactors[pattern]));
        if (subsetIndex == subsetCount - 1)
            dResult[pattern] = (log(dResult[pattern] + sumTotal) + dMaxScalingFactors[pattern]);
        else
            dResult[pattern] += sumTotal;
    }
#else // GPU implementation
    INTEGRATE_PARTIALS_X_GPU();
    #ifdef IS_POWER_OF_TWO
        SUM_STATES_POWER_OF_TWO_X_GPU();
    #else // not power-of-2
        SUM_STATES_X_GPU();
    #endif // IS_POWER_OF_TWO
    REAL cumulativeScalingFactor = (dScalingFactors + dPtrQueue[subsetIndex])[pattern];
    if (subsetIndex == 0) {
        int indexMaxScalingFactor = 0;
        REAL maxScalingFactor = cumulativeScalingFactor;
        for (int j = 1; j < subsetCount; j++) {
            REAL tmpScalingFactor = (dScalingFactors + dPtrQueue[j])[pattern];
            if (tmpScalingFactor > maxScalingFactor) {
                indexMaxScalingFactor = j;
                maxScalingFactor = tmpScalingFactor;
            }
        }
        dIndexMaxScalingFactors[pattern] = indexMaxScalingFactor;
        dMaxScalingFactors[pattern] = maxScalingFactor;
        if (indexMaxScalingFactor != 0)
            sum[state] *= exp((REAL)(cumulativeScalingFactor - maxScalingFactor));
        if (state == 0)
            dResult[pattern] = sum[state];
        KW_LOCAL_FENCE;
    } else {
        if (subsetIndex != dIndexMaxScalingFactors[pattern])
            sum[state] *= exp((REAL)(cumulativeScalingFactor - dMaxScalingFactors[pattern]));
        if (state == 0) {
            if (subsetIndex == subsetCount - 1)
                dResult[pattern] = (log(dResult[pattern] + sum[state]) + dMaxScalingFactors[pattern]);
            else
                dResult[pattern] += sum[state];
        }
    }
#endif // FW_OPENCL_CPU
}

////////////////////////////////////////////////////////////////////////////////////////////////
// edge and deriv kernels

KW_GLOBAL_KERNEL void kernelPartialsPartialsEdgeLikelihoods(KW_GLOBAL_VAR REAL* KW_RESTRICT dPartialsTmp,
                                                            KW_GLOBAL_VAR REAL* KW_RESTRICT dParentPartials,
                                                            KW_GLOBAL_VAR REAL* KW_RESTRICT dChildParials,
                                                            KW_GLOBAL_VAR REAL* KW_RESTRICT dTransMatrix,
                                                            int totalPatterns) {

#ifdef FW_OPENCL_CPU // CPU/MIC implementation
    DETERMINE_INDICES_X_CPU();
    int deltaPartials = deltaPartialsByMatrix + deltaPartialsByState;
    KW_GLOBAL_VAR REAL* KW_RESTRICT sMatrix1 = dTransMatrix + deltaMatrix;
    KW_GLOBAL_VAR REAL* KW_RESTRICT sPartials1 = dParentPartials + deltaPartials;
    KW_GLOBAL_VAR REAL* KW_RESTRICT sPartials2 = dChildParials + deltaPartials;
    REAL sum1 = 0;
    for(int i = 0; i < PADDED_STATE_COUNT; i++) {
        FMA(sMatrix1[i * PADDED_STATE_COUNT + state],  sPartials1[i], sum1);
    }
    dPartialsTmp[u] = sum1 * sPartials2[state];
#else // GPU implementation
    DETERMINE_INDICES_X_GPU();
    KW_GLOBAL_VAR REAL* KW_RESTRICT matrix1 = dTransMatrix + deltaMatrix;
    int y = deltaPartialsByState + deltaPartialsByMatrix;
    KW_LOCAL_MEM REAL sMatrix1[BLOCK_PEELING_SIZE][PADDED_STATE_COUNT];
    KW_LOCAL_MEM REAL sPartials1[PATTERN_BLOCK_SIZE][PADDED_STATE_COUNT];
    KW_LOCAL_MEM REAL sPartials2[PATTERN_BLOCK_SIZE][PADDED_STATE_COUNT];
    if (pattern < totalPatterns) {
        sPartials1[patIdx][state] = dParentPartials[y + state];
        sPartials2[patIdx][state] = dChildParials[y + state];
    } else {
        sPartials1[patIdx][state] = 0;
        sPartials2[patIdx][state] = 0;
    }
    REAL sum1 = 0;
    int i;
    for (i = 0; i < PADDED_STATE_COUNT; i += BLOCK_PEELING_SIZE) {
        if (patIdx < BLOCK_PEELING_SIZE) {
            sMatrix1[patIdx][state] = matrix1[patIdx * PADDED_STATE_COUNT + state];
            matrix1 += BLOCK_PEELING_SIZE * PADDED_STATE_COUNT;
        }
        KW_LOCAL_FENCE;
        int j;
        for(j = 0; j < BLOCK_PEELING_SIZE; j++) {
            FMA(sMatrix1[j][state], sPartials1[patIdx][i + j], sum1);
        }
        KW_LOCAL_FENCE;
    }
    if (pattern < totalPatterns)
        dPartialsTmp[u] = sum1 * sPartials2[patIdx][state];
#endif // FW_OPENCL_CPU
}


KW_GLOBAL_KERNEL void
#ifdef CUDA
__launch_bounds__(PATTERN_BLOCK_SIZE * PADDED_STATE_COUNT)
#endif
kernelPartialsPartialsEdgeLikelihoodsSecondDeriv(KW_GLOBAL_VAR REAL* KW_RESTRICT dPartialsTmp,
                                                 KW_GLOBAL_VAR REAL* KW_RESTRICT dFirstDerivTmp,
                                                 KW_GLOBAL_VAR REAL* KW_RESTRICT dSecondDerivTmp,
                                                 KW_GLOBAL_VAR REAL* KW_RESTRICT dParentPartials,
                                                 KW_GLOBAL_VAR REAL* KW_RESTRICT dChildParials,
                                                 KW_GLOBAL_VAR REAL* KW_RESTRICT dTransMatrix,
                                                 KW_GLOBAL_VAR REAL* KW_RESTRICT dFirstDerivMatrix,
                                                 KW_GLOBAL_VAR REAL* KW_RESTRICT dSecondDerivMatrix,
                                                 int totalPatterns) {
#ifdef FW_OPENCL_CPU // CPU/MIC implementation
    DETERMINE_INDICES_X_CPU();
    int deltaPartials = deltaPartialsByMatrix + deltaPartialsByState;
    KW_GLOBAL_VAR REAL* KW_RESTRICT sMatrix1 = dTransMatrix + deltaMatrix;
    KW_GLOBAL_VAR REAL* KW_RESTRICT sMatrixFirstDeriv = dFirstDerivMatrix + deltaMatrix;
    KW_GLOBAL_VAR REAL* KW_RESTRICT sMatrixSecondDeriv = dSecondDerivMatrix + deltaMatrix;
    KW_GLOBAL_VAR REAL* KW_RESTRICT sPartials1 = dParentPartials + deltaPartials;
    KW_GLOBAL_VAR REAL* KW_RESTRICT sPartials2 = dChildParials + deltaPartials;
    REAL sum1 = 0;
    REAL sumFirstDeriv = 0;
    REAL sumSecondDeriv = 0;
    for(int i = 0; i < PADDED_STATE_COUNT; i++) {
        FMA(sMatrix1[          i * PADDED_STATE_COUNT + state], sPartials1[i], sum1);
        FMA(sMatrixFirstDeriv[ i * PADDED_STATE_COUNT + state], sPartials1[i], sumFirstDeriv);
        FMA(sMatrixSecondDeriv[i * PADDED_STATE_COUNT + state], sPartials1[i], sumSecondDeriv);
    }
    dPartialsTmp[u]    = sum1           * sPartials2[state];
    dFirstDerivTmp[u]  = sumFirstDeriv  * sPartials2[state];
    dSecondDerivTmp[u] = sumSecondDeriv * sPartials2[state];
#else // GPU implementation
    DETERMINE_INDICES_X_GPU();
    KW_GLOBAL_VAR REAL* KW_RESTRICT matrix1 = dTransMatrix + deltaMatrix; // Points to *this* matrix
    KW_GLOBAL_VAR REAL* KW_RESTRICT matrixFirstDeriv = dFirstDerivMatrix + deltaMatrix;
    KW_GLOBAL_VAR REAL* KW_RESTRICT matrixSecondDeriv = dSecondDerivMatrix + deltaMatrix;
    int y = deltaPartialsByState + deltaPartialsByMatrix;
    KW_LOCAL_MEM REAL sMatrix1[BLOCK_PEELING_SIZE/2][PADDED_STATE_COUNT];
    KW_LOCAL_MEM REAL sMatrixFirstDeriv[BLOCK_PEELING_SIZE/2][PADDED_STATE_COUNT];
    KW_LOCAL_MEM REAL sMatrixSecondDeriv[BLOCK_PEELING_SIZE/2][PADDED_STATE_COUNT];
    KW_LOCAL_MEM REAL sPartials1[PATTERN_BLOCK_SIZE][PADDED_STATE_COUNT];
    KW_LOCAL_MEM REAL sPartials2[PATTERN_BLOCK_SIZE][PADDED_STATE_COUNT];
    if (pattern < totalPatterns) {
        sPartials1[patIdx][state] = dParentPartials[y + state];
        sPartials2[patIdx][state] = dChildParials[y + state];
    } else {
        sPartials1[patIdx][state] = 0;
        sPartials2[patIdx][state] = 0;
    }
    REAL sum1 = 0;
    REAL sumFirstDeriv = 0;
    REAL sumSecondDeriv = 0;
    int i;
    for (i = 0; i < PADDED_STATE_COUNT; i += BLOCK_PEELING_SIZE/2) {
        if (patIdx < BLOCK_PEELING_SIZE/2) {
            sMatrix1[patIdx][state] = matrix1[patIdx * PADDED_STATE_COUNT + state];
            sMatrixFirstDeriv[patIdx][state] = matrixFirstDeriv[patIdx * PADDED_STATE_COUNT + state];
            sMatrixSecondDeriv[patIdx][state] = matrixSecondDeriv[patIdx * PADDED_STATE_COUNT + state];
            matrix1 += BLOCK_PEELING_SIZE/2 * PADDED_STATE_COUNT;
            matrixFirstDeriv += BLOCK_PEELING_SIZE/2 * PADDED_STATE_COUNT;
            matrixSecondDeriv += BLOCK_PEELING_SIZE/2 * PADDED_STATE_COUNT;
        }
        KW_LOCAL_FENCE;
        int j;
        for(j = 0; j < BLOCK_PEELING_SIZE/2; j++) {
            FMA(sMatrix1[j][state]          , sPartials1[patIdx][i + j], sum1          );
            FMA(sMatrixFirstDeriv[j][state] , sPartials1[patIdx][i + j], sumFirstDeriv );
            FMA(sMatrixSecondDeriv[j][state], sPartials1[patIdx][i + j], sumSecondDeriv);
        }
        KW_LOCAL_FENCE;
    }
    if (pattern < totalPatterns) {
        dPartialsTmp[u] = sum1 * sPartials2[patIdx][state];
        dFirstDerivTmp[u] = sumFirstDeriv * sPartials2[patIdx][state];
        dSecondDerivTmp[u] = sumSecondDeriv * sPartials2[patIdx][state];
    }
#endif // FW_OPENCL_CPU
}

KW_GLOBAL_KERNEL void kernelStatesPartialsEdgeLikelihoods(KW_GLOBAL_VAR REAL* KW_RESTRICT dPartialsTmp,
                                                          KW_GLOBAL_VAR REAL* KW_RESTRICT dParentPartials,
                                                          KW_GLOBAL_VAR int* KW_RESTRICT dChildStates,
                                                          KW_GLOBAL_VAR REAL* KW_RESTRICT dTransMatrix,
                                                          int totalPatterns) {
#ifdef FW_OPENCL_CPU // CPU/MIC implementation
    DETERMINE_INDICES_X_CPU();
    int deltaPartials = deltaPartialsByMatrix + deltaPartialsByState;
    KW_GLOBAL_VAR REAL* KW_RESTRICT sMatrix1 = dTransMatrix + deltaMatrix;
    KW_GLOBAL_VAR REAL* KW_RESTRICT sPartials2 = dParentPartials + deltaPartials;
    REAL sum1 = 0;
    int state1 = dChildStates[pattern];
    if (state1 < PADDED_STATE_COUNT)
        sum1 = sMatrix1[state1 * PADDED_STATE_COUNT + state];
    else
        sum1 = 1.0;
    dPartialsTmp[u] = sum1 * sPartials2[state];
#else // GPU implementation
    DETERMINE_INDICES_X_GPU();
    int y = deltaPartialsByState + deltaPartialsByMatrix;
    KW_LOCAL_MEM REAL sPartials2[PATTERN_BLOCK_SIZE][PADDED_STATE_COUNT];
    if (pattern < totalPatterns) {
        sPartials2[patIdx][state] = dParentPartials[y + state];
    } else {
        sPartials2[patIdx][state] = 0;
    }
    REAL sum1 = 0;
    if (pattern < totalPatterns) {
        int state1 = dChildStates[pattern];
        KW_GLOBAL_VAR REAL* KW_RESTRICT matrix1 = dTransMatrix + deltaMatrix + state1 * PADDED_STATE_COUNT;
        if (state1 < PADDED_STATE_COUNT)
            sum1 = matrix1[state];
        else
            sum1 = 1.0;
    }
    if (pattern < totalPatterns)
        dPartialsTmp[u] = sum1 * sPartials2[patIdx][state];
#endif // FW_OPENCL_CPU
}

KW_GLOBAL_KERNEL void kernelStatesPartialsEdgeLikelihoodsSecondDeriv(KW_GLOBAL_VAR REAL* KW_RESTRICT dPartialsTmp,
                                                                     KW_GLOBAL_VAR REAL* KW_RESTRICT dFirstDerivTmp,
                                                                     KW_GLOBAL_VAR REAL* KW_RESTRICT dSecondDerivTmp,
                                                                     KW_GLOBAL_VAR REAL* KW_RESTRICT dParentPartials,
                                                                     KW_GLOBAL_VAR int* KW_RESTRICT dChildStates,
                                                                     KW_GLOBAL_VAR REAL* KW_RESTRICT dTransMatrix,
                                                                     KW_GLOBAL_VAR REAL* KW_RESTRICT dFirstDerivMatrix,
                                                                     KW_GLOBAL_VAR REAL* KW_RESTRICT dSecondDerivMatrix,
                                                                     int totalPatterns) {
#ifdef FW_OPENCL_CPU // CPU/MIC implementation
    DETERMINE_INDICES_X_CPU();
    int deltaPartials = deltaPartialsByMatrix + deltaPartialsByState;
    KW_GLOBAL_VAR REAL* KW_RESTRICT sMatrix1 = dTransMatrix + deltaMatrix;
    KW_GLOBAL_VAR REAL* KW_RESTRICT sMatrixFirstDeriv = dFirstDerivMatrix + deltaMatrix;
    KW_GLOBAL_VAR REAL* KW_RESTRICT sMatrixSecondDeriv = dSecondDerivMatrix + deltaMatrix;
    KW_GLOBAL_VAR REAL* KW_RESTRICT sPartials2 = dParentPartials + deltaPartials;
    REAL sum1 = 0;
    REAL sumFirstDeriv = 0;
    REAL sumSecondDeriv = 0;
    int state1 = dChildStates[pattern];
    if (state1 < PADDED_STATE_COUNT) {
        sum1           = sMatrix1[          state1 * PADDED_STATE_COUNT + state];
        sumFirstDeriv  = sMatrixFirstDeriv[ state1 * PADDED_STATE_COUNT + state];
        sumSecondDeriv = sMatrixSecondDeriv[state1 * PADDED_STATE_COUNT + state];
    } else {
        sum1 = 1.0;
    }
    dPartialsTmp[u]    = sum1           * sPartials2[state];
    dFirstDerivTmp[u]  = sumFirstDeriv  * sPartials2[state];
    dSecondDerivTmp[u] = sumSecondDeriv * sPartials2[state];
#else // GPU implementation
    DETERMINE_INDICES_X_GPU();
    int y = deltaPartialsByState + deltaPartialsByMatrix;
    KW_LOCAL_MEM REAL sPartials2[PATTERN_BLOCK_SIZE][PADDED_STATE_COUNT];
    if (pattern < totalPatterns) {
        sPartials2[patIdx][state] = dParentPartials[y + state];
    } else {
        sPartials2[patIdx][state] = 0;
    }
    REAL sum1 = 0;
    REAL sumFirstDeriv = 0;
    REAL sumSecondDeriv = 0;
    if (pattern < totalPatterns) {
        int state1 = dChildStates[pattern]; // Coalesced; no need to share
        KW_GLOBAL_VAR REAL* KW_RESTRICT matrix1 = dTransMatrix + deltaMatrix + state1 * PADDED_STATE_COUNT;
        KW_GLOBAL_VAR REAL* KW_RESTRICT matrixFirstDeriv = dFirstDerivMatrix + deltaMatrix + state1 * PADDED_STATE_COUNT;
        KW_GLOBAL_VAR REAL* KW_RESTRICT matrixSecondDeriv = dSecondDerivMatrix + deltaMatrix + state1 * PADDED_STATE_COUNT;
        if (state1 < PADDED_STATE_COUNT) {
            sum1 = matrix1[state];
            sumFirstDeriv = matrixFirstDeriv[state];
            sumSecondDeriv = matrixSecondDeriv[state];
        } else {
            sum1 = 1.0;
            sumFirstDeriv = 0.0;
            sumSecondDeriv = 0.0;
        }
    }
    if (pattern < totalPatterns) {
        dPartialsTmp[u] = sum1 * sPartials2[patIdx][state];
        dFirstDerivTmp[u] = sumFirstDeriv * sPartials2[patIdx][state];
        dSecondDerivTmp[u] = sumSecondDeriv * sPartials2[patIdx][state];
    }
#endif // FW_OPENCL_CPU
}

KW_GLOBAL_KERNEL void kernelIntegrateLikelihoodsSecondDeriv(KW_GLOBAL_VAR REAL* KW_RESTRICT dResult,
                                                            KW_GLOBAL_VAR REAL* KW_RESTRICT dFirstDerivResult,
                                                            KW_GLOBAL_VAR REAL* KW_RESTRICT dSecondDerivResult,
                                                            KW_GLOBAL_VAR REAL* KW_RESTRICT dRootPartials,
                                                            KW_GLOBAL_VAR REAL* KW_RESTRICT dRootFirstDeriv,
                                                            KW_GLOBAL_VAR REAL* KW_RESTRICT dRootSecondDeriv,
                                                            KW_GLOBAL_VAR REAL* KW_RESTRICT dWeights,
                                                            KW_GLOBAL_VAR REAL* KW_RESTRICT dFrequencies,
                                                            int matrixCount,
                                                            int patternCount) {
#ifdef FW_OPENCL_CPU // CPU/MIC implementation
    INTEGRATE_PARTIALS_DERIV_X_CPU();
    tmpLogLike = sumTotal;
    dResult[pattern] = log(tmpLogLike);
    tmpFirstDeriv = sumTotalD1 / tmpLogLike;
    dFirstDerivResult[pattern] = tmpFirstDeriv;
    dSecondDerivResult[pattern] = (sumTotalD2 / tmpLogLike - tmpFirstDeriv * tmpFirstDeriv);
#else // GPU implementation
    INTEGRATE_PARTIALS_DERIV_X_GPU();
#ifdef IS_POWER_OF_TWO
    SUM_STATES_DERIVS_POWER_OF_TWO_X_GPU();
#else // not power-of-2
    SUM_STATES_DERIVS_X_GPU();
#endif // IS_POWER_OF_TWO
    if (state == 0) {
        tmpLogLike = sum[state];
        dResult[pattern] = log(tmpLogLike);
        tmpFirstDeriv = sumD1[state] / tmpLogLike;
        dFirstDerivResult[pattern] = tmpFirstDeriv;
        dSecondDerivResult[pattern] = (sumD2[state] / tmpLogLike - tmpFirstDeriv * tmpFirstDeriv);
    }
#endif // FW_OPENCL_CPU
}

KW_GLOBAL_KERNEL void kernelIntegrateLikelihoodsFixedScaleSecondDeriv(KW_GLOBAL_VAR REAL* KW_RESTRICT dResult,
                                                                      KW_GLOBAL_VAR REAL* KW_RESTRICT dFirstDerivResult,
                                                                      KW_GLOBAL_VAR REAL* KW_RESTRICT dSecondDerivResult,
                                                                      KW_GLOBAL_VAR REAL* KW_RESTRICT dRootPartials,
                                                                      KW_GLOBAL_VAR REAL* KW_RESTRICT dRootFirstDeriv,
                                                                      KW_GLOBAL_VAR REAL* KW_RESTRICT dRootSecondDeriv,
                                                                      KW_GLOBAL_VAR REAL* KW_RESTRICT dWeights,
                                                                      KW_GLOBAL_VAR REAL* KW_RESTRICT dFrequencies,
                                                                      KW_GLOBAL_VAR REAL* KW_RESTRICT dRootScalingFactors,
                                                                      int matrixCount,
                                                                      int patternCount) {
#ifdef FW_OPENCL_CPU // CPU/MIC implementation
    INTEGRATE_PARTIALS_DERIV_X_CPU();
    tmpLogLike = sumTotal;
    dResult[pattern] = log(tmpLogLike) + dRootScalingFactors[pattern];
    tmpFirstDeriv = sumTotalD1 / tmpLogLike;
    dFirstDerivResult[pattern] = tmpFirstDeriv;
    dSecondDerivResult[pattern] = (sumTotalD2 / tmpLogLike - tmpFirstDeriv * tmpFirstDeriv);
#else // GPU implementation
    INTEGRATE_PARTIALS_DERIV_X_GPU();
#ifdef IS_POWER_OF_TWO
    SUM_STATES_DERIVS_POWER_OF_TWO_X_GPU();
#else // not power-of-2
    SUM_STATES_DERIVS_X_GPU();
#endif // IS_POWER_OF_TWO
    if (state == 0) {
        tmpLogLike = sum[state];
        dResult[pattern] = log(tmpLogLike) + dRootScalingFactors[pattern];
        tmpFirstDeriv = sumD1[state] / tmpLogLike;
        dFirstDerivResult[pattern] = tmpFirstDeriv;
        dSecondDerivResult[pattern] = (sumD2[state] / tmpLogLike - tmpFirstDeriv * tmpFirstDeriv);
    }
#endif // FW_OPENCL_CPU
}


////////////////////////////////////////////////////////////////////////////////////////////////
// scaling experiments kernels

KW_GLOBAL_KERNEL void kernelPartialsPartialsAutoScale(KW_GLOBAL_VAR REAL* partials1,
                                                             KW_GLOBAL_VAR REAL* partials2,
                                                             KW_GLOBAL_VAR REAL* partials3,
                                                             KW_GLOBAL_VAR REAL* matrices1,
                                                             KW_GLOBAL_VAR REAL* matrices2,
                                                             KW_GLOBAL_VAR signed char* scalingFactors,
                                                             int totalPatterns) {
    REAL sum1 = 0;
    REAL sum2 = 0;
    int i;

    DETERMINE_INDICES_X_GPU();

    KW_GLOBAL_VAR REAL* matrix1 = matrices1 + deltaMatrix; // Points to *this* matrix
    KW_GLOBAL_VAR REAL* matrix2 = matrices2 + deltaMatrix;

    int y = deltaPartialsByState + deltaPartialsByMatrix;

    // Load values into shared memory
    KW_LOCAL_MEM REAL sMatrix1[BLOCK_PEELING_SIZE][PADDED_STATE_COUNT];
    KW_LOCAL_MEM REAL sMatrix2[BLOCK_PEELING_SIZE][PADDED_STATE_COUNT];

    KW_LOCAL_MEM REAL sPartials1[PATTERN_BLOCK_SIZE][PADDED_STATE_COUNT];
    KW_LOCAL_MEM REAL sPartials2[PATTERN_BLOCK_SIZE][PADDED_STATE_COUNT];

    // copy PADDED_STATE_COUNT*PATTERN_BLOCK_SIZE lengthed partials
    if (pattern < totalPatterns) {
        // These are all coherent global memory reads; checked in Profiler
        sPartials1[patIdx][state] = partials1[y + state];
        sPartials2[patIdx][state] = partials2[y + state];
    } else {
        sPartials1[patIdx][state] = 0;
        sPartials2[patIdx][state] = 0;
    }

    for (i = 0; i < PADDED_STATE_COUNT; i += BLOCK_PEELING_SIZE) {
        // load one row of matrices
        if (patIdx < BLOCK_PEELING_SIZE) {
            // These are all coherent global memory reads.
            sMatrix1[patIdx][state] = matrix1[patIdx * PADDED_STATE_COUNT + state];
            sMatrix2[patIdx][state] = matrix2[patIdx * PADDED_STATE_COUNT + state];

            // sMatrix now filled with starting in state and ending in i
            matrix1 += BLOCK_PEELING_SIZE * PADDED_STATE_COUNT;
            matrix2 += BLOCK_PEELING_SIZE * PADDED_STATE_COUNT;
        }
        KW_LOCAL_FENCE;

        int j;
        for(j = 0; j < BLOCK_PEELING_SIZE; j++) {
            sum1 += sMatrix1[j][state] * sPartials1[patIdx][i + j];
            sum2 += sMatrix2[j][state] * sPartials2[patIdx][i + j];
        }

        KW_LOCAL_FENCE; // GTX280 FIX HERE

    }

    REAL tmpPartial = sum1 * sum2;
    int expTmp;
    REAL sigTmp = frexp(tmpPartial, &expTmp);

    if (pattern < totalPatterns) {
        if (abs(expTmp) > SCALING_EXPONENT_THRESHOLD) {
            // now using sPartials2 to hold scaling trigger boolean
            sPartials2[patIdx][0] = 1;
        } else {
            partials3[u] = tmpPartial;
            sPartials2[patIdx][0] = 0;
            sPartials1[patIdx][0] = 0;
        }
    }

    KW_LOCAL_FENCE;

    int scalingActive = sPartials2[patIdx][0];

    if (scalingActive) {
        // now using sPartials1 to store max unscaled partials3
        sPartials1[patIdx][state] = tmpPartial;
    }

    KW_LOCAL_FENCE;

    // Unrolled parallel max-reduction
    if (scalingActive && state < 2) {
        REAL compare = sPartials1[patIdx][state + 2];
        if (compare >  sPartials1[patIdx][state])
            sPartials1[patIdx][state] = compare;
    }

    KW_LOCAL_FENCE;

    if (scalingActive && state < 1) {
        REAL maxPartial = sPartials1[patIdx][1];
        if (maxPartial < sPartials1[patIdx][0])
            maxPartial = sPartials1[patIdx][0];
        int expMax;
        frexp(maxPartial, &expMax);
        sPartials1[patIdx][0] = expMax;
    }

    KW_LOCAL_FENCE;

    if (scalingActive)
        partials3[u] = ldexp(sigTmp, expTmp - sPartials1[patIdx][0]);

    int myIdx = (patIdx * PADDED_STATE_COUNT) + state; // threadId in block
    if ((myIdx < PATTERN_BLOCK_SIZE) && (myIdx + (KW_GROUP_ID_0 * PATTERN_BLOCK_SIZE) < totalPatterns))
        scalingFactors[(KW_GROUP_ID_0 * PATTERN_BLOCK_SIZE) + (matrix * totalPatterns) + myIdx] = sPartials1[myIdx][0];

}

KW_GLOBAL_KERNEL void kernelIntegrateLikelihoodsAutoScaling(KW_GLOBAL_VAR REAL* dResult,
                                                     KW_GLOBAL_VAR REAL* dRootPartials,
                                                     KW_GLOBAL_VAR REAL* dWeights,
                                                     KW_GLOBAL_VAR REAL* dFrequencies,
                                                     KW_GLOBAL_VAR int* dRootScalingFactors,
                                                     int matrixCount,
                                                     int patternCount) {
    int state   = KW_LOCAL_ID_0;
    int pattern = KW_GROUP_ID_0;
//    int patternCount = KW_NUM_GROUPS_0;

    KW_LOCAL_MEM REAL stateFreq[PADDED_STATE_COUNT];
    // TODO: Currently assumes MATRIX_BLOCK_SIZE >> matrixCount
    KW_LOCAL_MEM REAL matrixProp[MATRIX_BLOCK_SIZE];
    KW_LOCAL_MEM REAL matrixScalers[MATRIX_BLOCK_SIZE];
    KW_LOCAL_MEM REAL sum[PADDED_STATE_COUNT];

    // Load shared memory

    stateFreq[state] = dFrequencies[state];
    sum[state] = 0;

    for(int matrixEdge = 0; matrixEdge < matrixCount; matrixEdge += PADDED_STATE_COUNT) {
        int x = matrixEdge + state;
        if (x < matrixCount) {
            matrixProp[x] = dWeights[x];
            matrixScalers[x] = dRootScalingFactors[pattern + (x * patternCount)];
        }
    }

    KW_LOCAL_FENCE;

    int u = state + pattern * PADDED_STATE_COUNT;
    int delta = patternCount * PADDED_STATE_COUNT;

    short maxScaleFactor = matrixScalers[0];
    for(int r = 1; r < matrixCount; r++) {
        int tmpFactor = matrixScalers[r];
        if (tmpFactor > maxScaleFactor)
            maxScaleFactor = tmpFactor;
    }

    for(int r = 0; r < matrixCount; r++) {
        int tmpFactor = matrixScalers[r];
        if (tmpFactor != maxScaleFactor) {
            int expTmp;
            sum[state] += ldexp(frexp(dRootPartials[u + delta * r], &expTmp), expTmp + (tmpFactor - maxScaleFactor)) * matrixProp[r];
        } else {
            sum[state] += dRootPartials[u + delta * r] * matrixProp[r];
        }
    }

    sum[state] *= stateFreq[state];
    KW_LOCAL_FENCE;

#ifdef IS_POWER_OF_TWO
    // parallelized reduction *** only works for powers-of-2 ****
    for (int i = PADDED_STATE_COUNT / 2; i > 0; i >>= 1) {
        if (state < i) {
#else
    for (int i = SMALLEST_POWER_OF_TWO / 2; i > 0; i >>= 1) {
        if (state < i && state + i < PADDED_STATE_COUNT ) {
#endif // IS_POWER_OF_TWO
            sum[state] += sum[state + i];
        }
        KW_LOCAL_FENCE;
    }

    if (state == 0)
        dResult[pattern] = (log(sum[state]) + (M_LN2 * maxScaleFactor));
}

#ifdef CUDA
    #include "kernelsXDerivatives.cu"
#endif // CUDA

#ifdef CUDA
} // extern "C"
#endif //CUDA
