#include "hip/hip_runtime.h"
/*
 *
 * Copyright 2009 Phylogenetic Likelihood Working Group
 *
 * This file is part of BEAGLE.
 *
 * Use of this source code is governed by an MIT-style
 * license that can be found in the LICENSE file or at
 * https://opensource.org/licenses/MIT.
 *
 * @author Marc Suchard
 * @author Daniel Ayres
 */

#ifdef CUDA
    #include "libhmsbeagle/GPU/GPUImplDefs.h"
    #include <stdlib.h>
    #include <string.h>
    #include <stdio.h>
    extern "C" {
#elif defined(FW_OPENCL)
    #ifdef DOUBLE_PRECISION
        #pragma OPENCL EXTENSION cl_khr_fp64: enable
    #endif
    #define __umul24(x, y) (x * y)
#endif //FW_OPENCL

#if (!defined DOUBLE_PRECISION && defined FP_FAST_FMAF) || (defined DOUBLE_PRECISION && defined FP_FAST_FMA)
    #define FMA(x, y, z) (z = fma(x, y, z))
#else //FP_FAST_FMA
    #define FMA(x, y, z) (z += x * y)
#endif //FP_FAST_FMA

#if (defined CUDA) && (defined DOUBLE_PRECISION) &&  (__CUDA_ARCH__ < 600)
    __device__ double atomicAdd(double* address, double val)
    {
        unsigned long long int* address_as_ull =
                                  (unsigned long long int*)address;
        unsigned long long int old = *address_as_ull, assumed;

        do {
            assumed = old;
            old = atomicCAS(address_as_ull, assumed,
                            __double_as_longlong(val +
                                   __longlong_as_double(assumed)));

        // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
        } while (assumed != old);

        return __longlong_as_double(old);
    }
#endif

///////////////////////////////////////////////////////////////////////////////

KW_GLOBAL_KERNEL void kernelReorderPatterns(      KW_GLOBAL_VAR REAL*             dPartials,
                                                  KW_GLOBAL_VAR int*              dStates,
                                                  KW_GLOBAL_VAR int*              dStatesSort,
                                            const KW_GLOBAL_VAR int*  KW_RESTRICT dTipOffsets,
                                            const KW_GLOBAL_VAR int*  KW_RESTRICT dTipTypes,
                                            const KW_GLOBAL_VAR int*  KW_RESTRICT dPatternsNewOrder,
                                            const KW_GLOBAL_VAR REAL* KW_RESTRICT dPatternWeights,
                                                  KW_GLOBAL_VAR REAL* KW_RESTRICT dPatternWeightsSort,
                                                                int               patternCount,
                                                                int               paddedPatternCount) {
#ifdef FW_OPENCL_CPU
    int state      = 0;
    int pattern    = KW_LOCAL_ID_0 + KW_GROUP_ID_0 * KW_LOCAL_SIZE_0;
#else
    int state      = KW_LOCAL_ID_0;
    int pattern    = KW_LOCAL_ID_1 + KW_GROUP_ID_0 * KW_LOCAL_SIZE_1;
#endif
    int stateCount = PADDED_STATE_COUNT;
    int category   = KW_GROUP_ID_1;
    int tip        = KW_GROUP_ID_2;
    int tipCount   = KW_NUM_GROUPS_2;

    if (pattern < patternCount) {
        int patternSorted  = dPatternsNewOrder[pattern];

        if (dTipTypes[tip] == 0) {
            int categoryOffset = category * stateCount * paddedPatternCount;

            int sortIndex   = categoryOffset + patternSorted * stateCount;
            int originIndex = categoryOffset + pattern       * stateCount;

            const KW_GLOBAL_VAR REAL* KW_RESTRICT partialOriginal = dPartials + dTipOffsets[tip];
                  KW_GLOBAL_VAR REAL* KW_RESTRICT partialSorted   = dPartials + dTipOffsets[tip+tipCount];

#ifdef FW_OPENCL_CPU
            for (int i=0; i < stateCount; i++) {
                partialSorted[sortIndex+i] = partialOriginal[originIndex+i];
            }
#else
            sortIndex += state;
            originIndex += state;
            partialSorted[sortIndex] = partialOriginal[originIndex];
#endif
        } else if (state == 0) {
            const KW_GLOBAL_VAR int* KW_RESTRICT stateOriginal = dStates     + dTipOffsets[tip];
                  KW_GLOBAL_VAR int* KW_RESTRICT stateSorted   = dStatesSort + dTipOffsets[tip+tipCount];

            stateSorted[patternSorted] = stateOriginal[pattern];
        }

        if (state == 0 && category == 0 && tip == 0) {
            dPatternWeightsSort[patternSorted] = dPatternWeights[pattern];
        }
    }
}

KW_GLOBAL_KERNEL void kernelMatrixMulADBMulti(KW_GLOBAL_VAR REAL* dMatrices,
                                              KW_GLOBAL_VAR unsigned int* offsets,
                                              KW_GLOBAL_VAR REAL* Alist,
                                              KW_GLOBAL_VAR REAL* Dlist,
                                              KW_GLOBAL_VAR REAL* Blist,
                                              KW_GLOBAL_VAR REAL* distanceQueue,
                                              int length,
                                              int wB,
                                              int totalMatrix) {

    int wMatrix = KW_GROUP_ID_0 % totalMatrix;
    int offIndex = wMatrix * 3;

    // Block index
    int bx = KW_GROUP_ID_0 / totalMatrix;
    int by = KW_GROUP_ID_1;

    // Thread index
    int tx = KW_LOCAL_ID_0;
    int ty = KW_LOCAL_ID_1;
    int BLOCKS = KW_NUM_GROUPS_1;

    KW_GLOBAL_VAR REAL* C = dMatrices + offsets[offIndex];
    KW_GLOBAL_VAR REAL* B = Blist + offsets[offIndex + 1]; // dEvec
    KW_GLOBAL_VAR REAL* A = Alist + offsets[offIndex + 1]; // dIevc
    KW_GLOBAL_VAR REAL* D = Dlist + offsets[offIndex + 2]; // dEigenValues
    REAL distance = distanceQueue[wMatrix];

    const int EDGE = PADDED_STATE_COUNT - (BLOCKS - 1) * MULTIPLY_BLOCK_SIZE;

    // Step size used to iterate through the sub-matrices of A
    int aStep = MULTIPLY_BLOCK_SIZE;

    // Step size used to iterate through the sub-matrices of B
    int bStep = MULTIPLY_BLOCK_SIZE * PADDED_STATE_COUNT;

    // Csub is used to store the element of the block sub-matrix
    // that is computed by the thread
    REAL Csub = 0;

    int a = PADDED_STATE_COUNT * MULTIPLY_BLOCK_SIZE * by;
    int b = MULTIPLY_BLOCK_SIZE * bx;
    int d = 0; //MULTIPLY_BLOCK_SIZE * bx;

    KW_LOCAL_MEM REAL As[MULTIPLY_BLOCK_SIZE][MULTIPLY_BLOCK_SIZE];
    KW_LOCAL_MEM REAL Bs[MULTIPLY_BLOCK_SIZE][MULTIPLY_BLOCK_SIZE];
    KW_LOCAL_MEM REAL Ds[MULTIPLY_BLOCK_SIZE];

    for (int i = 0; i < BLOCKS - 1; i++) {

        if (ty == 0)
            Ds[tx] = exp(D[d + tx] * distance);

        As[ty][tx] = A[a + PADDED_STATE_COUNT * ty + tx];
        Bs[ty][tx] = B[b + PADDED_STATE_COUNT * ty + tx];

        KW_LOCAL_FENCE;

        for (int k = 0; k < MULTIPLY_BLOCK_SIZE; ++k)
            Csub += As[ty][k] * Ds[k] * Bs[k][tx];

        KW_LOCAL_FENCE;

        a += aStep;
        b += bStep;
        d += MULTIPLY_BLOCK_SIZE;
    }

    // Last block is too long
    if (tx < EDGE && ty < EDGE) {
        if (ty == 0)
            Ds[tx] = exp(D[d + tx] * distance);

        As[ty][tx] = A[a + PADDED_STATE_COUNT * ty + tx];
        Bs[ty][tx] = B[b + PADDED_STATE_COUNT * ty + tx];

    } else {

        if (ty == 0)
            Ds[tx] = 0;

        As[ty][tx] = 0;
        Bs[ty][tx] = 0;
    }

    KW_LOCAL_FENCE;

    for (int k = 0; k < EDGE; k++)
        Csub += As[ty][k] * Ds[k] * Bs[k][tx];

    KW_LOCAL_FENCE;

    // Write the block sub-matrix to device memory;
    // each thread writes one element

    if ((tx < EDGE || bx < BLOCKS - 1) && (ty < EDGE || by < BLOCKS - 1)) { // It's OK to write
        if (Csub < 0)
            C[PADDED_STATE_COUNT* MULTIPLY_BLOCK_SIZE * by + MULTIPLY_BLOCK_SIZE * bx +
              PADDED_STATE_COUNT * ty + tx] = 0;
        else
            C[PADDED_STATE_COUNT* MULTIPLY_BLOCK_SIZE * by + MULTIPLY_BLOCK_SIZE * bx +
              PADDED_STATE_COUNT * ty + tx] = Csub;
    }
}

KW_GLOBAL_KERNEL void kernelMatrixMulADB(KW_GLOBAL_VAR REAL* dMatrices,
                                   KW_GLOBAL_VAR unsigned int* listC,
                                   KW_GLOBAL_VAR REAL* A,
                                   KW_GLOBAL_VAR REAL* D,
                                   KW_GLOBAL_VAR REAL* B,
                                   KW_GLOBAL_VAR REAL* distanceQueue,
                                   int length,
                                   int wB,
                                   int totalMatrix) {

    int wMatrix = KW_GROUP_ID_0 % totalMatrix;

    // Block index
    int bx = KW_GROUP_ID_0 / totalMatrix;
    int by = KW_GROUP_ID_1;

    // Thread index
    int tx = KW_LOCAL_ID_0;
    int ty = KW_LOCAL_ID_1;
    int BLOCKS = KW_NUM_GROUPS_1;

#ifdef CUDA
    KW_LOCAL_MEM REAL* C;
    KW_LOCAL_MEM REAL distance;
    if (tx == 0 && ty == 0) {
        C = dMatrices + listC[wMatrix]; // Non-coalescent read
        distance = distanceQueue[wMatrix]; // Non-coalescent read
    }
#elif defined(FW_OPENCL)
    KW_GLOBAL_VAR REAL* C;
    REAL distance;
    C = dMatrices + listC[wMatrix];
    distance = distanceQueue[wMatrix];
#endif

    KW_LOCAL_FENCE;

    const int EDGE = PADDED_STATE_COUNT - (BLOCKS - 1) * MULTIPLY_BLOCK_SIZE;

    // Step size used to iterate through the sub-matrices of A
    int aStep = MULTIPLY_BLOCK_SIZE;

    // Step size used to iterate through the sub-matrices of B
    int bStep = MULTIPLY_BLOCK_SIZE * PADDED_STATE_COUNT;

    // Csub is used to store the element of the block sub-matrix
    // that is computed by the thread
    REAL Csub = 0;

    int a = PADDED_STATE_COUNT * MULTIPLY_BLOCK_SIZE * by;
    int b = MULTIPLY_BLOCK_SIZE * bx;
    int d = 0; //MULTIPLY_BLOCK_SIZE * bx;

    KW_LOCAL_MEM REAL As[MULTIPLY_BLOCK_SIZE][MULTIPLY_BLOCK_SIZE];
    KW_LOCAL_MEM REAL Bs[MULTIPLY_BLOCK_SIZE][MULTIPLY_BLOCK_SIZE];
    KW_LOCAL_MEM REAL Ds[MULTIPLY_BLOCK_SIZE];

    for (int i = 0; i < BLOCKS - 1; i++) {

        if (ty == 0)
            Ds[tx] = exp(D[d + tx] * distance);

        As[ty][tx] = A[a + PADDED_STATE_COUNT * ty + tx];
        Bs[ty][tx] = B[b + PADDED_STATE_COUNT * ty + tx];

        KW_LOCAL_FENCE;

        for (int k = 0; k < MULTIPLY_BLOCK_SIZE; ++k)
            Csub += As[ty][k] * Ds[k] * Bs[k][tx];

        KW_LOCAL_FENCE;

        a += aStep;
        b += bStep;
        d += MULTIPLY_BLOCK_SIZE;
    }

    // Last block is too long
    if (tx < EDGE && ty < EDGE) {
        if (ty == 0)
            Ds[tx] = exp(D[d + tx] * distance);

        As[ty][tx] = A[a + PADDED_STATE_COUNT * ty + tx];
        Bs[ty][tx] = B[b + PADDED_STATE_COUNT * ty + tx];

    } else {

        if (ty == 0)
            Ds[tx] = 0;

        As[ty][tx] = 0;
        Bs[ty][tx] = 0;
    }

    KW_LOCAL_FENCE;

    for (int k = 0; k < EDGE; k++)
        Csub += As[ty][k] * Ds[k] * Bs[k][tx];

    KW_LOCAL_FENCE;

    // Write the block sub-matrix to device memory;
    // each thread writes one element

    if ((tx < EDGE || bx < BLOCKS - 1) && (ty < EDGE || by < BLOCKS - 1)) { // It's OK to write
        if (Csub < 0)
            C[PADDED_STATE_COUNT* MULTIPLY_BLOCK_SIZE * by + MULTIPLY_BLOCK_SIZE * bx +
              PADDED_STATE_COUNT * ty + tx] = 0;
        else
            C[PADDED_STATE_COUNT* MULTIPLY_BLOCK_SIZE * by + MULTIPLY_BLOCK_SIZE * bx +
              PADDED_STATE_COUNT * ty + tx] = Csub;
    }
}

KW_GLOBAL_KERNEL void kernelMatrixMulADBFirstDeriv(KW_GLOBAL_VAR REAL* dMatrices,
                                           KW_GLOBAL_VAR unsigned int* listC,
                                           KW_GLOBAL_VAR REAL* A,
                                           KW_GLOBAL_VAR REAL* D,
                                           KW_GLOBAL_VAR REAL* B,
                                           KW_GLOBAL_VAR REAL* distanceQueue,
                                           int length,
                                           int wB,
                                           int totalMatrix) {

    int wMatrix = KW_GROUP_ID_0 % totalMatrix;

    // Block index
    int bx = KW_GROUP_ID_0 / totalMatrix;
    int by = KW_GROUP_ID_1;

    // Thread index
    int tx = KW_LOCAL_ID_0;
    int ty = KW_LOCAL_ID_1;
    int BLOCKS = KW_NUM_GROUPS_1;

#ifdef CUDA
    KW_LOCAL_MEM REAL* C;
    KW_LOCAL_MEM REAL* CFirstDeriv;
    KW_LOCAL_MEM REAL distanceLength;
    KW_LOCAL_MEM REAL distanceRate;
    if (tx == 0 && ty == 0) {
        C = dMatrices + listC[wMatrix];
        CFirstDeriv = dMatrices + listC[wMatrix + totalMatrix];
        distanceLength = distanceQueue[wMatrix]; // Non-coalescent read
        distanceRate = distanceQueue[wMatrix + totalMatrix]; // Non-coalescent read
    }
#elif defined(FW_OPENCL)
    KW_GLOBAL_VAR REAL* C;
    KW_GLOBAL_VAR REAL* CFirstDeriv;
    REAL distanceLength;
    REAL distanceRate;
    C = dMatrices + listC[wMatrix];
    CFirstDeriv = dMatrices + listC[wMatrix + totalMatrix];
    distanceLength = distanceQueue[wMatrix];
    distanceRate = distanceQueue[wMatrix + totalMatrix];
#endif

    KW_LOCAL_FENCE;

    const int EDGE = PADDED_STATE_COUNT - (BLOCKS - 1) * MULTIPLY_BLOCK_SIZE;

    // Step size used to iterate through the sub-matrices of A
    int aStep = MULTIPLY_BLOCK_SIZE;

    // Step size used to iterate through the sub-matrices of B
    int bStep = MULTIPLY_BLOCK_SIZE * PADDED_STATE_COUNT;

    // Csub is used to store the element of the block sub-matrix
    // that is computed by the thread
    REAL Csub = 0;
    REAL CFirstDerivSub = 0;

    int a = PADDED_STATE_COUNT * MULTIPLY_BLOCK_SIZE * by;
    int b = MULTIPLY_BLOCK_SIZE * bx;
    int d = 0; //MULTIPLY_BLOCK_SIZE * bx;

    KW_LOCAL_MEM REAL As[MULTIPLY_BLOCK_SIZE][MULTIPLY_BLOCK_SIZE];
    KW_LOCAL_MEM REAL Bs[MULTIPLY_BLOCK_SIZE][MULTIPLY_BLOCK_SIZE];
    KW_LOCAL_MEM REAL Ds[MULTIPLY_BLOCK_SIZE][2];

    for (int i = 0; i < BLOCKS - 1; i++) {

        if (ty == 0) {
            REAL scaledEigenTmp = D[d + tx] * distanceRate;
            Ds[tx][0] = exp(scaledEigenTmp * distanceLength);
            Ds[tx][1] = scaledEigenTmp * Ds[tx][0];
        }

        As[ty][tx] = A[a + PADDED_STATE_COUNT * ty + tx];
        Bs[ty][tx] = B[b + PADDED_STATE_COUNT * ty + tx];

        KW_LOCAL_FENCE;

        for (int k = 0; k < MULTIPLY_BLOCK_SIZE; ++k) {
            Csub += As[ty][k] * Ds[k][0] * Bs[k][tx];
            CFirstDerivSub += As[ty][k] * Ds[k][1] * Bs[k][tx];
        }

        KW_LOCAL_FENCE;

        a += aStep;
        b += bStep;
        d += MULTIPLY_BLOCK_SIZE;
    }

    // Last block is too long
    if (tx < EDGE && ty < EDGE) {
        if (ty == 0) {
            REAL scaledEigenTmp = D[d + tx] * distanceRate;
            Ds[tx][0] = exp(scaledEigenTmp * distanceLength);
            Ds[tx][1] = scaledEigenTmp * Ds[tx][0];
                }

        As[ty][tx] = A[a + PADDED_STATE_COUNT * ty + tx];
        Bs[ty][tx] = B[b + PADDED_STATE_COUNT * ty + tx];

    } else {

        if (ty == 0) {
            Ds[tx][0] = 0;
            Ds[tx][1] = 0;
        }

        As[ty][tx] = 0;
        Bs[ty][tx] = 0;
    }

    KW_LOCAL_FENCE;

    for (int k = 0; k < EDGE; k++) {
        Csub += As[ty][k] * Ds[k][0] * Bs[k][tx];
        CFirstDerivSub += As[ty][k] * Ds[k][1] * Bs[k][tx];
    }

    KW_LOCAL_FENCE;

    // Write the block sub-matrix to device memory;
    // each thread writes one element

    if ((tx < EDGE || bx < BLOCKS - 1) && (ty < EDGE || by < BLOCKS - 1)) { // It's OK to write
        if (Csub < 0)
            C[PADDED_STATE_COUNT* MULTIPLY_BLOCK_SIZE * by + MULTIPLY_BLOCK_SIZE * bx +
              PADDED_STATE_COUNT * ty + tx] = 0;
        else
            C[PADDED_STATE_COUNT* MULTIPLY_BLOCK_SIZE * by + MULTIPLY_BLOCK_SIZE * bx +
              PADDED_STATE_COUNT * ty + tx] = Csub;

        CFirstDeriv[PADDED_STATE_COUNT* MULTIPLY_BLOCK_SIZE * by + MULTIPLY_BLOCK_SIZE * bx +
          PADDED_STATE_COUNT * ty + tx] = CFirstDerivSub;
    }
}

KW_GLOBAL_KERNEL void kernelMatrixMulADBSecondDeriv(KW_GLOBAL_VAR REAL* dMatrices,
                                           KW_GLOBAL_VAR unsigned int* listC,
                                           KW_GLOBAL_VAR REAL* A,
                                           KW_GLOBAL_VAR REAL* D,
                                           KW_GLOBAL_VAR REAL* B,
                                           KW_GLOBAL_VAR REAL* distanceQueue,
                                           int length,
                                           int wB,
                                           int totalMatrix) {

    int wMatrix = KW_GROUP_ID_0 % totalMatrix;

    // Block index
    int bx = KW_GROUP_ID_0 / totalMatrix;
    int by = KW_GROUP_ID_1;

    // Thread index
    int tx = KW_LOCAL_ID_0;
    int ty = KW_LOCAL_ID_1;
    int BLOCKS = KW_NUM_GROUPS_1;

#ifdef CUDA
    KW_LOCAL_MEM REAL* C;
    KW_LOCAL_MEM REAL* CFirstDeriv;
    KW_LOCAL_MEM REAL* CSecondDeriv;
    KW_LOCAL_MEM REAL distanceLength;
    KW_LOCAL_MEM REAL distanceRate;
    if (tx == 0 && ty == 0) {
        C = dMatrices + listC[wMatrix];
        CFirstDeriv = dMatrices + listC[wMatrix + totalMatrix];
        CSecondDeriv = dMatrices + listC[wMatrix + totalMatrix * 2];
        distanceLength = distanceQueue[wMatrix]; // Non-coalescent read
        distanceRate = distanceQueue[wMatrix + totalMatrix]; // Non-coalescent read
    }
#elif defined(FW_OPENCL)
    KW_GLOBAL_VAR REAL* C;
    KW_GLOBAL_VAR REAL* CFirstDeriv;
    KW_GLOBAL_VAR REAL* CSecondDeriv;
    REAL distanceLength;
    REAL distanceRate;
    C = dMatrices + listC[wMatrix];
    CFirstDeriv = dMatrices + listC[wMatrix + totalMatrix];
    CSecondDeriv = dMatrices + listC[wMatrix + totalMatrix * 2];
    distanceLength = distanceQueue[wMatrix];
    distanceRate = distanceQueue[wMatrix + totalMatrix];
#endif

    KW_LOCAL_FENCE;

    const int EDGE = PADDED_STATE_COUNT - (BLOCKS - 1) * MULTIPLY_BLOCK_SIZE;

    // Step size used to iterate through the sub-matrices of A
    int aStep = MULTIPLY_BLOCK_SIZE;

    // Step size used to iterate through the sub-matrices of B
    int bStep = MULTIPLY_BLOCK_SIZE * PADDED_STATE_COUNT;

    // Csub is used to store the element of the block sub-matrix
    // that is computed by the thread
    REAL Csub = 0;
    REAL CFirstDerivSub = 0;
    REAL CSecondDerivSub = 0;

    int a = PADDED_STATE_COUNT * MULTIPLY_BLOCK_SIZE * by;
    int b = MULTIPLY_BLOCK_SIZE * bx;
    int d = 0; //MULTIPLY_BLOCK_SIZE * bx;

    KW_LOCAL_MEM REAL As[MULTIPLY_BLOCK_SIZE][MULTIPLY_BLOCK_SIZE];
    KW_LOCAL_MEM REAL Bs[MULTIPLY_BLOCK_SIZE][MULTIPLY_BLOCK_SIZE];
    KW_LOCAL_MEM REAL Ds[MULTIPLY_BLOCK_SIZE][3];

    for (int i = 0; i < BLOCKS - 1; i++) {

        if (ty == 0) {
            REAL scaledEigenTmp = D[d + tx] * distanceRate;
            Ds[tx][0] = exp(scaledEigenTmp * distanceLength);
            Ds[tx][1] = scaledEigenTmp * Ds[tx][0];
            Ds[tx][2] = scaledEigenTmp * Ds[tx][1];
        }

        As[ty][tx] = A[a + PADDED_STATE_COUNT * ty + tx];
        Bs[ty][tx] = B[b + PADDED_STATE_COUNT * ty + tx];

        KW_LOCAL_FENCE;

        for (int k = 0; k < MULTIPLY_BLOCK_SIZE; ++k) {
            Csub += As[ty][k] * Ds[k][0] * Bs[k][tx];
            CFirstDerivSub += As[ty][k] * Ds[k][1] * Bs[k][tx];
            CSecondDerivSub += As[ty][k] * Ds[k][2] * Bs[k][tx];
        }

        KW_LOCAL_FENCE;

        a += aStep;
        b += bStep;
        d += MULTIPLY_BLOCK_SIZE;
    }

    // Last block is too long
    if (tx < EDGE && ty < EDGE) {
        if (ty == 0) {
            REAL scaledEigenTmp = D[d + tx] * distanceRate;
            Ds[tx][0] = exp(scaledEigenTmp * distanceLength);
            Ds[tx][1] = scaledEigenTmp * Ds[tx][0];
            Ds[tx][2] = scaledEigenTmp * Ds[tx][1];
                }

        As[ty][tx] = A[a + PADDED_STATE_COUNT * ty + tx];
        Bs[ty][tx] = B[b + PADDED_STATE_COUNT * ty + tx];

    } else {

        if (ty == 0) {
            Ds[tx][0] = 0;
            Ds[tx][1] = 0;
            Ds[tx][2] = 0;
        }

        As[ty][tx] = 0;
        Bs[ty][tx] = 0;
    }

    KW_LOCAL_FENCE;

    for (int k = 0; k < EDGE; k++) {
        Csub += As[ty][k] * Ds[k][0] * Bs[k][tx];
        CFirstDerivSub += As[ty][k] * Ds[k][1] * Bs[k][tx];
        CSecondDerivSub += As[ty][k] * Ds[k][2] * Bs[k][tx];
    }

    KW_LOCAL_FENCE;

    // Write the block sub-matrix to device memory;
    // each thread writes one element

    if ((tx < EDGE || bx < BLOCKS - 1) && (ty < EDGE || by < BLOCKS - 1)) { // It's OK to write
        if (Csub < 0)
            C[PADDED_STATE_COUNT* MULTIPLY_BLOCK_SIZE * by + MULTIPLY_BLOCK_SIZE * bx +
              PADDED_STATE_COUNT * ty + tx] = 0;
        else
            C[PADDED_STATE_COUNT* MULTIPLY_BLOCK_SIZE * by + MULTIPLY_BLOCK_SIZE * bx +
              PADDED_STATE_COUNT * ty + tx] = Csub;

        CFirstDeriv[PADDED_STATE_COUNT* MULTIPLY_BLOCK_SIZE * by + MULTIPLY_BLOCK_SIZE * bx +
          PADDED_STATE_COUNT * ty + tx] = CFirstDerivSub;

        CSecondDeriv[PADDED_STATE_COUNT* MULTIPLY_BLOCK_SIZE * by + MULTIPLY_BLOCK_SIZE * bx +
          PADDED_STATE_COUNT * ty + tx] = CSecondDerivSub;
    }
}

KW_GLOBAL_KERNEL void kernelMatrixConvolution(KW_GLOBAL_VAR REAL* dMatrices,
								        KW_GLOBAL_VAR unsigned int* list,
								        int totalMatrixCount
								        ) {

	    int wMatrix = KW_GROUP_ID_0 % totalMatrixCount;

	    // Block index
	    int bx = KW_GROUP_ID_0 / totalMatrixCount;
	    int by = KW_GROUP_ID_1;

	    // Thread index
	    int tx = KW_LOCAL_ID_0;
	    int ty = KW_LOCAL_ID_1;
	    int BLOCKS = KW_NUM_GROUPS_1;


#ifdef CUDA
        KW_LOCAL_MEM REAL* A;
        KW_LOCAL_MEM REAL* B;
        KW_LOCAL_MEM REAL* C;
        if (tx == 0 && ty == 0) {
            A = dMatrices + list[wMatrix]; // Non-coalescent read
            B = dMatrices + list[wMatrix + totalMatrixCount]; // Non-coalescent read
            C = dMatrices + list[wMatrix + totalMatrixCount*2]; // Non-coalescent read
        }
#elif defined(FW_OPENCL)
        KW_GLOBAL_VAR REAL* A;
        KW_GLOBAL_VAR REAL* B;
        KW_GLOBAL_VAR REAL* C;
        A = dMatrices + list[wMatrix];
        B = dMatrices + list[wMatrix + totalMatrixCount];
        C = dMatrices + list[wMatrix + totalMatrixCount*2];
#endif

	    KW_LOCAL_FENCE;

	    const int EDGE = PADDED_STATE_COUNT - (BLOCKS - 1) * MULTIPLY_BLOCK_SIZE;

	    // Step size used to iterate through the sub-matrices of A
	    int aStep = MULTIPLY_BLOCK_SIZE;

	    // Step size used to iterate through the sub-matrices of B
	    int bStep = MULTIPLY_BLOCK_SIZE * PADDED_STATE_COUNT;

	    // Csub is used to store the element of the block sub-matrix
	    // that is computed by the thread
	    REAL Csub = 0;

	    int a = PADDED_STATE_COUNT * MULTIPLY_BLOCK_SIZE * by;
	    int b = MULTIPLY_BLOCK_SIZE * bx;

	    KW_LOCAL_MEM REAL As[MULTIPLY_BLOCK_SIZE][MULTIPLY_BLOCK_SIZE];
	    KW_LOCAL_MEM REAL Bs[MULTIPLY_BLOCK_SIZE][MULTIPLY_BLOCK_SIZE];

	    for (int i = 0; i < BLOCKS - 1; i++) {

	        As[ty][tx] = A[a + PADDED_STATE_COUNT * ty + tx];
	        Bs[ty][tx] = B[b + PADDED_STATE_COUNT * ty + tx];

	        KW_LOCAL_FENCE;

	        for (int k = 0; k < MULTIPLY_BLOCK_SIZE; ++k)
	            Csub += As[ty][k]  * Bs[k][tx];

	        KW_LOCAL_FENCE;

	        a += aStep;
	        b += bStep;
	    }//END: BLOCKS loop

	    // Last block is too long
	    if (tx < EDGE && ty < EDGE) {

	#ifndef KERNEL_PRINT_ENABLED
	        KW_LOCAL_FENCE;
	#endif

	        As[ty][tx] = A[a + PADDED_STATE_COUNT * ty + tx];
	        Bs[ty][tx] = B[b + PADDED_STATE_COUNT * ty + tx];

	    } else {

	        As[ty][tx] = 0;
	        Bs[ty][tx] = 0;

	    }//END: EDGE check

	    KW_LOCAL_FENCE;

	    for (int k = 0; k < EDGE; k++) {
	        Csub += As[ty][k] *  Bs[k][tx];
	    }

	    KW_LOCAL_FENCE;

	    // Write the block sub-matrix to device memory;
	    // each thread writes one element

	    if ((tx < EDGE || bx < BLOCKS - 1) && (ty < EDGE || by < BLOCKS - 1)) { // It's OK to write
	        if (Csub < 0) {

	        	C[PADDED_STATE_COUNT* MULTIPLY_BLOCK_SIZE * by + MULTIPLY_BLOCK_SIZE * bx +
	              PADDED_STATE_COUNT * ty + tx] = 0;

	        } else {

	        	C[PADDED_STATE_COUNT* MULTIPLY_BLOCK_SIZE * by + MULTIPLY_BLOCK_SIZE * bx +
	              PADDED_STATE_COUNT * ty + tx] = Csub;

	        }//END: Csub check
	    }//END: EDGE check

}//END: kernelMatrixConvolution

KW_GLOBAL_KERNEL void kernelMatrixTranspose(KW_GLOBAL_VAR REAL* dMatrices,
                                            KW_GLOBAL_VAR unsigned int* list,
                                            int totalMatrixCount) {

	    int wMatrix = KW_GROUP_ID_0 % totalMatrixCount;

	    // Block index
	    int bx = KW_GROUP_ID_0 / totalMatrixCount;
	    int by = KW_GROUP_ID_1;

	    // Thread index
	    int tx = KW_LOCAL_ID_0;
	    int ty = KW_LOCAL_ID_1;

#ifdef CUDA
        KW_LOCAL_MEM REAL* A;
        KW_LOCAL_MEM REAL* C;
        if (tx == 0 && ty == 0) {
            A = dMatrices + list[wMatrix]; // Non-coalescent read
            C = dMatrices + list[wMatrix + totalMatrixCount]; // Non-coalescent read
        }
#elif defined(FW_OPENCL)
        KW_GLOBAL_VAR REAL* A;
        KW_GLOBAL_VAR REAL* C;
        A = dMatrices + list[wMatrix];
        C = dMatrices + list[wMatrix + totalMatrixCount];
#endif

	    KW_LOCAL_FENCE;

        const int rowOffset = MULTIPLY_BLOCK_SIZE * bx;
        const int colOffset = MULTIPLY_BLOCK_SIZE * by;

        const int row = rowOffset + tx;
        const int col = colOffset + ty;

	    KW_LOCAL_MEM REAL As[MULTIPLY_BLOCK_SIZE][MULTIPLY_BLOCK_SIZE];

	    if (row < PADDED_STATE_COUNT && col < PADDED_STATE_COUNT) {
	        As[ty][tx] = A[PADDED_STATE_COUNT * colOffset + rowOffset +
                           PADDED_STATE_COUNT * ty + tx];
	    }

	    KW_LOCAL_FENCE;

	    if (row < PADDED_STATE_COUNT && col < PADDED_STATE_COUNT) {
		    C[PADDED_STATE_COUNT * rowOffset + colOffset +
		      PADDED_STATE_COUNT * ty + tx] = As[tx][ty];
	    }
}

KW_GLOBAL_KERNEL void kernelMatrixMulADBComplexMulti(KW_GLOBAL_VAR REAL* dMatrices,
                                   KW_GLOBAL_VAR unsigned int* offsets,
                                   KW_GLOBAL_VAR REAL* Alist,
                                   KW_GLOBAL_VAR REAL* Dlist,
                                   KW_GLOBAL_VAR REAL* Blist,
                                   KW_GLOBAL_VAR REAL* distanceQueue,
                                   int length,
                                   int wB,
                                   int totalMatrix) {
#if !(defined(FW_OPENCL_APPLEAMDGPU) && defined(DOUBLE_PRECISION)) // TODO: fix this issue
    int wMatrix = KW_GROUP_ID_0 % totalMatrix;
    int offIndex = wMatrix * 3;

    // Block index
    int bx = KW_GROUP_ID_0 / totalMatrix;
    int by = KW_GROUP_ID_1;
    int BLOCKS = KW_NUM_GROUPS_1;

    // Thread index
    int tx = KW_LOCAL_ID_0;
    int ty = KW_LOCAL_ID_1;

    KW_GLOBAL_VAR REAL* C = dMatrices + offsets[offIndex];
    KW_GLOBAL_VAR REAL* B = Blist + offsets[offIndex + 1]; // dEvec
    KW_GLOBAL_VAR REAL* A = Alist + offsets[offIndex + 1]; // dIevc
    KW_GLOBAL_VAR REAL* D = Dlist + offsets[offIndex + 2]; // dEigenValues
    REAL distance = distanceQueue[wMatrix];

    const int EDGE = PADDED_STATE_COUNT - (BLOCKS - 1) * MULTIPLY_BLOCK_SIZE;

    // Step size used to iterate through the sub-matrices of A
    int aStep = MULTIPLY_BLOCK_SIZE;

    // Step size used to iterate through the sub-matrices of B
    int bStep = MULTIPLY_BLOCK_SIZE * PADDED_STATE_COUNT;

    // Csub is used to store the element of the block sub-matrix
    // that is computed by the thread
    REAL Csub = 0;

    int a = PADDED_STATE_COUNT * MULTIPLY_BLOCK_SIZE * by;
    int b = MULTIPLY_BLOCK_SIZE * bx;
    int d = 0; //MULTIPLY_BLOCK_SIZE * bx;

    KW_LOCAL_MEM REAL As[MULTIPLY_BLOCK_SIZE][MULTIPLY_BLOCK_SIZE];
    KW_LOCAL_MEM REAL Bs[MULTIPLY_BLOCK_SIZE + 2][MULTIPLY_BLOCK_SIZE];
    KW_LOCAL_MEM REAL Cs[MULTIPLY_BLOCK_SIZE];
    KW_LOCAL_MEM REAL Ds[MULTIPLY_BLOCK_SIZE];
    KW_LOCAL_MEM REAL Es[MULTIPLY_BLOCK_SIZE + 2];

#ifdef CUDA
    REAL* B0  = &Bs[1][0];
    REAL* Bm1 = &Bs[0][0];
    REAL* Bp1 = &Bs[2][0];
    REAL* E0  = &Es[1];
#elif defined(FW_OPENCL)
    KW_LOCAL_MEM REAL* B0  = &Bs[1][0];
    KW_LOCAL_MEM REAL* Bm1 = &Bs[0][0];
    KW_LOCAL_MEM REAL* Bp1 = &Bs[2][0];
    KW_LOCAL_MEM REAL* E0  = &Es[1];
#endif

    // Zero first row of Bs and Es
    if (ty == 0) {
        Bs[0][tx] = 0;
        if (tx == 0) {
            Es[0] = 0;
        }
    }

    while (d + MULTIPLY_BLOCK_SIZE < PADDED_STATE_COUNT) {

//      READ_SCHUR_VALUES();
        if (ty == 0) {
            Ds[tx] = exp(D[d + tx] * distance);
            Cs[tx] = D[d + PADDED_STATE_COUNT + tx] * distance;
            if (Cs[tx]) {
                REAL expat = Ds[tx];
                REAL cosbt = cos(Cs[tx]);
#ifdef FW_OPENCL_AMDGPU
                Cs[tx] = -expat * sin(Cs[tx] + 0.0);
#else
                Cs[tx] = -expat * sin(Cs[tx]);
#endif
                Ds[tx] *= cosbt;
            }
        }

        // Block read A and B sub-matrices
        As[ty][tx] = A[a + PADDED_STATE_COUNT * ty + tx];
        B0[ty * MULTIPLY_BLOCK_SIZE + tx] = B[b + PADDED_STATE_COUNT * ty + tx];

        // Read extra row of B for Bp1
        if (ty == 0) {
            B0[MULTIPLY_BLOCK_SIZE * MULTIPLY_BLOCK_SIZE + tx] =
                    B[b + PADDED_STATE_COUNT * MULTIPLY_BLOCK_SIZE + tx];
        }

        // All necessary values loaded
        KW_LOCAL_FENCE;

//      POPULATE_SCHUR_BAND(MULTIPLY_BLOCK_SIZE);
        if (ty == 0 && tx == 0) {
            for(int k=0; k<MULTIPLY_BLOCK_SIZE; k++) {
                if (Cs[k] && !Es[k]) {
                    E0[k] = Cs[k];
                } else {
                    E0[k] = 0;
                }
            }
        }


        KW_LOCAL_FENCE;

//      DO_MULTIPLICATION(MULTIPLY_BLOCK_SIZE);
        for (int k = 0; k < MULTIPLY_BLOCK_SIZE; k++) {
            Csub += As[ty][k] * (
                    Ds[k] * B0 [k * MULTIPLY_BLOCK_SIZE + tx]
                  + E0[k] * Bp1[k * MULTIPLY_BLOCK_SIZE + tx]
                  - Es[k] * Bm1[k * MULTIPLY_BLOCK_SIZE + tx]
            );
        }


        // Move last entries in B0 and E0 to first entries in Bs and Es
        if (ty == 0) {
            Bm1[tx] = Bm1[MULTIPLY_BLOCK_SIZE*MULTIPLY_BLOCK_SIZE + tx];
            if (tx == 0) {
                Es[0] = Es[MULTIPLY_BLOCK_SIZE];
            }
        }

        KW_LOCAL_FENCE;

        // Increment sub-matrices
        a += aStep;
        b += bStep;
        d += MULTIPLY_BLOCK_SIZE;

    }

    if (tx < EDGE && ty < EDGE) { // Last block is too long

//      READ_SCHUR_VALUES();
        if (ty == 0) {
            Ds[tx] = exp(D[d + tx] * distance);
            Cs[tx] = D[d + PADDED_STATE_COUNT + tx] * distance;
            if (Cs[tx]) {
                REAL expat = Ds[tx];
                REAL cosbt = cos(Cs[tx]);
#ifdef FW_OPENCL_AMDGPU
                Cs[tx] = -expat * sin(Cs[tx] + 0.0);
#else
                Cs[tx] = -expat * sin(Cs[tx]);
#endif
                Ds[tx] *= cosbt;
            }
        }

        As[ty][tx] = A[a + PADDED_STATE_COUNT * ty + tx];
        B0[ty * MULTIPLY_BLOCK_SIZE + tx] = B[b + PADDED_STATE_COUNT * ty + tx];

    } else {
        if (ty == 0) {
            Ds[tx] = 0;
            Cs[tx] = 0;
        }
        As[ty][tx] = 0;
        B0[ty * MULTIPLY_BLOCK_SIZE + tx] = 0;
    }

    // Zero last row of Bs and Es (only for unrolled iteration at end)
    if (ty == 0) {
        Bs[MULTIPLY_BLOCK_SIZE+1][tx] = 0;
    }

    // All necessary values loaded
    KW_LOCAL_FENCE;

//  POPULATE_SCHUR_BAND(EDGE);
    if (ty == 0 && tx == 0) {
        for(int k=0; k<EDGE; k++) {
            if (Cs[k] && !Es[k]) {
                E0[k] = Cs[k];
            } else {
                E0[k] = 0;
            }
        }
    }

    KW_LOCAL_FENCE;

    // Do matrix multiplication
//  DO_MULTIPLICATION(EDGE);
    for (int k = 0; k < EDGE; k++) {
        Csub += As[ty][k] * (
                Ds[k] * B0 [k * MULTIPLY_BLOCK_SIZE + tx]
              + E0[k] * Bp1[k * MULTIPLY_BLOCK_SIZE + tx]
              - Es[k] * Bm1[k * MULTIPLY_BLOCK_SIZE + tx]
        );
    }


    KW_LOCAL_FENCE;

    // Write the block sub-matrix to device memory;
    // each thread writes one element

    if (Csub < 0)
        Csub = 0;

    if ((tx < EDGE || bx < BLOCKS - 1) && (ty < EDGE || by < BLOCKS - 1)) { // It's OK to write
        C[PADDED_STATE_COUNT* MULTIPLY_BLOCK_SIZE * by + MULTIPLY_BLOCK_SIZE * bx +
              PADDED_STATE_COUNT * ty + tx] = Csub;
    }
#endif
}


KW_GLOBAL_KERNEL void kernelMatrixMulADBComplex(KW_GLOBAL_VAR REAL* dMatrices,
                                   KW_GLOBAL_VAR unsigned int* listC,
                                   KW_GLOBAL_VAR REAL* A,
                                   KW_GLOBAL_VAR REAL* D,
                                   KW_GLOBAL_VAR REAL* B,
                                   KW_GLOBAL_VAR REAL* distanceQueue,
                                   int length,
                                   int wB,
                                   int totalMatrix) {
#if !(defined(FW_OPENCL_APPLEAMDGPU) && defined(DOUBLE_PRECISION)) // TODO: fix this issue
    int wMatrix = KW_GROUP_ID_0 % totalMatrix;

    // Block index
    int bx = KW_GROUP_ID_0 / totalMatrix;
    int by = KW_GROUP_ID_1;
    int BLOCKS = KW_NUM_GROUPS_1;

    // Thread index
    int tx = KW_LOCAL_ID_0;
    int ty = KW_LOCAL_ID_1;

#ifdef CUDA
    KW_LOCAL_MEM REAL* C;
    KW_LOCAL_MEM REAL distance;
    if (tx == 0 && ty == 0) {
        C = dMatrices + listC[wMatrix];
        distance = distanceQueue[wMatrix]; // Non-coalescent read
    }
#elif defined(FW_OPENCL)
    KW_GLOBAL_VAR REAL* C;
    REAL distance;
    C = dMatrices + listC[wMatrix];
    distance = distanceQueue[wMatrix];
#endif

    KW_LOCAL_FENCE;

    const int EDGE = PADDED_STATE_COUNT - (BLOCKS - 1) * MULTIPLY_BLOCK_SIZE;

    // Step size used to iterate through the sub-matrices of A
    int aStep = MULTIPLY_BLOCK_SIZE;

    // Step size used to iterate through the sub-matrices of B
    int bStep = MULTIPLY_BLOCK_SIZE * PADDED_STATE_COUNT;

    // Csub is used to store the element of the block sub-matrix
    // that is computed by the thread
    REAL Csub = 0;

    int a = PADDED_STATE_COUNT * MULTIPLY_BLOCK_SIZE * by;
    int b = MULTIPLY_BLOCK_SIZE * bx;
    int d = 0; //MULTIPLY_BLOCK_SIZE * bx;

    KW_LOCAL_MEM REAL As[MULTIPLY_BLOCK_SIZE][MULTIPLY_BLOCK_SIZE];
    KW_LOCAL_MEM REAL Bs[MULTIPLY_BLOCK_SIZE + 2][MULTIPLY_BLOCK_SIZE];
    KW_LOCAL_MEM REAL Cs[MULTIPLY_BLOCK_SIZE];
    KW_LOCAL_MEM REAL Ds[MULTIPLY_BLOCK_SIZE];
    KW_LOCAL_MEM REAL Es[MULTIPLY_BLOCK_SIZE + 2];

#ifdef CUDA
   	REAL* B0  = &Bs[1][0];
   	REAL* Bm1 = &Bs[0][0];
   	REAL* Bp1 = &Bs[2][0];
   	REAL* E0  = &Es[1];
#elif defined(FW_OPENCL)
   	KW_LOCAL_MEM REAL* B0  = &Bs[1][0];
   	KW_LOCAL_MEM REAL* Bm1 = &Bs[0][0];
   	KW_LOCAL_MEM REAL* Bp1 = &Bs[2][0];
   	KW_LOCAL_MEM REAL* E0  = &Es[1];
#endif

   	// Zero first row of Bs and Es
   	if (ty == 0) {
   		Bs[0][tx] = 0;
   		if (tx == 0) {
   			Es[0] = 0;
   		}
   	}

    while (d + MULTIPLY_BLOCK_SIZE < PADDED_STATE_COUNT) {

//      READ_SCHUR_VALUES();
		if (ty == 0) {
			Ds[tx] = exp(D[d + tx] * distance);
			Cs[tx] = D[d + PADDED_STATE_COUNT + tx] * distance;
			if (Cs[tx]) {
            	REAL expat = Ds[tx];
            	REAL cosbt = cos(Cs[tx]);
#ifdef FW_OPENCL_AMDGPU
                Cs[tx] = -expat * sin(Cs[tx] + 0.0);
#else
                Cs[tx] = -expat * sin(Cs[tx]);
#endif
            	Ds[tx] *= cosbt;
            }
        }

        // Block read A and B sub-matrices
        As[ty][tx] = A[a + PADDED_STATE_COUNT * ty + tx];
        B0[ty * MULTIPLY_BLOCK_SIZE + tx] = B[b + PADDED_STATE_COUNT * ty + tx];

        // Read extra row of B for Bp1
        if (ty == 0) {
        	B0[MULTIPLY_BLOCK_SIZE * MULTIPLY_BLOCK_SIZE + tx] =
        			B[b + PADDED_STATE_COUNT * MULTIPLY_BLOCK_SIZE + tx];
        }

        // All necessary values loaded
    	KW_LOCAL_FENCE;

//    	POPULATE_SCHUR_BAND(MULTIPLY_BLOCK_SIZE);
		if (ty == 0 && tx == 0) {
			for(int k=0; k<MULTIPLY_BLOCK_SIZE; k++) {
				if (Cs[k] && !Es[k]) {
					E0[k] = Cs[k];
				} else {
					E0[k] = 0;
				}
			}
		}


    	KW_LOCAL_FENCE;

//      DO_MULTIPLICATION(MULTIPLY_BLOCK_SIZE);
		for (int k = 0; k < MULTIPLY_BLOCK_SIZE; k++) {
			Csub += As[ty][k] * (
					Ds[k] * B0 [k * MULTIPLY_BLOCK_SIZE + tx]
				  + E0[k] * Bp1[k * MULTIPLY_BLOCK_SIZE + tx]
				  - Es[k] * Bm1[k * MULTIPLY_BLOCK_SIZE + tx]
			);
		}


        // Move last entries in B0 and E0 to first entries in Bs and Es
        if (ty == 0) {
        	Bm1[tx] = Bm1[MULTIPLY_BLOCK_SIZE*MULTIPLY_BLOCK_SIZE + tx];
        	if (tx == 0) {
        		Es[0] = Es[MULTIPLY_BLOCK_SIZE];
        	}
        }

        KW_LOCAL_FENCE;

        // Increment sub-matrices
        a += aStep;
        b += bStep;
        d += MULTIPLY_BLOCK_SIZE;

    }

    if (tx < EDGE && ty < EDGE) { // Last block is too long

//      READ_SCHUR_VALUES();
		if (ty == 0) {
			Ds[tx] = exp(D[d + tx] * distance);
			Cs[tx] = D[d + PADDED_STATE_COUNT + tx] * distance;
			if (Cs[tx]) {
            	REAL expat = Ds[tx];
            	REAL cosbt = cos(Cs[tx]);
#ifdef FW_OPENCL_AMDGPU
            	Cs[tx] = -expat * sin(Cs[tx] + 0.0);
#else
                Cs[tx] = -expat * sin(Cs[tx]);
#endif
            	Ds[tx] *= cosbt;
            }
        }

        As[ty][tx] = A[a + PADDED_STATE_COUNT * ty + tx];
        B0[ty * MULTIPLY_BLOCK_SIZE + tx] = B[b + PADDED_STATE_COUNT * ty + tx];

    } else {
    	if (ty == 0) {
    		Ds[tx] = 0;
    		Cs[tx] = 0;
    	}
    	As[ty][tx] = 0;
    	B0[ty * MULTIPLY_BLOCK_SIZE + tx] = 0;
    }

	// Zero last row of Bs and Es (only for unrolled iteration at end)
    if (ty == 0) {
    	Bs[MULTIPLY_BLOCK_SIZE+1][tx] = 0;
    }

    // All necessary values loaded
	KW_LOCAL_FENCE;

//	POPULATE_SCHUR_BAND(EDGE);
    if (ty == 0 && tx == 0) {
        for(int k=0; k<EDGE; k++) {
            if (Cs[k] && !Es[k]) {
                E0[k] = Cs[k];
            } else {
                E0[k] = 0;
            }
        }
    }

	KW_LOCAL_FENCE;

	// Do matrix multiplication
//	DO_MULTIPLICATION(EDGE);
    for (int k = 0; k < EDGE; k++) {
        Csub += As[ty][k] * (
                Ds[k] * B0 [k * MULTIPLY_BLOCK_SIZE + tx]
              + E0[k] * Bp1[k * MULTIPLY_BLOCK_SIZE + tx]
              - Es[k] * Bm1[k * MULTIPLY_BLOCK_SIZE + tx]
        );
    }


    KW_LOCAL_FENCE;

    // Write the block sub-matrix to device memory;
    // each thread writes one element

    if (Csub < 0)
    	Csub = 0;

    if ((tx < EDGE || bx < BLOCKS - 1) && (ty < EDGE || by < BLOCKS - 1)) { // It's OK to write
        C[PADDED_STATE_COUNT* MULTIPLY_BLOCK_SIZE * by + MULTIPLY_BLOCK_SIZE * bx +
              PADDED_STATE_COUNT * ty + tx] = Csub;
    }
#endif
}

KW_GLOBAL_KERNEL void kernelSumSites1(KW_GLOBAL_VAR REAL* dArray,
                                      KW_GLOBAL_VAR REAL* dSum,
                                      KW_GLOBAL_VAR REAL* dPatternWeights,
                                      int patternCount) {
#ifdef FW_OPENCL_CPU

    REAL sum = 0;

    int pattern = KW_GROUP_ID_0 * SUM_SITES_BLOCK_SIZE;
    int maxPattern = (KW_GROUP_ID_0 + 1) * SUM_SITES_BLOCK_SIZE;

    if (maxPattern > patternCount)
        maxPattern = patternCount;

    while (pattern < maxPattern) {
        FMA(dArray[pattern],  dPatternWeights[pattern], sum);
        pattern++;
    }

    dSum[KW_GROUP_ID_0] = sum;

#else

    KW_LOCAL_MEM REAL sum[SUM_SITES_BLOCK_SIZE];

    int tx = KW_LOCAL_ID_0;
    int pattern = KW_LOCAL_ID_0 + KW_GROUP_ID_0 * SUM_SITES_BLOCK_SIZE;

    if (pattern < patternCount)
        sum[tx] = dArray[pattern] * dPatternWeights[pattern];
    else
        sum[tx] = 0.0;

    KW_LOCAL_FENCE;

    for (unsigned int s = SUM_SITES_BLOCK_SIZE / 2; s > 0; s >>= 1) {
        if (tx < s)
            sum[tx] += sum[tx + s];
        KW_LOCAL_FENCE;
    }

    if (tx == 0)
        dSum[KW_GROUP_ID_0] = sum[0];

#endif
}

KW_GLOBAL_KERNEL void kernelSumSites1Partition(KW_GLOBAL_VAR REAL* dArray,
                                               KW_GLOBAL_VAR REAL* dSum,
                                               KW_GLOBAL_VAR REAL* dPatternWeights,
                                               int startPattern,
                                               int endPattern) {
#ifdef FW_OPENCL_CPU

    REAL sum = 0;

    int pattern = startPattern + KW_GROUP_ID_0 * SUM_SITES_BLOCK_SIZE;
    int maxPattern = startPattern + (KW_GROUP_ID_0 + 1) * SUM_SITES_BLOCK_SIZE;

    if (maxPattern > endPattern)
        maxPattern = endPattern;

    while (pattern < maxPattern) {
        FMA(dArray[pattern],  dPatternWeights[pattern], sum);
        pattern++;
    }

    dSum[KW_GROUP_ID_0] = sum;

#else

    KW_LOCAL_MEM REAL sum[SUM_SITES_BLOCK_SIZE];

    int tx = KW_LOCAL_ID_0;
    int pattern = startPattern + KW_LOCAL_ID_0 + KW_GROUP_ID_0 * SUM_SITES_BLOCK_SIZE;

    if (pattern < endPattern)
        sum[tx] = dArray[pattern] * dPatternWeights[pattern];
    else
        sum[tx] = 0.0;

    KW_LOCAL_FENCE;

    for (unsigned int s = SUM_SITES_BLOCK_SIZE / 2; s > 0; s >>= 1) {
        if (tx < s)
            sum[tx] += sum[tx + s];
        KW_LOCAL_FENCE;
    }

    if (tx == 0)
        dSum[KW_GROUP_ID_0] = sum[0];

#endif
}

// KW_GLOBAL_KERNEL void kernelSumSites1Partition(KW_GLOBAL_VAR REAL*         dArray,
//                                                KW_GLOBAL_VAR REAL*         dSum,
//                                                KW_GLOBAL_VAR REAL*         dPatternWeights,
//                                                KW_GLOBAL_VAR unsigned int* dPtrOffsets) {

//     int opIndexPtr = KW_GROUP_ID_0 * 2;
//     int startPattern = dPtrOffsets[opIndexPtr    ];
//     int endPattern   = dPtrOffsets[opIndexPtr + 1];

// #ifdef FW_OPENCL_CPU

//     REAL sum = 0;

//     int pattern = startPattern + KW_GROUP_ID_0 * SUM_SITES_BLOCK_SIZE;

//     while (pattern < endPattern) {
//         FMA(dArray[pattern],  dPatternWeights[pattern], sum);
//         pattern++;
//     }

//     dSum[KW_GROUP_ID_0] = sum;

// #else

//     KW_LOCAL_MEM REAL sum[SUM_SITES_BLOCK_SIZE];

//     int tx = KW_LOCAL_ID_0;
//     int pattern = startPattern + KW_LOCAL_ID_0 + KW_GROUP_ID_0 * SUM_SITES_BLOCK_SIZE;

//     if (pattern < endPattern)
//         sum[tx] = dArray[pattern] * dPatternWeights[pattern];
//     else
//         sum[tx] = 0.0;

//     KW_LOCAL_FENCE;

//     for (unsigned int s = SUM_SITES_BLOCK_SIZE / 2; s > 0; s >>= 1) {
//         if (tx < s)
//             sum[tx] += sum[tx + s];
//         KW_LOCAL_FENCE;
//     }

//     if (tx == 0)
//         dSum[KW_GROUP_ID_0] = sum[0];

// #endif
// }

KW_GLOBAL_KERNEL void kernelSumSites2(KW_GLOBAL_VAR REAL* dArray1,
                                      KW_GLOBAL_VAR REAL* dSum1,
                                      KW_GLOBAL_VAR REAL* dArray2,
                                      KW_GLOBAL_VAR REAL* dSum2,
                                      KW_GLOBAL_VAR REAL* dPatternWeights,
                                      int patternCount) {

#ifdef FW_OPENCL_CPU

    REAL sum1 = 0, sum2 = 0;

    int pattern = KW_GROUP_ID_0 * SUM_SITES_BLOCK_SIZE;
    int maxPattern = (KW_GROUP_ID_0 + 1) * SUM_SITES_BLOCK_SIZE;

    if (maxPattern > patternCount)
        maxPattern = patternCount;

    while (pattern < maxPattern) {
        FMA(dArray1[pattern],  dPatternWeights[pattern], sum1);
        FMA(dArray2[pattern],  dPatternWeights[pattern], sum2);
        pattern++;
    }

    dSum1[KW_GROUP_ID_0] = sum1;
    dSum2[KW_GROUP_ID_0] = sum2;

#else

    KW_LOCAL_MEM REAL sum1[SUM_SITES_BLOCK_SIZE];
    KW_LOCAL_MEM REAL sum2[SUM_SITES_BLOCK_SIZE];

    int tx = KW_LOCAL_ID_0;
    int pattern = KW_LOCAL_ID_0 + KW_GROUP_ID_0 * SUM_SITES_BLOCK_SIZE;

    if (pattern < patternCount) {
        REAL pWeight = dPatternWeights[pattern];
        sum1[tx] = dArray1[pattern] * pWeight;
        sum2[tx] = dArray2[pattern] * pWeight;
    } else {
        sum1[tx] = 0.0;
        sum2[tx] = 0.0;
    }

    KW_LOCAL_FENCE;

    for (unsigned int s = SUM_SITES_BLOCK_SIZE / 2; s > 0; s >>= 1) {
        if (tx < s) {
            sum1[tx] += sum1[tx + s];
            sum2[tx] += sum2[tx + s];
        }
        KW_LOCAL_FENCE;
    }

    if (tx == 0) {
        dSum1[KW_GROUP_ID_0] = sum1[0];
        dSum2[KW_GROUP_ID_0] = sum2[0];
    }

#endif
}

KW_GLOBAL_KERNEL void kernelSumSites3(KW_GLOBAL_VAR REAL* dArray1,
                                      KW_GLOBAL_VAR REAL* dSum1,
                                      KW_GLOBAL_VAR REAL* dArray2,
                                      KW_GLOBAL_VAR REAL* dSum2,
                                      KW_GLOBAL_VAR REAL* dArray3,
                                      KW_GLOBAL_VAR REAL* dSum3,
                                      KW_GLOBAL_VAR REAL* dPatternWeights,
                                      int patternCount) {

#ifdef FW_OPENCL_CPU

    REAL sum1 = 0, sum2 = 0, sum3 = 0;

    int pattern = KW_GROUP_ID_0 * SUM_SITES_BLOCK_SIZE;
    int maxPattern = (KW_GROUP_ID_0 + 1) * SUM_SITES_BLOCK_SIZE;

    if (maxPattern > patternCount)
        maxPattern = patternCount;

    while (pattern < maxPattern) {
        FMA(dArray1[pattern],  dPatternWeights[pattern], sum1);
        FMA(dArray2[pattern],  dPatternWeights[pattern], sum2);
        FMA(dArray3[pattern],  dPatternWeights[pattern], sum3);

        pattern++;
    }

    dSum1[KW_GROUP_ID_0] = sum1;
    dSum2[KW_GROUP_ID_0] = sum2;
    dSum3[KW_GROUP_ID_0] = sum3;

#else

    KW_LOCAL_MEM REAL sum1[SUM_SITES_BLOCK_SIZE];
    KW_LOCAL_MEM REAL sum2[SUM_SITES_BLOCK_SIZE];
    KW_LOCAL_MEM REAL sum3[SUM_SITES_BLOCK_SIZE];

    int tx = KW_LOCAL_ID_0;
    int pattern = KW_LOCAL_ID_0 + KW_GROUP_ID_0 * SUM_SITES_BLOCK_SIZE;

    if (pattern < patternCount) {
        REAL pWeight = dPatternWeights[pattern];
        sum1[tx] = dArray1[pattern] * pWeight;
        sum2[tx] = dArray2[pattern] * pWeight;
        sum3[tx] = dArray3[pattern] * pWeight;
    } else {
        sum1[tx] = 0.0;
        sum2[tx] = 0.0;
        sum3[tx] = 0.0;
    }

    KW_LOCAL_FENCE;

    for (unsigned int s = SUM_SITES_BLOCK_SIZE / 2; s > 0; s >>= 1) {
        if (tx < s) {
            sum1[tx] += sum1[tx + s];
            sum2[tx] += sum2[tx + s];
            sum3[tx] += sum3[tx + s];
        }
        KW_LOCAL_FENCE;
    }

    if (tx == 0) {
        dSum1[KW_GROUP_ID_0] = sum1[0];
        dSum2[KW_GROUP_ID_0] = sum2[0];
        dSum3[KW_GROUP_ID_0] = sum3[0];
    }

#endif
}

KW_GLOBAL_KERNEL void kernelAccumulateFactors(KW_GLOBAL_VAR REAL* dScalingFactors,
                                              KW_GLOBAL_VAR unsigned int* dNodePtrQueue,
                                              KW_GLOBAL_VAR REAL* rootScaling,
                                              int nodeCount,
                                              int patternCount) {

    int pattern = KW_LOCAL_ID_0 + KW_GROUP_ID_0 * PATTERN_BLOCK_SIZE;

    REAL total = 0;
    KW_GLOBAL_VAR REAL* nodeScales;

    int n;
    for(n = 0; n < nodeCount; n++) {
//      if (KW_LOCAL_ID_0 == 0) // TODO Why does this not work???
        nodeScales = dScalingFactors + dNodePtrQueue[n];
//      KW_LOCAL_FENCE;

    #ifdef KERNEL_PRINT_ENABLED
        if (pattern == 1)
            printf("added %1.2e\n", nodeScales[pattern]);
    #endif
        REAL factor = nodeScales[pattern];
        if (factor != 1.0) {
            total += log(factor);
        }
    }

#ifdef FW_OPENCL_CPU // CPU/MIC implementation
    rootScaling[pattern] += total;
#else // GPU implementation
    if (pattern < patternCount)
        rootScaling[pattern] += total;
#endif // FW_OPENCL_CPU
}

KW_GLOBAL_KERNEL void kernelAccumulateFactorsByPartition(KW_GLOBAL_VAR REAL* dScalingFactors,
                                                         KW_GLOBAL_VAR unsigned int* dNodePtrQueue,
                                                         KW_GLOBAL_VAR REAL* rootScaling,
                                                         int nodeCount,
                                                         int startPattern,
                                                         int endPattern) {

    int pattern = startPattern + KW_LOCAL_ID_0 + KW_GROUP_ID_0 * PATTERN_BLOCK_SIZE;

    REAL total = 0;
    KW_GLOBAL_VAR REAL* nodeScales;

    int n;
    for(n = 0; n < nodeCount; n++) {
        nodeScales = dScalingFactors + dNodePtrQueue[n];

        REAL factor = nodeScales[pattern];
        if (factor != 1.0) {
            total += log(factor);
        }
    }

    if (pattern < endPattern) {
        rootScaling[pattern] += total;
    }
}

KW_GLOBAL_KERNEL void kernelAccumulateFactorsScalersLog(KW_GLOBAL_VAR REAL* dScalingFactors,
                                                 KW_GLOBAL_VAR unsigned int* dNodePtrQueue,
                                                 KW_GLOBAL_VAR REAL* rootScaling,
                                                 int nodeCount,
                                                 int patternCount) {
    int pattern = KW_LOCAL_ID_0 + KW_GROUP_ID_0 * PATTERN_BLOCK_SIZE;

    REAL total = 0;
    KW_GLOBAL_VAR REAL* nodeScales;

    int n;
    for(n = 0; n < nodeCount; n++) {
//      if (KW_LOCAL_ID_0 == 0) // TODO Why does this not work???
        nodeScales = dScalingFactors + dNodePtrQueue[n];
//      KW_LOCAL_FENCE;

#ifdef KERNEL_PRINT_ENABLED
        if (pattern == 1)
            printf("added %1.2e\n", nodeScales[pattern]);
#endif
        total += nodeScales[pattern];
    }

#ifdef FW_OPENCL_CPU // CPU/MIC implementation
    rootScaling[pattern] += total;
#else // GPU implementation
    if (pattern < patternCount)
        rootScaling[pattern] += total;
#endif // FW_OPENCL_CPU
}

KW_GLOBAL_KERNEL void kernelAccumulateFactorsScalersLogByPartition(
                                                KW_GLOBAL_VAR REAL* dScalingFactors,
                                                KW_GLOBAL_VAR unsigned int* dNodePtrQueue,
                                                KW_GLOBAL_VAR REAL* rootScaling,
                                                int nodeCount,
                                                int startPattern,
                                                int endPattern) {

    int pattern = startPattern + KW_LOCAL_ID_0 + KW_GROUP_ID_0 * PATTERN_BLOCK_SIZE;

    REAL total = 0;
    KW_GLOBAL_VAR REAL* nodeScales;

    int n;
    for(n = 0; n < nodeCount; n++) {
        nodeScales = dScalingFactors + dNodePtrQueue[n];

        total += nodeScales[pattern];
    }

    if (pattern < endPattern) {
        rootScaling[pattern] += total;
    }
}

KW_GLOBAL_KERNEL void kernelRemoveFactors(KW_GLOBAL_VAR REAL* dScalingFactors,
                                    KW_GLOBAL_VAR unsigned int* dNodePtrQueue,
                                                   KW_GLOBAL_VAR REAL* rootScaling,
                                                   int nodeCount,
                                                   int patternCount) {
    int pattern = KW_LOCAL_ID_0 + KW_GROUP_ID_0 * PATTERN_BLOCK_SIZE;

    REAL total = 0;
    KW_GLOBAL_VAR REAL* nodeScales;

    int n;
    for(n = 0; n < nodeCount; n++) {
//      if (KW_LOCAL_ID_0 == 0) // TODO Why does this not work???
        nodeScales = dScalingFactors + dNodePtrQueue[n];
//      KW_LOCAL_FENCE;

#ifdef KERNEL_PRINT_ENABLED
        if (pattern == 1)
            printf("added %1.2e\n", nodeScales[pattern]);
#endif
        REAL factor = nodeScales[pattern];
        if (factor != 1.0) {
            total += log(factor);
        }
    }

#ifdef FW_OPENCL_CPU // CPU/MIC implementation
    rootScaling[pattern] -= total;
#else // GPU implementation
    if (pattern < patternCount)
        rootScaling[pattern] -= total;
#endif // FW_OPENCL_CPU
}

KW_GLOBAL_KERNEL void kernelRemoveFactorsByPartition(KW_GLOBAL_VAR REAL* dScalingFactors,
                                                     KW_GLOBAL_VAR unsigned int* dNodePtrQueue,
                                                     KW_GLOBAL_VAR REAL* rootScaling,
                                                     int nodeCount,
                                                     int startPattern,
                                                     int endPattern) {
    int pattern = startPattern + KW_LOCAL_ID_0 + KW_GROUP_ID_0 * PATTERN_BLOCK_SIZE;

    REAL total = 0;
    KW_GLOBAL_VAR REAL* nodeScales;

    int n;
    for(n = 0; n < nodeCount; n++) {
        nodeScales = dScalingFactors + dNodePtrQueue[n];

        REAL factor = nodeScales[pattern];
        if (factor != 1.0) {
            total += log(factor);
        }
    }

    if (pattern < endPattern) {
        rootScaling[pattern] -= total;
    }
}

KW_GLOBAL_KERNEL void kernelRemoveFactorsScalersLog(KW_GLOBAL_VAR REAL* dScalingFactors,
                                             KW_GLOBAL_VAR unsigned int* dNodePtrQueue,
                                             KW_GLOBAL_VAR REAL* rootScaling,
                                             int nodeCount,
                                             int patternCount) {
    int pattern = KW_LOCAL_ID_0 + KW_GROUP_ID_0 * PATTERN_BLOCK_SIZE;

    REAL total = 0;
    KW_GLOBAL_VAR REAL* nodeScales;

    int n;
    for(n = 0; n < nodeCount; n++) {
//      if (KW_LOCAL_ID_0 == 0) // TODO Why does this not work???
        nodeScales = dScalingFactors + dNodePtrQueue[n];
//      KW_LOCAL_FENCE;

#ifdef KERNEL_PRINT_ENABLED
        if (pattern == 1)
            printf("added %1.2e\n", nodeScales[pattern]);
#endif

        total += nodeScales[pattern];
    }

#ifdef FW_OPENCL_CPU // CPU/MIC implementation
    rootScaling[pattern] -= total;
#else // GPU implementation
    if (pattern < patternCount)
        rootScaling[pattern] -= total;
#endif // FW_OPENCL_CPU
}

KW_GLOBAL_KERNEL void kernelRemoveFactorsScalersLogByPartition(KW_GLOBAL_VAR REAL* dScalingFactors,
                                                               KW_GLOBAL_VAR unsigned int* dNodePtrQueue,
                                                               KW_GLOBAL_VAR REAL* rootScaling,
                                                               int nodeCount,
                                                               int startPattern,
                                                               int endPattern) {
    int pattern = startPattern + KW_LOCAL_ID_0 + KW_GROUP_ID_0 * PATTERN_BLOCK_SIZE;

    REAL total = 0;
    KW_GLOBAL_VAR REAL* nodeScales;

    int n;
    for(n = 0; n < nodeCount; n++) {
        nodeScales = dScalingFactors + dNodePtrQueue[n];

        total += nodeScales[pattern];
    }

    if (pattern < endPattern)
        rootScaling[pattern] -= total;

}

KW_GLOBAL_KERNEL void kernelResetFactorsByPartition(KW_GLOBAL_VAR REAL* dScalingFactors,
                                                    int startPattern,
                                                    int endPattern) {
    int pattern = startPattern + KW_LOCAL_ID_0 + KW_GROUP_ID_0 * PATTERN_BLOCK_SIZE;

    if (pattern < endPattern) {
        dScalingFactors[pattern] = 0.0;
    }
}


KW_GLOBAL_KERNEL void kernelPartialsDynamicScalingSlow(KW_GLOBAL_VAR REAL* allPartials,
                                                 KW_GLOBAL_VAR REAL* scalingFactors,
                                                 int matrixCount) {
    int state = KW_LOCAL_ID_0;
    int pattern = KW_GROUP_ID_0;
    int patternCount = KW_NUM_GROUPS_0;

    KW_LOCAL_MEM REAL partials[PADDED_STATE_COUNT];

    KW_LOCAL_MEM REAL max;

    if (state == 0)
        max = 0.0;

    int m;
    for(m = 0; m < matrixCount; m++) {
        partials[state] = allPartials[m * patternCount * PADDED_STATE_COUNT + pattern *
                                      PADDED_STATE_COUNT + state];
        KW_LOCAL_FENCE;

#ifdef IS_POWER_OF_TWO
    // parallelized reduction *** only works for powers-of-2 ****
    for (int i = PADDED_STATE_COUNT / 2; i > 0; i >>= 1) {
        if (state < i) {
#else
    for (int i = SMALLEST_POWER_OF_TWO / 2; i > 0; i >>= 1) {
        if (state < i && state + i < PADDED_STATE_COUNT ) {
#endif // IS_POWER_OF_TWO
                REAL compare1 = partials[state];
                REAL compare2 = partials[state + i];
                if(compare2 > compare1)
                    partials[state] = compare2;
            }
            KW_LOCAL_FENCE;
        }
        if(state == 0) {
            if( partials[0] > max)
                max = partials[0];
        }
    }

    if(state == 0) {
        if (max == 0)
        	max = 1.0;
        scalingFactors[pattern] = max;
    }


    KW_LOCAL_FENCE;

    for(m = 0; m < matrixCount; m++)
        allPartials[m * patternCount * PADDED_STATE_COUNT + pattern * PADDED_STATE_COUNT +
                    state] /= max;

}

KW_GLOBAL_KERNEL void kernelPartialsDynamicScalingSlowScalersLog(KW_GLOBAL_VAR REAL* allPartials,
                                                          KW_GLOBAL_VAR REAL* scalingFactors,
                                                          int matrixCount) {
    int state = KW_LOCAL_ID_0;
    int pattern = KW_GROUP_ID_0;
    int patternCount = KW_NUM_GROUPS_0;

    KW_LOCAL_MEM REAL partials[PADDED_STATE_COUNT];

    KW_LOCAL_MEM REAL max;

    if (state == 0)
        max = 0.0;

    int m;
    for(m = 0; m < matrixCount; m++) {
        partials[state] = allPartials[m * patternCount * PADDED_STATE_COUNT + pattern *
                                      PADDED_STATE_COUNT + state];
        KW_LOCAL_FENCE;

#ifdef IS_POWER_OF_TWO
    // parallelized reduction *** only works for powers-of-2 ****
    for (int i = PADDED_STATE_COUNT / 2; i > 0; i >>= 1) {
        if (state < i) {
#else
    for (int i = SMALLEST_POWER_OF_TWO / 2; i > 0; i >>= 1) {
        if (state < i && state + i < PADDED_STATE_COUNT ) {
#endif // IS_POWER_OF_TWO
                REAL compare1 = partials[state];
                REAL compare2 = partials[state + i];
                if(compare2 > compare1)
                    partials[state] = compare2;
            }
            KW_LOCAL_FENCE;
        }
        if(state == 0) {
            if( partials[0] > max)
                max = partials[0];
        }
    }

    if(state == 0) {
        if (max == 0) {
        	max = 1.0;
            scalingFactors[pattern] = 0.0;
        } else {
            scalingFactors[pattern] = log(max);
        }
    }


    KW_LOCAL_FENCE;

    for(m = 0; m < matrixCount; m++)
        allPartials[m * patternCount * PADDED_STATE_COUNT + pattern * PADDED_STATE_COUNT +
                    state] /= max;

}

KW_GLOBAL_KERNEL void kernelMultipleNodeSiteReduction(KW_GLOBAL_VAR REAL* dOut,
                                                      KW_GLOBAL_VAR REAL* dIn,
                                                      KW_GLOBAL_VAR REAL* dPatternWeights,
                                                      int outOffset,
                                                      int patternCount) {
#ifdef FW_OPENCL_CPU
    // TODO
#else

    KW_LOCAL_MEM REAL reduce[MULTI_NODE_SUM_BLOCK_SIZE];

    int tx = KW_LOCAL_ID_0;
    int node = KW_GROUP_ID_0;
    int offset = patternCount * node;
    int pattern = tx;

    REAL sum = 0;

    while (pattern < patternCount) {
        FMA(dIn[offset + pattern], dPatternWeights[pattern], sum);
        pattern += MULTI_NODE_SUM_BLOCK_SIZE;
    }

    reduce[tx] = sum;

    KW_LOCAL_FENCE;

    for (unsigned int s = MULTI_NODE_SUM_BLOCK_SIZE / 2; s > 0; s >>= 1) {
        if (tx < s) {
            reduce[tx] += reduce[tx + s];
        }
        KW_LOCAL_FENCE;
    }

    if (tx == 0) {
        dOut[outOffset + node] = reduce[0];
    }
#endif
}

KW_GLOBAL_KERNEL void kernelMultipleNodeSiteSquaredReduction(KW_GLOBAL_VAR REAL* dOut,
                                                             KW_GLOBAL_VAR REAL* dIn,
                                                             KW_GLOBAL_VAR REAL* dPatternWeights,
                                                             int outOffset,
                                                             int patternCount) {
#ifdef FW_OPENCL_CPU
    // TODO
#else

    KW_LOCAL_MEM REAL reduce[MULTI_NODE_SUM_BLOCK_SIZE];

    int tx = KW_LOCAL_ID_0;
    int node = KW_GROUP_ID_0;
    int offset = patternCount * node;
    int pattern = tx;

    REAL sum = 0;

    while (pattern < patternCount) {
        REAL value = dIn[offset + pattern];
        FMA(value * value, dPatternWeights[pattern], sum);
        pattern += MULTI_NODE_SUM_BLOCK_SIZE;
    }

    reduce[tx] = sum;

    KW_LOCAL_FENCE;

    for (unsigned int s = MULTI_NODE_SUM_BLOCK_SIZE / 2; s > 0; s >>= 1) {
        if (tx < s) {
            reduce[tx] += reduce[tx + s];
        }
        KW_LOCAL_FENCE;
    }

    if (tx == 0) {
        dOut[outOffset + node] = reduce[0];
    }
#endif
}

////////////////////////////////////////////////////////////////////////////////////////////////
// scaling experiments kernels

KW_GLOBAL_KERNEL void kernelAccumulateFactorsAutoScaling(KW_GLOBAL_VAR signed char* dScalingFactors,
                                                   KW_GLOBAL_VAR unsigned int* dNodePtrQueue,
                                                   KW_GLOBAL_VAR int* rootScaling,
                                                   int nodeCount,
                                                   int patternCount,
                                                   int scaleBufferSize) {
    int pattern = KW_LOCAL_ID_0 + KW_GROUP_ID_0 * PATTERN_BLOCK_SIZE;
    int index = pattern + KW_GROUP_ID_1 * patternCount;

    int total = 0;
    KW_GLOBAL_VAR signed char* nodeScales;

    int n;
    for(n = 0; n < nodeCount; n++) {
//        int sIndex = dNodePtrQueue[n];
        nodeScales = dScalingFactors + dNodePtrQueue[n] * scaleBufferSize;

        total += nodeScales[index];
    }

    if (pattern < patternCount)
        rootScaling[index] = total;
}

#ifdef CUDA
} // extern "C"
#endif //CUDA
