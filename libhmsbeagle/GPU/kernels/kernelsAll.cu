#include "hip/hip_runtime.h"
/*
 *
 * Copyright 2009 Phylogenetic Likelihood Working Group
 *
 * This file is part of BEAGLE.
 *
 * Use of this source code is governed by an MIT-style
 * license that can be found in the LICENSE file or at
 * https://opensource.org/licenses/MIT.
 *
 * @author Marc Suchard
 * @author Daniel Ayres
 */

#ifdef CUDA
    #include "libhmsbeagle/GPU/GPUImplDefs.h"
    #include <stdlib.h>
    #include <string.h>
    #include <stdio.h>
    #include <math.h>
    extern "C" {
#elif defined(FW_OPENCL)
    #ifdef DOUBLE_PRECISION
        #pragma OPENCL EXTENSION cl_khr_fp64: enable
		#pragma OPENCL EXTENSION cl_khr_int64_base_atomics : enable
    #endif
    #define __umul24(x, y) (x * y)
#endif //FW_OPENCL

#if (!defined DOUBLE_PRECISION && defined FP_FAST_FMAF) || (defined DOUBLE_PRECISION && defined FP_FAST_FMA)
    #define FMA(x, y, z) (z = fma(x, y, z))
#else //FP_FAST_FMA
    #define FMA(x, y, z) (z += x * y)
#endif //FP_FAST_FMA

#if (defined CUDA) && (defined DOUBLE_PRECISION) &&  (__CUDA_ARCH__ < 600)
    __device__ double atomicAdd(double* address, double val)
    {
        unsigned long long int* address_as_ull =
                                  (unsigned long long int*)address;
        unsigned long long int old = *address_as_ull, assumed;

        do {
            assumed = old;
            old = atomicCAS(address_as_ull, assumed,
                            __double_as_longlong(val +
                                   __longlong_as_double(assumed)));

        // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
        } while (assumed != old);

        return __longlong_as_double(old);
    }
#endif

#if (defined FW_OPENCL) && (defined DOUBLE_PRECISION)
    double atomicAdd(__global double* address, double val) {
    __global long* address_as_ull =
        (__global long*)address;
    long old = *address_as_ull;
    long assumed;

    do {
        assumed = old;
        old = atom_cmpxchg(address_as_ull, assumed,
            as_long(val + as_double(assumed)));
    } while (assumed != old);

    return as_double(old);
}

//void atomicAdd(volatile global float* addr, const float val) {
//    private float old, sum;
//    do {
//        old = *addr;
//        sum = old+val;
//    } while(atomic_cmpxchg((volatile global int*)addr, as_int(old), as_int(sum))!=as_int(old));
//}
#endif
///////////////////////////////////////////////////////////////////////////////

KW_GLOBAL_KERNEL void kernelReorderPatterns(      KW_GLOBAL_VAR REAL*             dPartials,
                                                  KW_GLOBAL_VAR int*              dStates,
                                                  KW_GLOBAL_VAR int*              dStatesSort,
                                            const KW_GLOBAL_VAR int*  KW_RESTRICT dTipOffsets,
                                            const KW_GLOBAL_VAR int*  KW_RESTRICT dTipTypes,
                                            const KW_GLOBAL_VAR int*  KW_RESTRICT dPatternsNewOrder,
                                            const KW_GLOBAL_VAR REAL* KW_RESTRICT dPatternWeights,
                                                  KW_GLOBAL_VAR REAL* KW_RESTRICT dPatternWeightsSort,
                                                                int               patternCount,
                                                                int               paddedPatternCount) {
#ifdef FW_OPENCL_CPU
    int state      = 0;
    int pattern    = KW_LOCAL_ID_0 + KW_GROUP_ID_0 * KW_LOCAL_SIZE_0;
#else
    int state      = KW_LOCAL_ID_0;
    int pattern    = KW_LOCAL_ID_1 + KW_GROUP_ID_0 * KW_LOCAL_SIZE_1;
#endif
    int stateCount = PADDED_STATE_COUNT;
    int category   = KW_GROUP_ID_1;
    int tip        = KW_GROUP_ID_2;
    int tipCount   = KW_NUM_GROUPS_2;

    if (pattern < patternCount) {
        int patternSorted  = dPatternsNewOrder[pattern];

        if (dTipTypes[tip] == 0) {
            int categoryOffset = category * stateCount * paddedPatternCount;

            int sortIndex   = categoryOffset + patternSorted * stateCount;
            int originIndex = categoryOffset + pattern       * stateCount;

            const KW_GLOBAL_VAR REAL* KW_RESTRICT partialOriginal = dPartials + dTipOffsets[tip];
                  KW_GLOBAL_VAR REAL* KW_RESTRICT partialSorted   = dPartials + dTipOffsets[tip+tipCount];

#ifdef FW_OPENCL_CPU
            for (int i=0; i < stateCount; i++) {
                partialSorted[sortIndex+i] = partialOriginal[originIndex+i];
            }
#else
            sortIndex += state;
            originIndex += state;
            partialSorted[sortIndex] = partialOriginal[originIndex];
#endif
        } else if (state == 0) {
            const KW_GLOBAL_VAR int* KW_RESTRICT stateOriginal = dStates     + dTipOffsets[tip];
                  KW_GLOBAL_VAR int* KW_RESTRICT stateSorted   = dStatesSort + dTipOffsets[tip+tipCount];

            stateSorted[patternSorted] = stateOriginal[pattern];
        }

        if (state == 0 && category == 0 && tip == 0) {
            dPatternWeightsSort[patternSorted] = dPatternWeights[pattern];
        }
    }
}

KW_GLOBAL_KERNEL void kernelMatrixMulADBMulti(KW_GLOBAL_VAR REAL* dMatrices,
                                              KW_GLOBAL_VAR unsigned int* offsets,
                                              KW_GLOBAL_VAR REAL* Alist,
                                              KW_GLOBAL_VAR REAL* Dlist,
                                              KW_GLOBAL_VAR REAL* Blist,
                                              KW_GLOBAL_VAR REAL* distanceQueue,
                                              int length,
                                              int wB,
                                              int totalMatrix) {

    int wMatrix = KW_GROUP_ID_0 % totalMatrix;
    int offIndex = wMatrix * 3;

    // Block index
    int bx = KW_GROUP_ID_0 / totalMatrix;
    int by = KW_GROUP_ID_1;

    // Thread index
    int tx = KW_LOCAL_ID_0;
    int ty = KW_LOCAL_ID_1;
    int BLOCKS = KW_NUM_GROUPS_1;

    KW_GLOBAL_VAR REAL* C = dMatrices + offsets[offIndex];
    KW_GLOBAL_VAR REAL* B = Blist + offsets[offIndex + 1]; // dEvec
    KW_GLOBAL_VAR REAL* A = Alist + offsets[offIndex + 1]; // dIevc
    KW_GLOBAL_VAR REAL* D = Dlist + offsets[offIndex + 2]; // dEigenValues
    REAL distance = distanceQueue[wMatrix];

    const int EDGE = PADDED_STATE_COUNT - (BLOCKS - 1) * MULTIPLY_BLOCK_SIZE;

    // Step size used to iterate through the sub-matrices of A
    int aStep = MULTIPLY_BLOCK_SIZE;

    // Step size used to iterate through the sub-matrices of B
    int bStep = MULTIPLY_BLOCK_SIZE * PADDED_STATE_COUNT;

    // Csub is used to store the element of the block sub-matrix
    // that is computed by the thread
    REAL Csub = 0;

    int a = PADDED_STATE_COUNT * MULTIPLY_BLOCK_SIZE * by;
    int b = MULTIPLY_BLOCK_SIZE * bx;
    int d = 0; //MULTIPLY_BLOCK_SIZE * bx;

    KW_LOCAL_MEM REAL As[MULTIPLY_BLOCK_SIZE][MULTIPLY_BLOCK_SIZE];
    KW_LOCAL_MEM REAL Bs[MULTIPLY_BLOCK_SIZE][MULTIPLY_BLOCK_SIZE];
    KW_LOCAL_MEM REAL Ds[MULTIPLY_BLOCK_SIZE];

    for (int i = 0; i < BLOCKS - 1; i++) {

        if (ty == 0)
            Ds[tx] = exp(D[d + tx] * distance);

        As[ty][tx] = A[a + PADDED_STATE_COUNT * ty + tx];
        Bs[ty][tx] = B[b + PADDED_STATE_COUNT * ty + tx];

        KW_LOCAL_FENCE;

        for (int k = 0; k < MULTIPLY_BLOCK_SIZE; ++k)
            Csub += As[ty][k] * Ds[k] * Bs[k][tx];

        KW_LOCAL_FENCE;

        a += aStep;
        b += bStep;
        d += MULTIPLY_BLOCK_SIZE;
    }

    // Last block is too long
    if (tx < EDGE && ty < EDGE) {
        if (ty == 0)
            Ds[tx] = exp(D[d + tx] * distance);

        As[ty][tx] = A[a + PADDED_STATE_COUNT * ty + tx];
        Bs[ty][tx] = B[b + PADDED_STATE_COUNT * ty + tx];

    } else {

        if (ty == 0)
            Ds[tx] = 0;

        As[ty][tx] = 0;
        Bs[ty][tx] = 0;
    }

    KW_LOCAL_FENCE;

    for (int k = 0; k < EDGE; k++)
        Csub += As[ty][k] * Ds[k] * Bs[k][tx];

    KW_LOCAL_FENCE;

    // Write the block sub-matrix to device memory;
    // each thread writes one element

    if ((tx < EDGE || bx < BLOCKS - 1) && (ty < EDGE || by < BLOCKS - 1)) { // It's OK to write
        if (Csub < 0)
            C[PADDED_STATE_COUNT* MULTIPLY_BLOCK_SIZE * by + MULTIPLY_BLOCK_SIZE * bx +
              PADDED_STATE_COUNT * ty + tx] = 0;
        else
            C[PADDED_STATE_COUNT* MULTIPLY_BLOCK_SIZE * by + MULTIPLY_BLOCK_SIZE * bx +
              PADDED_STATE_COUNT * ty + tx] = Csub;
    }
}

KW_GLOBAL_KERNEL void kernelMatrixMulADB(KW_GLOBAL_VAR REAL* dMatrices,
                                   KW_GLOBAL_VAR unsigned int* listC,
                                   KW_GLOBAL_VAR REAL* A,
                                   KW_GLOBAL_VAR REAL* D,
                                   KW_GLOBAL_VAR REAL* B,
                                   KW_GLOBAL_VAR REAL* distanceQueue,
                                   int length,
                                   int wB,
                                   int totalMatrix) {

    int wMatrix = KW_GROUP_ID_0 % totalMatrix;

    // Block index
    int bx = KW_GROUP_ID_0 / totalMatrix;
    int by = KW_GROUP_ID_1;

    // Thread index
    int tx = KW_LOCAL_ID_0;
    int ty = KW_LOCAL_ID_1;
    int BLOCKS = KW_NUM_GROUPS_1;

#ifdef CUDA
    KW_LOCAL_MEM REAL* C;
    KW_LOCAL_MEM REAL distance;
    if (tx == 0 && ty == 0) {
        C = dMatrices + listC[wMatrix]; // Non-coalescent read
        distance = distanceQueue[wMatrix]; // Non-coalescent read
    }
#elif defined(FW_OPENCL)
    KW_GLOBAL_VAR REAL* C;
    REAL distance;
    C = dMatrices + listC[wMatrix];
    distance = distanceQueue[wMatrix];
#endif

    KW_LOCAL_FENCE;

    const int EDGE = PADDED_STATE_COUNT - (BLOCKS - 1) * MULTIPLY_BLOCK_SIZE;

    // Step size used to iterate through the sub-matrices of A
    int aStep = MULTIPLY_BLOCK_SIZE;

    // Step size used to iterate through the sub-matrices of B
    int bStep = MULTIPLY_BLOCK_SIZE * PADDED_STATE_COUNT;

    // Csub is used to store the element of the block sub-matrix
    // that is computed by the thread
    REAL Csub = 0;

    int a = PADDED_STATE_COUNT * MULTIPLY_BLOCK_SIZE * by;
    int b = MULTIPLY_BLOCK_SIZE * bx;
    int d = 0; //MULTIPLY_BLOCK_SIZE * bx;

    KW_LOCAL_MEM REAL As[MULTIPLY_BLOCK_SIZE][MULTIPLY_BLOCK_SIZE];
    KW_LOCAL_MEM REAL Bs[MULTIPLY_BLOCK_SIZE][MULTIPLY_BLOCK_SIZE];
    KW_LOCAL_MEM REAL Ds[MULTIPLY_BLOCK_SIZE];

    for (int i = 0; i < BLOCKS - 1; i++) {

        if (ty == 0)
            Ds[tx] = exp(D[d + tx] * distance);

        As[ty][tx] = A[a + PADDED_STATE_COUNT * ty + tx];
        Bs[ty][tx] = B[b + PADDED_STATE_COUNT * ty + tx];

        KW_LOCAL_FENCE;

        for (int k = 0; k < MULTIPLY_BLOCK_SIZE; ++k)
            Csub += As[ty][k] * Ds[k] * Bs[k][tx];

        KW_LOCAL_FENCE;

        a += aStep;
        b += bStep;
        d += MULTIPLY_BLOCK_SIZE;
    }

    // Last block is too long
    if (tx < EDGE && ty < EDGE) {
        if (ty == 0)
            Ds[tx] = exp(D[d + tx] * distance);

        As[ty][tx] = A[a + PADDED_STATE_COUNT * ty + tx];
        Bs[ty][tx] = B[b + PADDED_STATE_COUNT * ty + tx];

    } else {

        if (ty == 0)
            Ds[tx] = 0;

        As[ty][tx] = 0;
        Bs[ty][tx] = 0;
    }

    KW_LOCAL_FENCE;

    for (int k = 0; k < EDGE; k++)
        Csub += As[ty][k] * Ds[k] * Bs[k][tx];

    KW_LOCAL_FENCE;

    // Write the block sub-matrix to device memory;
    // each thread writes one element

    if ((tx < EDGE || bx < BLOCKS - 1) && (ty < EDGE || by < BLOCKS - 1)) { // It's OK to write
        if (Csub < 0)
            C[PADDED_STATE_COUNT* MULTIPLY_BLOCK_SIZE * by + MULTIPLY_BLOCK_SIZE * bx +
              PADDED_STATE_COUNT * ty + tx] = 0;
        else
            C[PADDED_STATE_COUNT* MULTIPLY_BLOCK_SIZE * by + MULTIPLY_BLOCK_SIZE * bx +
              PADDED_STATE_COUNT * ty + tx] = Csub;
    }
}

KW_GLOBAL_KERNEL void kernelMatrixMulADBFirstDeriv(KW_GLOBAL_VAR REAL* dMatrices,
                                           KW_GLOBAL_VAR unsigned int* listC,
                                           KW_GLOBAL_VAR REAL* A,
                                           KW_GLOBAL_VAR REAL* D,
                                           KW_GLOBAL_VAR REAL* B,
                                           KW_GLOBAL_VAR REAL* distanceQueue,
                                           int length,
                                           int wB,
                                           int totalMatrix) {

    int wMatrix = KW_GROUP_ID_0 % totalMatrix;

    // Block index
    int bx = KW_GROUP_ID_0 / totalMatrix;
    int by = KW_GROUP_ID_1;

    // Thread index
    int tx = KW_LOCAL_ID_0;
    int ty = KW_LOCAL_ID_1;
    int BLOCKS = KW_NUM_GROUPS_1;

#ifdef CUDA
    KW_LOCAL_MEM REAL* C;
    KW_LOCAL_MEM REAL* CFirstDeriv;
    KW_LOCAL_MEM REAL distanceLength;
    KW_LOCAL_MEM REAL distanceRate;
    if (tx == 0 && ty == 0) {
        C = dMatrices + listC[wMatrix];
        CFirstDeriv = dMatrices + listC[wMatrix + totalMatrix];
        distanceLength = distanceQueue[wMatrix]; // Non-coalescent read
        distanceRate = distanceQueue[wMatrix + totalMatrix]; // Non-coalescent read
    }
#elif defined(FW_OPENCL)
    KW_GLOBAL_VAR REAL* C;
    KW_GLOBAL_VAR REAL* CFirstDeriv;
    REAL distanceLength;
    REAL distanceRate;
    C = dMatrices + listC[wMatrix];
    CFirstDeriv = dMatrices + listC[wMatrix + totalMatrix];
    distanceLength = distanceQueue[wMatrix];
    distanceRate = distanceQueue[wMatrix + totalMatrix];
#endif

    KW_LOCAL_FENCE;

    const int EDGE = PADDED_STATE_COUNT - (BLOCKS - 1) * MULTIPLY_BLOCK_SIZE;

    // Step size used to iterate through the sub-matrices of A
    int aStep = MULTIPLY_BLOCK_SIZE;

    // Step size used to iterate through the sub-matrices of B
    int bStep = MULTIPLY_BLOCK_SIZE * PADDED_STATE_COUNT;

    // Csub is used to store the element of the block sub-matrix
    // that is computed by the thread
    REAL Csub = 0;
    REAL CFirstDerivSub = 0;

    int a = PADDED_STATE_COUNT * MULTIPLY_BLOCK_SIZE * by;
    int b = MULTIPLY_BLOCK_SIZE * bx;
    int d = 0; //MULTIPLY_BLOCK_SIZE * bx;

    KW_LOCAL_MEM REAL As[MULTIPLY_BLOCK_SIZE][MULTIPLY_BLOCK_SIZE];
    KW_LOCAL_MEM REAL Bs[MULTIPLY_BLOCK_SIZE][MULTIPLY_BLOCK_SIZE];
    KW_LOCAL_MEM REAL Ds[MULTIPLY_BLOCK_SIZE][2];

    for (int i = 0; i < BLOCKS - 1; i++) {

        if (ty == 0) {
            REAL scaledEigenTmp = D[d + tx] * distanceRate;
            Ds[tx][0] = exp(scaledEigenTmp * distanceLength);
            Ds[tx][1] = scaledEigenTmp * Ds[tx][0];
        }

        As[ty][tx] = A[a + PADDED_STATE_COUNT * ty + tx];
        Bs[ty][tx] = B[b + PADDED_STATE_COUNT * ty + tx];

        KW_LOCAL_FENCE;

        for (int k = 0; k < MULTIPLY_BLOCK_SIZE; ++k) {
            Csub += As[ty][k] * Ds[k][0] * Bs[k][tx];
            CFirstDerivSub += As[ty][k] * Ds[k][1] * Bs[k][tx];
        }

        KW_LOCAL_FENCE;

        a += aStep;
        b += bStep;
        d += MULTIPLY_BLOCK_SIZE;
    }

    // Last block is too long
    if (tx < EDGE && ty < EDGE) {
        if (ty == 0) {
            REAL scaledEigenTmp = D[d + tx] * distanceRate;
            Ds[tx][0] = exp(scaledEigenTmp * distanceLength);
            Ds[tx][1] = scaledEigenTmp * Ds[tx][0];
                }

        As[ty][tx] = A[a + PADDED_STATE_COUNT * ty + tx];
        Bs[ty][tx] = B[b + PADDED_STATE_COUNT * ty + tx];

    } else {

        if (ty == 0) {
            Ds[tx][0] = 0;
            Ds[tx][1] = 0;
        }

        As[ty][tx] = 0;
        Bs[ty][tx] = 0;
    }

    KW_LOCAL_FENCE;

    for (int k = 0; k < EDGE; k++) {
        Csub += As[ty][k] * Ds[k][0] * Bs[k][tx];
        CFirstDerivSub += As[ty][k] * Ds[k][1] * Bs[k][tx];
    }

    KW_LOCAL_FENCE;

    // Write the block sub-matrix to device memory;
    // each thread writes one element

    if ((tx < EDGE || bx < BLOCKS - 1) && (ty < EDGE || by < BLOCKS - 1)) { // It's OK to write
        if (Csub < 0)
            C[PADDED_STATE_COUNT* MULTIPLY_BLOCK_SIZE * by + MULTIPLY_BLOCK_SIZE * bx +
              PADDED_STATE_COUNT * ty + tx] = 0;
        else
            C[PADDED_STATE_COUNT* MULTIPLY_BLOCK_SIZE * by + MULTIPLY_BLOCK_SIZE * bx +
              PADDED_STATE_COUNT * ty + tx] = Csub;

        CFirstDeriv[PADDED_STATE_COUNT* MULTIPLY_BLOCK_SIZE * by + MULTIPLY_BLOCK_SIZE * bx +
          PADDED_STATE_COUNT * ty + tx] = CFirstDerivSub;
    }
}

KW_GLOBAL_KERNEL void kernelMatrixMulADBSecondDeriv(KW_GLOBAL_VAR REAL* dMatrices,
                                           KW_GLOBAL_VAR unsigned int* listC,
                                           KW_GLOBAL_VAR REAL* A,
                                           KW_GLOBAL_VAR REAL* D,
                                           KW_GLOBAL_VAR REAL* B,
                                           KW_GLOBAL_VAR REAL* distanceQueue,
                                           int length,
                                           int wB,
                                           int totalMatrix) {

    int wMatrix = KW_GROUP_ID_0 % totalMatrix;

    // Block index
    int bx = KW_GROUP_ID_0 / totalMatrix;
    int by = KW_GROUP_ID_1;

    // Thread index
    int tx = KW_LOCAL_ID_0;
    int ty = KW_LOCAL_ID_1;
    int BLOCKS = KW_NUM_GROUPS_1;

#ifdef CUDA
    KW_LOCAL_MEM REAL* C;
    KW_LOCAL_MEM REAL* CFirstDeriv;
    KW_LOCAL_MEM REAL* CSecondDeriv;
    KW_LOCAL_MEM REAL distanceLength;
    KW_LOCAL_MEM REAL distanceRate;
    if (tx == 0 && ty == 0) {
        C = dMatrices + listC[wMatrix];
        CFirstDeriv = dMatrices + listC[wMatrix + totalMatrix];
        CSecondDeriv = dMatrices + listC[wMatrix + totalMatrix * 2];
        distanceLength = distanceQueue[wMatrix]; // Non-coalescent read
        distanceRate = distanceQueue[wMatrix + totalMatrix]; // Non-coalescent read
    }
#elif defined(FW_OPENCL)
    KW_GLOBAL_VAR REAL* C;
    KW_GLOBAL_VAR REAL* CFirstDeriv;
    KW_GLOBAL_VAR REAL* CSecondDeriv;
    REAL distanceLength;
    REAL distanceRate;
    C = dMatrices + listC[wMatrix];
    CFirstDeriv = dMatrices + listC[wMatrix + totalMatrix];
    CSecondDeriv = dMatrices + listC[wMatrix + totalMatrix * 2];
    distanceLength = distanceQueue[wMatrix];
    distanceRate = distanceQueue[wMatrix + totalMatrix];
#endif

    KW_LOCAL_FENCE;

    const int EDGE = PADDED_STATE_COUNT - (BLOCKS - 1) * MULTIPLY_BLOCK_SIZE;

    // Step size used to iterate through the sub-matrices of A
    int aStep = MULTIPLY_BLOCK_SIZE;

    // Step size used to iterate through the sub-matrices of B
    int bStep = MULTIPLY_BLOCK_SIZE * PADDED_STATE_COUNT;

    // Csub is used to store the element of the block sub-matrix
    // that is computed by the thread
    REAL Csub = 0;
    REAL CFirstDerivSub = 0;
    REAL CSecondDerivSub = 0;

    int a = PADDED_STATE_COUNT * MULTIPLY_BLOCK_SIZE * by;
    int b = MULTIPLY_BLOCK_SIZE * bx;
    int d = 0; //MULTIPLY_BLOCK_SIZE * bx;

    KW_LOCAL_MEM REAL As[MULTIPLY_BLOCK_SIZE][MULTIPLY_BLOCK_SIZE];
    KW_LOCAL_MEM REAL Bs[MULTIPLY_BLOCK_SIZE][MULTIPLY_BLOCK_SIZE];
    KW_LOCAL_MEM REAL Ds[MULTIPLY_BLOCK_SIZE][3];

    for (int i = 0; i < BLOCKS - 1; i++) {

        if (ty == 0) {
            REAL scaledEigenTmp = D[d + tx] * distanceRate;
            Ds[tx][0] = exp(scaledEigenTmp * distanceLength);
            Ds[tx][1] = scaledEigenTmp * Ds[tx][0];
            Ds[tx][2] = scaledEigenTmp * Ds[tx][1];
        }

        As[ty][tx] = A[a + PADDED_STATE_COUNT * ty + tx];
        Bs[ty][tx] = B[b + PADDED_STATE_COUNT * ty + tx];

        KW_LOCAL_FENCE;

        for (int k = 0; k < MULTIPLY_BLOCK_SIZE; ++k) {
            Csub += As[ty][k] * Ds[k][0] * Bs[k][tx];
            CFirstDerivSub += As[ty][k] * Ds[k][1] * Bs[k][tx];
            CSecondDerivSub += As[ty][k] * Ds[k][2] * Bs[k][tx];
        }

        KW_LOCAL_FENCE;

        a += aStep;
        b += bStep;
        d += MULTIPLY_BLOCK_SIZE;
    }

    // Last block is too long
    if (tx < EDGE && ty < EDGE) {
        if (ty == 0) {
            REAL scaledEigenTmp = D[d + tx] * distanceRate;
            Ds[tx][0] = exp(scaledEigenTmp * distanceLength);
            Ds[tx][1] = scaledEigenTmp * Ds[tx][0];
            Ds[tx][2] = scaledEigenTmp * Ds[tx][1];
                }

        As[ty][tx] = A[a + PADDED_STATE_COUNT * ty + tx];
        Bs[ty][tx] = B[b + PADDED_STATE_COUNT * ty + tx];

    } else {

        if (ty == 0) {
            Ds[tx][0] = 0;
            Ds[tx][1] = 0;
            Ds[tx][2] = 0;
        }

        As[ty][tx] = 0;
        Bs[ty][tx] = 0;
    }

    KW_LOCAL_FENCE;

    for (int k = 0; k < EDGE; k++) {
        Csub += As[ty][k] * Ds[k][0] * Bs[k][tx];
        CFirstDerivSub += As[ty][k] * Ds[k][1] * Bs[k][tx];
        CSecondDerivSub += As[ty][k] * Ds[k][2] * Bs[k][tx];
    }

    KW_LOCAL_FENCE;

    // Write the block sub-matrix to device memory;
    // each thread writes one element

    if ((tx < EDGE || bx < BLOCKS - 1) && (ty < EDGE || by < BLOCKS - 1)) { // It's OK to write
        if (Csub < 0)
            C[PADDED_STATE_COUNT* MULTIPLY_BLOCK_SIZE * by + MULTIPLY_BLOCK_SIZE * bx +
              PADDED_STATE_COUNT * ty + tx] = 0;
        else
            C[PADDED_STATE_COUNT* MULTIPLY_BLOCK_SIZE * by + MULTIPLY_BLOCK_SIZE * bx +
              PADDED_STATE_COUNT * ty + tx] = Csub;

        CFirstDeriv[PADDED_STATE_COUNT* MULTIPLY_BLOCK_SIZE * by + MULTIPLY_BLOCK_SIZE * bx +
          PADDED_STATE_COUNT * ty + tx] = CFirstDerivSub;

        CSecondDeriv[PADDED_STATE_COUNT* MULTIPLY_BLOCK_SIZE * by + MULTIPLY_BLOCK_SIZE * bx +
          PADDED_STATE_COUNT * ty + tx] = CSecondDerivSub;
    }
}

KW_GLOBAL_KERNEL void kernelMatrixConvolution(KW_GLOBAL_VAR REAL* dMatrices,
								        KW_GLOBAL_VAR unsigned int* list,
								        int totalMatrixCount
								        ) {

	    int wMatrix = KW_GROUP_ID_0 % totalMatrixCount;

	    // Block index
	    int bx = KW_GROUP_ID_0 / totalMatrixCount;
	    int by = KW_GROUP_ID_1;

	    // Thread index
	    int tx = KW_LOCAL_ID_0;
	    int ty = KW_LOCAL_ID_1;
	    int BLOCKS = KW_NUM_GROUPS_1;


#ifdef CUDA
        KW_LOCAL_MEM REAL* A;
        KW_LOCAL_MEM REAL* B;
        KW_LOCAL_MEM REAL* C;
        if (tx == 0 && ty == 0) {
            A = dMatrices + list[wMatrix]; // Non-coalescent read
            B = dMatrices + list[wMatrix + totalMatrixCount]; // Non-coalescent read
            C = dMatrices + list[wMatrix + totalMatrixCount*2]; // Non-coalescent read
        }
#elif defined(FW_OPENCL)
        KW_GLOBAL_VAR REAL* A;
        KW_GLOBAL_VAR REAL* B;
        KW_GLOBAL_VAR REAL* C;
        A = dMatrices + list[wMatrix];
        B = dMatrices + list[wMatrix + totalMatrixCount];
        C = dMatrices + list[wMatrix + totalMatrixCount*2];
#endif

	    KW_LOCAL_FENCE;

	    const int EDGE = PADDED_STATE_COUNT - (BLOCKS - 1) * MULTIPLY_BLOCK_SIZE;

	    // Step size used to iterate through the sub-matrices of A
	    int aStep = MULTIPLY_BLOCK_SIZE;

	    // Step size used to iterate through the sub-matrices of B
	    int bStep = MULTIPLY_BLOCK_SIZE * PADDED_STATE_COUNT;

	    // Csub is used to store the element of the block sub-matrix
	    // that is computed by the thread
	    REAL Csub = 0;

	    int a = PADDED_STATE_COUNT * MULTIPLY_BLOCK_SIZE * by;
	    int b = MULTIPLY_BLOCK_SIZE * bx;

	    KW_LOCAL_MEM REAL As[MULTIPLY_BLOCK_SIZE][MULTIPLY_BLOCK_SIZE];
	    KW_LOCAL_MEM REAL Bs[MULTIPLY_BLOCK_SIZE][MULTIPLY_BLOCK_SIZE];

	    for (int i = 0; i < BLOCKS - 1; i++) {

	        As[ty][tx] = A[a + PADDED_STATE_COUNT * ty + tx];
	        Bs[ty][tx] = B[b + PADDED_STATE_COUNT * ty + tx];

	        KW_LOCAL_FENCE;

	        for (int k = 0; k < MULTIPLY_BLOCK_SIZE; ++k)
	            Csub += As[ty][k]  * Bs[k][tx];

	        KW_LOCAL_FENCE;

	        a += aStep;
	        b += bStep;
	    }//END: BLOCKS loop

	    // Last block is too long
	    if (tx < EDGE && ty < EDGE) {

	#ifndef KERNEL_PRINT_ENABLED
	        KW_LOCAL_FENCE;
	#endif

	        As[ty][tx] = A[a + PADDED_STATE_COUNT * ty + tx];
	        Bs[ty][tx] = B[b + PADDED_STATE_COUNT * ty + tx];

	    } else {

	        As[ty][tx] = 0;
	        Bs[ty][tx] = 0;

	    }//END: EDGE check

	    KW_LOCAL_FENCE;

	    for (int k = 0; k < EDGE; k++) {
	        Csub += As[ty][k] *  Bs[k][tx];
	    }

	    KW_LOCAL_FENCE;

	    // Write the block sub-matrix to device memory;
	    // each thread writes one element

	    if ((tx < EDGE || bx < BLOCKS - 1) && (ty < EDGE || by < BLOCKS - 1)) { // It's OK to write
	        if (Csub < 0) {

	        	C[PADDED_STATE_COUNT* MULTIPLY_BLOCK_SIZE * by + MULTIPLY_BLOCK_SIZE * bx +
	              PADDED_STATE_COUNT * ty + tx] = 0;

	        } else {

	        	C[PADDED_STATE_COUNT* MULTIPLY_BLOCK_SIZE * by + MULTIPLY_BLOCK_SIZE * bx +
	              PADDED_STATE_COUNT * ty + tx] = Csub;

	        }//END: Csub check
	    }//END: EDGE check

}//END: kernelMatrixConvolution

KW_GLOBAL_KERNEL void kernelMatrixTranspose(KW_GLOBAL_VAR REAL* dMatrices,
                                            KW_GLOBAL_VAR unsigned int* list,
                                            int totalMatrixCount) {

	    int wMatrix = KW_GROUP_ID_0 % totalMatrixCount;

	    // Block index
	    int bx = KW_GROUP_ID_0 / totalMatrixCount;
	    int by = KW_GROUP_ID_1;

	    // Thread index
	    int tx = KW_LOCAL_ID_0;
	    int ty = KW_LOCAL_ID_1;

#ifdef CUDA
        KW_LOCAL_MEM REAL* A;
        KW_LOCAL_MEM REAL* C;
        if (tx == 0 && ty == 0) {
            A = dMatrices + list[wMatrix]; // Non-coalescent read
            C = dMatrices + list[wMatrix + totalMatrixCount]; // Non-coalescent read
        }
#elif defined(FW_OPENCL)
        KW_GLOBAL_VAR REAL* A;
        KW_GLOBAL_VAR REAL* C;
        A = dMatrices + list[wMatrix];
        C = dMatrices + list[wMatrix + totalMatrixCount];
#endif

	    KW_LOCAL_FENCE;

        const int rowOffset = MULTIPLY_BLOCK_SIZE * bx;
        const int colOffset = MULTIPLY_BLOCK_SIZE * by;

        const int row = rowOffset + tx;
        const int col = colOffset + ty;

	    KW_LOCAL_MEM REAL As[MULTIPLY_BLOCK_SIZE][MULTIPLY_BLOCK_SIZE];

	    if (row < PADDED_STATE_COUNT && col < PADDED_STATE_COUNT) {
	        As[ty][tx] = A[PADDED_STATE_COUNT * colOffset + rowOffset +
                           PADDED_STATE_COUNT * ty + tx];
	    }

	    KW_LOCAL_FENCE;

	    if (row < PADDED_STATE_COUNT && col < PADDED_STATE_COUNT) {
		    C[PADDED_STATE_COUNT * rowOffset + colOffset +
		      PADDED_STATE_COUNT * ty + tx] = As[tx][ty];
	    }
}

KW_GLOBAL_KERNEL void kernelMatrixMulADBComplexMulti(KW_GLOBAL_VAR REAL* dMatrices,
                                   KW_GLOBAL_VAR unsigned int* offsets,
                                   KW_GLOBAL_VAR REAL* Alist,
                                   KW_GLOBAL_VAR REAL* Dlist,
                                   KW_GLOBAL_VAR REAL* Blist,
                                   KW_GLOBAL_VAR REAL* distanceQueue,
                                   int length,
                                   int wB,
                                   int totalMatrix) {
#if !(defined(FW_OPENCL_APPLEAMDGPU) && defined(DOUBLE_PRECISION)) // TODO: fix this issue
    int wMatrix = KW_GROUP_ID_0 % totalMatrix;
    int offIndex = wMatrix * 3;

    // Block index
    int bx = KW_GROUP_ID_0 / totalMatrix;
    int by = KW_GROUP_ID_1;
    int BLOCKS = KW_NUM_GROUPS_1;

    // Thread index
    int tx = KW_LOCAL_ID_0;
    int ty = KW_LOCAL_ID_1;

    KW_GLOBAL_VAR REAL* C = dMatrices + offsets[offIndex];
    KW_GLOBAL_VAR REAL* B = Blist + offsets[offIndex + 1]; // dEvec
    KW_GLOBAL_VAR REAL* A = Alist + offsets[offIndex + 1]; // dIevc
    KW_GLOBAL_VAR REAL* D = Dlist + offsets[offIndex + 2]; // dEigenValues
    REAL distance = distanceQueue[wMatrix];

    const int EDGE = PADDED_STATE_COUNT - (BLOCKS - 1) * MULTIPLY_BLOCK_SIZE;

    // Step size used to iterate through the sub-matrices of A
    int aStep = MULTIPLY_BLOCK_SIZE;

    // Step size used to iterate through the sub-matrices of B
    int bStep = MULTIPLY_BLOCK_SIZE * PADDED_STATE_COUNT;

    // Csub is used to store the element of the block sub-matrix
    // that is computed by the thread
    REAL Csub = 0;

    int a = PADDED_STATE_COUNT * MULTIPLY_BLOCK_SIZE * by;
    int b = MULTIPLY_BLOCK_SIZE * bx;
    int d = 0; //MULTIPLY_BLOCK_SIZE * bx;

    KW_LOCAL_MEM REAL As[MULTIPLY_BLOCK_SIZE][MULTIPLY_BLOCK_SIZE];
    KW_LOCAL_MEM REAL Bs[MULTIPLY_BLOCK_SIZE + 2][MULTIPLY_BLOCK_SIZE];
    KW_LOCAL_MEM REAL Cs[MULTIPLY_BLOCK_SIZE];
    KW_LOCAL_MEM REAL Ds[MULTIPLY_BLOCK_SIZE];
    KW_LOCAL_MEM REAL Es[MULTIPLY_BLOCK_SIZE + 2];

#ifdef CUDA
    REAL* B0  = &Bs[1][0];
    REAL* Bm1 = &Bs[0][0];
    REAL* Bp1 = &Bs[2][0];
    REAL* E0  = &Es[1];
#elif defined(FW_OPENCL)
    KW_LOCAL_MEM REAL* B0  = &Bs[1][0];
    KW_LOCAL_MEM REAL* Bm1 = &Bs[0][0];
    KW_LOCAL_MEM REAL* Bp1 = &Bs[2][0];
    KW_LOCAL_MEM REAL* E0  = &Es[1];
#endif

    // Zero first row of Bs and Es
    if (ty == 0) {
        Bs[0][tx] = 0;
        if (tx == 0) {
            Es[0] = 0;
        }
    }

    while (d + MULTIPLY_BLOCK_SIZE < PADDED_STATE_COUNT) {

//      READ_SCHUR_VALUES();
        if (ty == 0) {
            Ds[tx] = exp(D[d + tx] * distance);
            Cs[tx] = D[d + PADDED_STATE_COUNT + tx] * distance;
            if (Cs[tx]) {
                REAL expat = Ds[tx];
                REAL cosbt = cos(Cs[tx]);
#ifdef FW_OPENCL_AMDGPU
                Cs[tx] = -expat * sin(Cs[tx] + 0.0);
#else
                Cs[tx] = -expat * sin(Cs[tx]);
#endif
                Ds[tx] *= cosbt;
            }
        }

        // Block read A and B sub-matrices
        As[ty][tx] = A[a + PADDED_STATE_COUNT * ty + tx];
        B0[ty * MULTIPLY_BLOCK_SIZE + tx] = B[b + PADDED_STATE_COUNT * ty + tx];

        // Read extra row of B for Bp1
        if (ty == 0) {
            B0[MULTIPLY_BLOCK_SIZE * MULTIPLY_BLOCK_SIZE + tx] =
                    B[b + PADDED_STATE_COUNT * MULTIPLY_BLOCK_SIZE + tx];
        }

        // All necessary values loaded
        KW_LOCAL_FENCE;

//      POPULATE_SCHUR_BAND(MULTIPLY_BLOCK_SIZE);
        if (ty == 0 && tx == 0) {
            for(int k=0; k<MULTIPLY_BLOCK_SIZE; k++) {
                if (Cs[k] && !Es[k]) {
                    E0[k] = Cs[k];
                } else {
                    E0[k] = 0;
                }
            }
        }


        KW_LOCAL_FENCE;

//      DO_MULTIPLICATION(MULTIPLY_BLOCK_SIZE);
        for (int k = 0; k < MULTIPLY_BLOCK_SIZE; k++) {
            Csub += As[ty][k] * (
                    Ds[k] * B0 [k * MULTIPLY_BLOCK_SIZE + tx]
                  + E0[k] * Bp1[k * MULTIPLY_BLOCK_SIZE + tx]
                  - Es[k] * Bm1[k * MULTIPLY_BLOCK_SIZE + tx]
            );
        }


        // Move last entries in B0 and E0 to first entries in Bs and Es
        if (ty == 0) {
            Bm1[tx] = Bm1[MULTIPLY_BLOCK_SIZE*MULTIPLY_BLOCK_SIZE + tx];
            if (tx == 0) {
                Es[0] = Es[MULTIPLY_BLOCK_SIZE];
            }
        }

        KW_LOCAL_FENCE;

        // Increment sub-matrices
        a += aStep;
        b += bStep;
        d += MULTIPLY_BLOCK_SIZE;

    }

    if (tx < EDGE && ty < EDGE) { // Last block is too long

//      READ_SCHUR_VALUES();
        if (ty == 0) {
            Ds[tx] = exp(D[d + tx] * distance);
            Cs[tx] = D[d + PADDED_STATE_COUNT + tx] * distance;
            if (Cs[tx]) {
                REAL expat = Ds[tx];
                REAL cosbt = cos(Cs[tx]);
#ifdef FW_OPENCL_AMDGPU
                Cs[tx] = -expat * sin(Cs[tx] + 0.0);
#else
                Cs[tx] = -expat * sin(Cs[tx]);
#endif
                Ds[tx] *= cosbt;
            }
        }

        As[ty][tx] = A[a + PADDED_STATE_COUNT * ty + tx];
        B0[ty * MULTIPLY_BLOCK_SIZE + tx] = B[b + PADDED_STATE_COUNT * ty + tx];

    } else {
        if (ty == 0) {
            Ds[tx] = 0;
            Cs[tx] = 0;
        }
        As[ty][tx] = 0;
        B0[ty * MULTIPLY_BLOCK_SIZE + tx] = 0;
    }

    // Zero last row of Bs and Es (only for unrolled iteration at end)
    if (ty == 0) {
        Bs[MULTIPLY_BLOCK_SIZE+1][tx] = 0;
    }

    // All necessary values loaded
    KW_LOCAL_FENCE;

//  POPULATE_SCHUR_BAND(EDGE);
    if (ty == 0 && tx == 0) {
        for(int k=0; k<EDGE; k++) {
            if (Cs[k] && !Es[k]) {
                E0[k] = Cs[k];
            } else {
                E0[k] = 0;
            }
        }
    }

    KW_LOCAL_FENCE;

    // Do matrix multiplication
//  DO_MULTIPLICATION(EDGE);
    for (int k = 0; k < EDGE; k++) {
        Csub += As[ty][k] * (
                Ds[k] * B0 [k * MULTIPLY_BLOCK_SIZE + tx]
              + E0[k] * Bp1[k * MULTIPLY_BLOCK_SIZE + tx]
              - Es[k] * Bm1[k * MULTIPLY_BLOCK_SIZE + tx]
        );
    }


    KW_LOCAL_FENCE;

    // Write the block sub-matrix to device memory;
    // each thread writes one element

    if (Csub < 0)
        Csub = 0;

    if ((tx < EDGE || bx < BLOCKS - 1) && (ty < EDGE || by < BLOCKS - 1)) { // It's OK to write
        C[PADDED_STATE_COUNT* MULTIPLY_BLOCK_SIZE * by + MULTIPLY_BLOCK_SIZE * bx +
              PADDED_STATE_COUNT * ty + tx] = Csub;
    }
#endif
}


KW_GLOBAL_KERNEL void kernelMatrixMulADBComplex(KW_GLOBAL_VAR REAL* dMatrices,
                                   KW_GLOBAL_VAR unsigned int* listC,
                                   KW_GLOBAL_VAR REAL* A,
                                   KW_GLOBAL_VAR REAL* D,
                                   KW_GLOBAL_VAR REAL* B,
                                   KW_GLOBAL_VAR REAL* distanceQueue,
                                   int length,
                                   int wB,
                                   int totalMatrix) {
#if !(defined(FW_OPENCL_APPLEAMDGPU) && defined(DOUBLE_PRECISION)) // TODO: fix this issue
    int wMatrix = KW_GROUP_ID_0 % totalMatrix;

    // Block index
    int bx = KW_GROUP_ID_0 / totalMatrix;
    int by = KW_GROUP_ID_1;
    int BLOCKS = KW_NUM_GROUPS_1;

    // Thread index
    int tx = KW_LOCAL_ID_0;
    int ty = KW_LOCAL_ID_1;

#ifdef CUDA
    KW_LOCAL_MEM REAL* C;
    KW_LOCAL_MEM REAL distance;
    if (tx == 0 && ty == 0) {
        C = dMatrices + listC[wMatrix];
        distance = distanceQueue[wMatrix]; // Non-coalescent read
    }
#elif defined(FW_OPENCL)
    KW_GLOBAL_VAR REAL* C;
    REAL distance;
    C = dMatrices + listC[wMatrix];
    distance = distanceQueue[wMatrix];
#endif

    KW_LOCAL_FENCE;

    const int EDGE = PADDED_STATE_COUNT - (BLOCKS - 1) * MULTIPLY_BLOCK_SIZE;

    // Step size used to iterate through the sub-matrices of A
    int aStep = MULTIPLY_BLOCK_SIZE;

    // Step size used to iterate through the sub-matrices of B
    int bStep = MULTIPLY_BLOCK_SIZE * PADDED_STATE_COUNT;

    // Csub is used to store the element of the block sub-matrix
    // that is computed by the thread
    REAL Csub = 0;

    int a = PADDED_STATE_COUNT * MULTIPLY_BLOCK_SIZE * by;
    int b = MULTIPLY_BLOCK_SIZE * bx;
    int d = 0; //MULTIPLY_BLOCK_SIZE * bx;

    KW_LOCAL_MEM REAL As[MULTIPLY_BLOCK_SIZE][MULTIPLY_BLOCK_SIZE];
    KW_LOCAL_MEM REAL Bs[MULTIPLY_BLOCK_SIZE + 2][MULTIPLY_BLOCK_SIZE];
    KW_LOCAL_MEM REAL Cs[MULTIPLY_BLOCK_SIZE];
    KW_LOCAL_MEM REAL Ds[MULTIPLY_BLOCK_SIZE];
    KW_LOCAL_MEM REAL Es[MULTIPLY_BLOCK_SIZE + 2];

#ifdef CUDA
   	REAL* B0  = &Bs[1][0];
   	REAL* Bm1 = &Bs[0][0];
   	REAL* Bp1 = &Bs[2][0];
   	REAL* E0  = &Es[1];
#elif defined(FW_OPENCL)
   	KW_LOCAL_MEM REAL* B0  = &Bs[1][0];
   	KW_LOCAL_MEM REAL* Bm1 = &Bs[0][0];
   	KW_LOCAL_MEM REAL* Bp1 = &Bs[2][0];
   	KW_LOCAL_MEM REAL* E0  = &Es[1];
#endif

   	// Zero first row of Bs and Es
   	if (ty == 0) {
   		Bs[0][tx] = 0;
   		if (tx == 0) {
   			Es[0] = 0;
   		}
   	}

    while (d + MULTIPLY_BLOCK_SIZE < PADDED_STATE_COUNT) {

//      READ_SCHUR_VALUES();
		if (ty == 0) {
			Ds[tx] = exp(D[d + tx] * distance);
			Cs[tx] = D[d + PADDED_STATE_COUNT + tx] * distance;
			if (Cs[tx]) {
            	REAL expat = Ds[tx];
            	REAL cosbt = cos(Cs[tx]);
#ifdef FW_OPENCL_AMDGPU
                Cs[tx] = -expat * sin(Cs[tx] + 0.0);
#else
                Cs[tx] = -expat * sin(Cs[tx]);
#endif
            	Ds[tx] *= cosbt;
            }
        }

        // Block read A and B sub-matrices
        As[ty][tx] = A[a + PADDED_STATE_COUNT * ty + tx];
        B0[ty * MULTIPLY_BLOCK_SIZE + tx] = B[b + PADDED_STATE_COUNT * ty + tx];

        // Read extra row of B for Bp1
        if (ty == 0) {
        	B0[MULTIPLY_BLOCK_SIZE * MULTIPLY_BLOCK_SIZE + tx] =
        			B[b + PADDED_STATE_COUNT * MULTIPLY_BLOCK_SIZE + tx];
        }

        // All necessary values loaded
    	KW_LOCAL_FENCE;

//    	POPULATE_SCHUR_BAND(MULTIPLY_BLOCK_SIZE);
		if (ty == 0 && tx == 0) {
			for(int k=0; k<MULTIPLY_BLOCK_SIZE; k++) {
				if (Cs[k] && !Es[k]) {
					E0[k] = Cs[k];
				} else {
					E0[k] = 0;
				}
			}
		}


    	KW_LOCAL_FENCE;

//      DO_MULTIPLICATION(MULTIPLY_BLOCK_SIZE);
		for (int k = 0; k < MULTIPLY_BLOCK_SIZE; k++) {
			Csub += As[ty][k] * (
					Ds[k] * B0 [k * MULTIPLY_BLOCK_SIZE + tx]
				  + E0[k] * Bp1[k * MULTIPLY_BLOCK_SIZE + tx]
				  - Es[k] * Bm1[k * MULTIPLY_BLOCK_SIZE + tx]
			);
		}


        // Move last entries in B0 and E0 to first entries in Bs and Es
        if (ty == 0) {
        	Bm1[tx] = Bm1[MULTIPLY_BLOCK_SIZE*MULTIPLY_BLOCK_SIZE + tx];
        	if (tx == 0) {
        		Es[0] = Es[MULTIPLY_BLOCK_SIZE];
        	}
        }

        KW_LOCAL_FENCE;

        // Increment sub-matrices
        a += aStep;
        b += bStep;
        d += MULTIPLY_BLOCK_SIZE;

    }

    if (tx < EDGE && ty < EDGE) { // Last block is too long

//      READ_SCHUR_VALUES();
		if (ty == 0) {
			Ds[tx] = exp(D[d + tx] * distance);
			Cs[tx] = D[d + PADDED_STATE_COUNT + tx] * distance;
			if (Cs[tx]) {
            	REAL expat = Ds[tx];
            	REAL cosbt = cos(Cs[tx]);
#ifdef FW_OPENCL_AMDGPU
            	Cs[tx] = -expat * sin(Cs[tx] + 0.0);
#else
                Cs[tx] = -expat * sin(Cs[tx]);
#endif
            	Ds[tx] *= cosbt;
            }
        }

        As[ty][tx] = A[a + PADDED_STATE_COUNT * ty + tx];
        B0[ty * MULTIPLY_BLOCK_SIZE + tx] = B[b + PADDED_STATE_COUNT * ty + tx];

    } else {
    	if (ty == 0) {
    		Ds[tx] = 0;
    		Cs[tx] = 0;
    	}
    	As[ty][tx] = 0;
    	B0[ty * MULTIPLY_BLOCK_SIZE + tx] = 0;
    }

	// Zero last row of Bs and Es (only for unrolled iteration at end)
    if (ty == 0) {
    	Bs[MULTIPLY_BLOCK_SIZE+1][tx] = 0;
    }

    // All necessary values loaded
	KW_LOCAL_FENCE;

//	POPULATE_SCHUR_BAND(EDGE);
    if (ty == 0 && tx == 0) {
        for(int k=0; k<EDGE; k++) {
            if (Cs[k] && !Es[k]) {
                E0[k] = Cs[k];
            } else {
                E0[k] = 0;
            }
        }
    }

	KW_LOCAL_FENCE;

	// Do matrix multiplication
//	DO_MULTIPLICATION(EDGE);
    for (int k = 0; k < EDGE; k++) {
        Csub += As[ty][k] * (
                Ds[k] * B0 [k * MULTIPLY_BLOCK_SIZE + tx]
              + E0[k] * Bp1[k * MULTIPLY_BLOCK_SIZE + tx]
              - Es[k] * Bm1[k * MULTIPLY_BLOCK_SIZE + tx]
        );
    }


    KW_LOCAL_FENCE;

    // Write the block sub-matrix to device memory;
    // each thread writes one element

    if (Csub < 0)
    	Csub = 0;

    if ((tx < EDGE || bx < BLOCKS - 1) && (ty < EDGE || by < BLOCKS - 1)) { // It's OK to write
        C[PADDED_STATE_COUNT* MULTIPLY_BLOCK_SIZE * by + MULTIPLY_BLOCK_SIZE * bx +
              PADDED_STATE_COUNT * ty + tx] = Csub;
    }
#endif
}

KW_GLOBAL_KERNEL void kernelSumSites1(KW_GLOBAL_VAR REAL* dArray,
                                      KW_GLOBAL_VAR REAL* dSum,
                                      KW_GLOBAL_VAR REAL* dPatternWeights,
                                      int patternCount) {
#ifdef FW_OPENCL_CPU

    REAL sum = 0;

    int pattern = KW_GROUP_ID_0 * SUM_SITES_BLOCK_SIZE;
    int maxPattern = (KW_GROUP_ID_0 + 1) * SUM_SITES_BLOCK_SIZE;

    if (maxPattern > patternCount)
        maxPattern = patternCount;

    while (pattern < maxPattern) {
        FMA(dArray[pattern],  dPatternWeights[pattern], sum);
        pattern++;
    }

    dSum[KW_GROUP_ID_0] = sum;

#else

    KW_LOCAL_MEM REAL sum[SUM_SITES_BLOCK_SIZE];

    int tx = KW_LOCAL_ID_0;
    int pattern = KW_LOCAL_ID_0 + KW_GROUP_ID_0 * SUM_SITES_BLOCK_SIZE;

    if (pattern < patternCount)
        sum[tx] = dArray[pattern] * dPatternWeights[pattern];
    else
        sum[tx] = 0.0;

    KW_LOCAL_FENCE;

    for (unsigned int s = SUM_SITES_BLOCK_SIZE / 2; s > 0; s >>= 1) {
        if (tx < s)
            sum[tx] += sum[tx + s];
        KW_LOCAL_FENCE;
    }

    if (tx == 0)
        dSum[KW_GROUP_ID_0] = sum[0];

#endif
}

KW_GLOBAL_KERNEL void kernelSumSites1Partition(KW_GLOBAL_VAR REAL* dArray,
                                               KW_GLOBAL_VAR REAL* dSum,
                                               KW_GLOBAL_VAR REAL* dPatternWeights,
                                               int startPattern,
                                               int endPattern) {
#ifdef FW_OPENCL_CPU

    REAL sum = 0;

    int pattern = startPattern + KW_GROUP_ID_0 * SUM_SITES_BLOCK_SIZE;
    int maxPattern = startPattern + (KW_GROUP_ID_0 + 1) * SUM_SITES_BLOCK_SIZE;

    if (maxPattern > endPattern)
        maxPattern = endPattern;

    while (pattern < maxPattern) {
        FMA(dArray[pattern],  dPatternWeights[pattern], sum);
        pattern++;
    }

    dSum[KW_GROUP_ID_0] = sum;

#else

    KW_LOCAL_MEM REAL sum[SUM_SITES_BLOCK_SIZE];

    int tx = KW_LOCAL_ID_0;
    int pattern = startPattern + KW_LOCAL_ID_0 + KW_GROUP_ID_0 * SUM_SITES_BLOCK_SIZE;

    if (pattern < endPattern)
        sum[tx] = dArray[pattern] * dPatternWeights[pattern];
    else
        sum[tx] = 0.0;

    KW_LOCAL_FENCE;

    for (unsigned int s = SUM_SITES_BLOCK_SIZE / 2; s > 0; s >>= 1) {
        if (tx < s)
            sum[tx] += sum[tx + s];
        KW_LOCAL_FENCE;
    }

    if (tx == 0)
        dSum[KW_GROUP_ID_0] = sum[0];

#endif
}

// KW_GLOBAL_KERNEL void kernelSumSites1Partition(KW_GLOBAL_VAR REAL*         dArray,
//                                                KW_GLOBAL_VAR REAL*         dSum,
//                                                KW_GLOBAL_VAR REAL*         dPatternWeights,
//                                                KW_GLOBAL_VAR unsigned int* dPtrOffsets) {

//     int opIndexPtr = KW_GROUP_ID_0 * 2;
//     int startPattern = dPtrOffsets[opIndexPtr    ];
//     int endPattern   = dPtrOffsets[opIndexPtr + 1];

// #ifdef FW_OPENCL_CPU

//     REAL sum = 0;

//     int pattern = startPattern + KW_GROUP_ID_0 * SUM_SITES_BLOCK_SIZE;

//     while (pattern < endPattern) {
//         FMA(dArray[pattern],  dPatternWeights[pattern], sum);
//         pattern++;
//     }

//     dSum[KW_GROUP_ID_0] = sum;

// #else

//     KW_LOCAL_MEM REAL sum[SUM_SITES_BLOCK_SIZE];

//     int tx = KW_LOCAL_ID_0;
//     int pattern = startPattern + KW_LOCAL_ID_0 + KW_GROUP_ID_0 * SUM_SITES_BLOCK_SIZE;

//     if (pattern < endPattern)
//         sum[tx] = dArray[pattern] * dPatternWeights[pattern];
//     else
//         sum[tx] = 0.0;

//     KW_LOCAL_FENCE;

//     for (unsigned int s = SUM_SITES_BLOCK_SIZE / 2; s > 0; s >>= 1) {
//         if (tx < s)
//             sum[tx] += sum[tx + s];
//         KW_LOCAL_FENCE;
//     }

//     if (tx == 0)
//         dSum[KW_GROUP_ID_0] = sum[0];

// #endif
// }

KW_GLOBAL_KERNEL void kernelSumSites2(KW_GLOBAL_VAR REAL* dArray1,
                                      KW_GLOBAL_VAR REAL* dSum1,
                                      KW_GLOBAL_VAR REAL* dArray2,
                                      KW_GLOBAL_VAR REAL* dSum2,
                                      KW_GLOBAL_VAR REAL* dPatternWeights,
                                      int patternCount) {

#ifdef FW_OPENCL_CPU

    REAL sum1 = 0, sum2 = 0;

    int pattern = KW_GROUP_ID_0 * SUM_SITES_BLOCK_SIZE;
    int maxPattern = (KW_GROUP_ID_0 + 1) * SUM_SITES_BLOCK_SIZE;

    if (maxPattern > patternCount)
        maxPattern = patternCount;

    while (pattern < maxPattern) {
        FMA(dArray1[pattern],  dPatternWeights[pattern], sum1);
        FMA(dArray2[pattern],  dPatternWeights[pattern], sum2);
        pattern++;
    }

    dSum1[KW_GROUP_ID_0] = sum1;
    dSum2[KW_GROUP_ID_0] = sum2;

#else

    KW_LOCAL_MEM REAL sum1[SUM_SITES_BLOCK_SIZE];
    KW_LOCAL_MEM REAL sum2[SUM_SITES_BLOCK_SIZE];

    int tx = KW_LOCAL_ID_0;
    int pattern = KW_LOCAL_ID_0 + KW_GROUP_ID_0 * SUM_SITES_BLOCK_SIZE;

    if (pattern < patternCount) {
        REAL pWeight = dPatternWeights[pattern];
        sum1[tx] = dArray1[pattern] * pWeight;
        sum2[tx] = dArray2[pattern] * pWeight;
    } else {
        sum1[tx] = 0.0;
        sum2[tx] = 0.0;
    }

    KW_LOCAL_FENCE;

    for (unsigned int s = SUM_SITES_BLOCK_SIZE / 2; s > 0; s >>= 1) {
        if (tx < s) {
            sum1[tx] += sum1[tx + s];
            sum2[tx] += sum2[tx + s];
        }
        KW_LOCAL_FENCE;
    }

    if (tx == 0) {
        dSum1[KW_GROUP_ID_0] = sum1[0];
        dSum2[KW_GROUP_ID_0] = sum2[0];
    }

#endif
}

KW_GLOBAL_KERNEL void kernelSumSites3(KW_GLOBAL_VAR REAL* dArray1,
                                      KW_GLOBAL_VAR REAL* dSum1,
                                      KW_GLOBAL_VAR REAL* dArray2,
                                      KW_GLOBAL_VAR REAL* dSum2,
                                      KW_GLOBAL_VAR REAL* dArray3,
                                      KW_GLOBAL_VAR REAL* dSum3,
                                      KW_GLOBAL_VAR REAL* dPatternWeights,
                                      int patternCount) {

#ifdef FW_OPENCL_CPU

    REAL sum1 = 0, sum2 = 0, sum3 = 0;

    int pattern = KW_GROUP_ID_0 * SUM_SITES_BLOCK_SIZE;
    int maxPattern = (KW_GROUP_ID_0 + 1) * SUM_SITES_BLOCK_SIZE;

    if (maxPattern > patternCount)
        maxPattern = patternCount;

    while (pattern < maxPattern) {
        FMA(dArray1[pattern],  dPatternWeights[pattern], sum1);
        FMA(dArray2[pattern],  dPatternWeights[pattern], sum2);
        FMA(dArray3[pattern],  dPatternWeights[pattern], sum3);

        pattern++;
    }

    dSum1[KW_GROUP_ID_0] = sum1;
    dSum2[KW_GROUP_ID_0] = sum2;
    dSum3[KW_GROUP_ID_0] = sum3;

#else

    KW_LOCAL_MEM REAL sum1[SUM_SITES_BLOCK_SIZE];
    KW_LOCAL_MEM REAL sum2[SUM_SITES_BLOCK_SIZE];
    KW_LOCAL_MEM REAL sum3[SUM_SITES_BLOCK_SIZE];

    int tx = KW_LOCAL_ID_0;
    int pattern = KW_LOCAL_ID_0 + KW_GROUP_ID_0 * SUM_SITES_BLOCK_SIZE;

    if (pattern < patternCount) {
        REAL pWeight = dPatternWeights[pattern];
        sum1[tx] = dArray1[pattern] * pWeight;
        sum2[tx] = dArray2[pattern] * pWeight;
        sum3[tx] = dArray3[pattern] * pWeight;
    } else {
        sum1[tx] = 0.0;
        sum2[tx] = 0.0;
        sum3[tx] = 0.0;
    }

    KW_LOCAL_FENCE;

    for (unsigned int s = SUM_SITES_BLOCK_SIZE / 2; s > 0; s >>= 1) {
        if (tx < s) {
            sum1[tx] += sum1[tx + s];
            sum2[tx] += sum2[tx + s];
            sum3[tx] += sum3[tx + s];
        }
        KW_LOCAL_FENCE;
    }

    if (tx == 0) {
        dSum1[KW_GROUP_ID_0] = sum1[0];
        dSum2[KW_GROUP_ID_0] = sum2[0];
        dSum3[KW_GROUP_ID_0] = sum3[0];
    }

#endif
}

KW_GLOBAL_KERNEL void kernelAccumulateFactors(KW_GLOBAL_VAR REAL* dScalingFactors,
                                              KW_GLOBAL_VAR unsigned int* dNodePtrQueue,
                                              KW_GLOBAL_VAR REAL* rootScaling,
                                              int nodeCount,
                                              int patternCount) {

    int pattern = KW_LOCAL_ID_0 + KW_GROUP_ID_0 * PATTERN_BLOCK_SIZE;

    REAL total = 0;
    KW_GLOBAL_VAR REAL* nodeScales;

    int n;
    for(n = 0; n < nodeCount; n++) {
//      if (KW_LOCAL_ID_0 == 0) // TODO Why does this not work???
        nodeScales = dScalingFactors + dNodePtrQueue[n];
//      KW_LOCAL_FENCE;

    #ifdef KERNEL_PRINT_ENABLED
        if (pattern == 1)
            printf("added %1.2e\n", nodeScales[pattern]);
    #endif
        REAL factor = nodeScales[pattern];
        if (factor != 1.0) {
            total += log(factor);
        }
    }

#ifdef FW_OPENCL_CPU // CPU/MIC implementation
    rootScaling[pattern] += total;
#else // GPU implementation
    if (pattern < patternCount)
        rootScaling[pattern] += total;
#endif // FW_OPENCL_CPU
}

KW_GLOBAL_KERNEL void kernelAccumulateFactorsByPartition(KW_GLOBAL_VAR REAL* dScalingFactors,
                                                         KW_GLOBAL_VAR unsigned int* dNodePtrQueue,
                                                         KW_GLOBAL_VAR REAL* rootScaling,
                                                         int nodeCount,
                                                         int startPattern,
                                                         int endPattern) {

    int pattern = startPattern + KW_LOCAL_ID_0 + KW_GROUP_ID_0 * PATTERN_BLOCK_SIZE;

    REAL total = 0;
    KW_GLOBAL_VAR REAL* nodeScales;

    int n;
    for(n = 0; n < nodeCount; n++) {
        nodeScales = dScalingFactors + dNodePtrQueue[n];

        REAL factor = nodeScales[pattern];
        if (factor != 1.0) {
            total += log(factor);
        }
    }

    if (pattern < endPattern) {
        rootScaling[pattern] += total;
    }
}

KW_GLOBAL_KERNEL void kernelAccumulateFactorsScalersLog(KW_GLOBAL_VAR REAL* dScalingFactors,
                                                 KW_GLOBAL_VAR unsigned int* dNodePtrQueue,
                                                 KW_GLOBAL_VAR REAL* rootScaling,
                                                 int nodeCount,
                                                 int patternCount) {
    int pattern = KW_LOCAL_ID_0 + KW_GROUP_ID_0 * PATTERN_BLOCK_SIZE;

    REAL total = 0;
    KW_GLOBAL_VAR REAL* nodeScales;

    int n;
    for(n = 0; n < nodeCount; n++) {
//      if (KW_LOCAL_ID_0 == 0) // TODO Why does this not work???
        nodeScales = dScalingFactors + dNodePtrQueue[n];
//      KW_LOCAL_FENCE;

#ifdef KERNEL_PRINT_ENABLED
        if (pattern == 1)
            printf("added %1.2e\n", nodeScales[pattern]);
#endif
        total += nodeScales[pattern];
    }

#ifdef FW_OPENCL_CPU // CPU/MIC implementation
    rootScaling[pattern] += total;
#else // GPU implementation
    if (pattern < patternCount)
        rootScaling[pattern] += total;
#endif // FW_OPENCL_CPU
}

KW_GLOBAL_KERNEL void kernelAccumulateFactorsScalersLogByPartition(
                                                KW_GLOBAL_VAR REAL* dScalingFactors,
                                                KW_GLOBAL_VAR unsigned int* dNodePtrQueue,
                                                KW_GLOBAL_VAR REAL* rootScaling,
                                                int nodeCount,
                                                int startPattern,
                                                int endPattern) {

    int pattern = startPattern + KW_LOCAL_ID_0 + KW_GROUP_ID_0 * PATTERN_BLOCK_SIZE;

    REAL total = 0;
    KW_GLOBAL_VAR REAL* nodeScales;

    int n;
    for(n = 0; n < nodeCount; n++) {
        nodeScales = dScalingFactors + dNodePtrQueue[n];

        total += nodeScales[pattern];
    }

    if (pattern < endPattern) {
        rootScaling[pattern] += total;
    }
}

KW_GLOBAL_KERNEL void kernelRemoveFactors(KW_GLOBAL_VAR REAL* dScalingFactors,
                                    KW_GLOBAL_VAR unsigned int* dNodePtrQueue,
                                                   KW_GLOBAL_VAR REAL* rootScaling,
                                                   int nodeCount,
                                                   int patternCount) {
    int pattern = KW_LOCAL_ID_0 + KW_GROUP_ID_0 * PATTERN_BLOCK_SIZE;

    REAL total = 0;
    KW_GLOBAL_VAR REAL* nodeScales;

    int n;
    for(n = 0; n < nodeCount; n++) {
//      if (KW_LOCAL_ID_0 == 0) // TODO Why does this not work???
        nodeScales = dScalingFactors + dNodePtrQueue[n];
//      KW_LOCAL_FENCE;

#ifdef KERNEL_PRINT_ENABLED
        if (pattern == 1)
            printf("added %1.2e\n", nodeScales[pattern]);
#endif
        REAL factor = nodeScales[pattern];
        if (factor != 1.0) {
            total += log(factor);
        }
    }

#ifdef FW_OPENCL_CPU // CPU/MIC implementation
    rootScaling[pattern] -= total;
#else // GPU implementation
    if (pattern < patternCount)
        rootScaling[pattern] -= total;
#endif // FW_OPENCL_CPU
}

KW_GLOBAL_KERNEL void kernelRemoveFactorsByPartition(KW_GLOBAL_VAR REAL* dScalingFactors,
                                                     KW_GLOBAL_VAR unsigned int* dNodePtrQueue,
                                                     KW_GLOBAL_VAR REAL* rootScaling,
                                                     int nodeCount,
                                                     int startPattern,
                                                     int endPattern) {
    int pattern = startPattern + KW_LOCAL_ID_0 + KW_GROUP_ID_0 * PATTERN_BLOCK_SIZE;

    REAL total = 0;
    KW_GLOBAL_VAR REAL* nodeScales;

    int n;
    for(n = 0; n < nodeCount; n++) {
        nodeScales = dScalingFactors + dNodePtrQueue[n];

        REAL factor = nodeScales[pattern];
        if (factor != 1.0) {
            total += log(factor);
        }
    }

    if (pattern < endPattern) {
        rootScaling[pattern] -= total;
    }
}

KW_GLOBAL_KERNEL void kernelRemoveFactorsScalersLog(KW_GLOBAL_VAR REAL* dScalingFactors,
                                             KW_GLOBAL_VAR unsigned int* dNodePtrQueue,
                                             KW_GLOBAL_VAR REAL* rootScaling,
                                             int nodeCount,
                                             int patternCount) {
    int pattern = KW_LOCAL_ID_0 + KW_GROUP_ID_0 * PATTERN_BLOCK_SIZE;

    REAL total = 0;
    KW_GLOBAL_VAR REAL* nodeScales;

    int n;
    for(n = 0; n < nodeCount; n++) {
//      if (KW_LOCAL_ID_0 == 0) // TODO Why does this not work???
        nodeScales = dScalingFactors + dNodePtrQueue[n];
//      KW_LOCAL_FENCE;

#ifdef KERNEL_PRINT_ENABLED
        if (pattern == 1)
            printf("added %1.2e\n", nodeScales[pattern]);
#endif

        total += nodeScales[pattern];
    }

#ifdef FW_OPENCL_CPU // CPU/MIC implementation
    rootScaling[pattern] -= total;
#else // GPU implementation
    if (pattern < patternCount)
        rootScaling[pattern] -= total;
#endif // FW_OPENCL_CPU
}

KW_GLOBAL_KERNEL void kernelRemoveFactorsScalersLogByPartition(KW_GLOBAL_VAR REAL* dScalingFactors,
                                                               KW_GLOBAL_VAR unsigned int* dNodePtrQueue,
                                                               KW_GLOBAL_VAR REAL* rootScaling,
                                                               int nodeCount,
                                                               int startPattern,
                                                               int endPattern) {
    int pattern = startPattern + KW_LOCAL_ID_0 + KW_GROUP_ID_0 * PATTERN_BLOCK_SIZE;

    REAL total = 0;
    KW_GLOBAL_VAR REAL* nodeScales;

    int n;
    for(n = 0; n < nodeCount; n++) {
        nodeScales = dScalingFactors + dNodePtrQueue[n];

        total += nodeScales[pattern];
    }

    if (pattern < endPattern)
        rootScaling[pattern] -= total;

}

KW_GLOBAL_KERNEL void kernelResetFactorsByPartition(KW_GLOBAL_VAR REAL* dScalingFactors,
                                                    int startPattern,
                                                    int endPattern) {
    int pattern = startPattern + KW_LOCAL_ID_0 + KW_GROUP_ID_0 * PATTERN_BLOCK_SIZE;

    if (pattern < endPattern) {
        dScalingFactors[pattern] = 0.0;
    }
}


KW_GLOBAL_KERNEL void kernelPartialsDynamicScalingSlow(KW_GLOBAL_VAR REAL* allPartials,
                                                 KW_GLOBAL_VAR REAL* scalingFactors,
                                                 int matrixCount) {
    int state = KW_LOCAL_ID_0;
    int pattern = KW_GROUP_ID_0;
    int patternCount = KW_NUM_GROUPS_0;

    KW_LOCAL_MEM REAL partials[PADDED_STATE_COUNT];

    KW_LOCAL_MEM REAL max;

    if (state == 0)
        max = 0.0;

    int m;
    for(m = 0; m < matrixCount; m++) {
        partials[state] = allPartials[m * patternCount * PADDED_STATE_COUNT + pattern *
                                      PADDED_STATE_COUNT + state];
        KW_LOCAL_FENCE;

#ifdef IS_POWER_OF_TWO
    // parallelized reduction *** only works for powers-of-2 ****
    for (int i = PADDED_STATE_COUNT / 2; i > 0; i >>= 1) {
        if (state < i) {
#else
    for (int i = SMALLEST_POWER_OF_TWO / 2; i > 0; i >>= 1) {
        if (state < i && state + i < PADDED_STATE_COUNT ) {
#endif // IS_POWER_OF_TWO
                REAL compare1 = partials[state];
                REAL compare2 = partials[state + i];
                if(compare2 > compare1)
                    partials[state] = compare2;
            }
            KW_LOCAL_FENCE;
        }
        if(state == 0) {
            if( partials[0] > max)
                max = partials[0];
        }
    }

    if(state == 0) {
        if (max == 0)
        	max = 1.0;
        scalingFactors[pattern] = max;
    }


    KW_LOCAL_FENCE;

    for(m = 0; m < matrixCount; m++)
        allPartials[m * patternCount * PADDED_STATE_COUNT + pattern * PADDED_STATE_COUNT +
                    state] /= max;

}

KW_GLOBAL_KERNEL void kernelPartialsDynamicScalingSlowScalersLog(KW_GLOBAL_VAR REAL* allPartials,
                                                          KW_GLOBAL_VAR REAL* scalingFactors,
                                                          int matrixCount) {
    int state = KW_LOCAL_ID_0;
    int pattern = KW_GROUP_ID_0;
    int patternCount = KW_NUM_GROUPS_0;

    KW_LOCAL_MEM REAL partials[PADDED_STATE_COUNT];

    KW_LOCAL_MEM REAL max;

    if (state == 0)
        max = 0.0;

    int m;
    for(m = 0; m < matrixCount; m++) {
        partials[state] = allPartials[m * patternCount * PADDED_STATE_COUNT + pattern *
                                      PADDED_STATE_COUNT + state];
        KW_LOCAL_FENCE;

#ifdef IS_POWER_OF_TWO
    // parallelized reduction *** only works for powers-of-2 ****
    for (int i = PADDED_STATE_COUNT / 2; i > 0; i >>= 1) {
        if (state < i) {
#else
    for (int i = SMALLEST_POWER_OF_TWO / 2; i > 0; i >>= 1) {
        if (state < i && state + i < PADDED_STATE_COUNT ) {
#endif // IS_POWER_OF_TWO
                REAL compare1 = partials[state];
                REAL compare2 = partials[state + i];
                if(compare2 > compare1)
                    partials[state] = compare2;
            }
            KW_LOCAL_FENCE;
        }
        if(state == 0) {
            if( partials[0] > max)
                max = partials[0];
        }
    }

    if(state == 0) {
        if (max == 0) {
        	max = 1.0;
            scalingFactors[pattern] = 0.0;
        } else {
            scalingFactors[pattern] = log(max);
        }
    }


    KW_LOCAL_FENCE;

    for(m = 0; m < matrixCount; m++)
        allPartials[m * patternCount * PADDED_STATE_COUNT + pattern * PADDED_STATE_COUNT +
                    state] /= max;

}

KW_GLOBAL_KERNEL void kernelMultipleNodeSiteReduction(KW_GLOBAL_VAR REAL* dOut,
                                                      KW_GLOBAL_VAR REAL* dIn,
                                                      KW_GLOBAL_VAR REAL* dPatternWeights,
                                                      int outOffset,
                                                      int patternCount) {
#ifdef FW_OPENCL_CPU
    // TODO
#else

    KW_LOCAL_MEM REAL reduce[MULTI_NODE_SUM_BLOCK_SIZE];

    int tx = KW_LOCAL_ID_0;
    int node = KW_GROUP_ID_0;
    int offset = patternCount * node;
    int pattern = tx;

    REAL sum = 0;

    while (pattern < patternCount) {
        FMA(dIn[offset + pattern], dPatternWeights[pattern], sum);
        pattern += MULTI_NODE_SUM_BLOCK_SIZE;
    }

    reduce[tx] = sum;

    KW_LOCAL_FENCE;

    for (unsigned int s = MULTI_NODE_SUM_BLOCK_SIZE / 2; s > 0; s >>= 1) {
        if (tx < s) {
            reduce[tx] += reduce[tx + s];
        }
        KW_LOCAL_FENCE;
    }

    if (tx == 0) {
        dOut[outOffset + node] = reduce[0];
    }
#endif
}

KW_GLOBAL_KERNEL void kernelMultipleNodeSiteSquaredReduction(KW_GLOBAL_VAR REAL* dOut,
                                                             KW_GLOBAL_VAR REAL* dIn,
                                                             KW_GLOBAL_VAR REAL* dPatternWeights,
                                                             int outOffset,
                                                             int patternCount) {
#ifdef FW_OPENCL_CPU
    // TODO
#else

    KW_LOCAL_MEM REAL reduce[MULTI_NODE_SUM_BLOCK_SIZE];

    int tx = KW_LOCAL_ID_0;
    int node = KW_GROUP_ID_0;
    int offset = patternCount * node;
    int pattern = tx;

    REAL sum = 0;

    while (pattern < patternCount) {
        REAL value = dIn[offset + pattern];
        FMA(value * value, dPatternWeights[pattern], sum);
        pattern += MULTI_NODE_SUM_BLOCK_SIZE;
    }

    reduce[tx] = sum;

    KW_LOCAL_FENCE;

    for (unsigned int s = MULTI_NODE_SUM_BLOCK_SIZE / 2; s > 0; s >>= 1) {
        if (tx < s) {
            reduce[tx] += reduce[tx + s];
        }
        KW_LOCAL_FENCE;
    }

    if (tx == 0) {
        dOut[outOffset + node] = reduce[0];
    }
#endif
}

////////////////////////////////////////////////////////////////////////////////////////////////
// scaling experiments kernels

KW_GLOBAL_KERNEL void kernelAccumulateFactorsAutoScaling(KW_GLOBAL_VAR signed char* dScalingFactors,
                                                   KW_GLOBAL_VAR unsigned int* dNodePtrQueue,
                                                   KW_GLOBAL_VAR int* rootScaling,
                                                   int nodeCount,
                                                   int patternCount,
                                                   int scaleBufferSize) {
    int pattern = KW_LOCAL_ID_0 + KW_GROUP_ID_0 * PATTERN_BLOCK_SIZE;
    int index = pattern + KW_GROUP_ID_1 * patternCount;

    int total = 0;
    KW_GLOBAL_VAR signed char* nodeScales;

    int n;
    for(n = 0; n < nodeCount; n++) {
//        int sIndex = dNodePtrQueue[n];
        nodeScales = dScalingFactors + dNodePtrQueue[n] * scaleBufferSize;

        total += nodeScales[index];
    }

    if (pattern < patternCount)
        rootScaling[index] = total;
}


/*
 * BASTA kernels
 */

KW_GLOBAL_KERNEL void kernelInnerBastaPartialsCoalescent(KW_GLOBAL_VAR REAL* KW_RESTRICT partials,
                                                    KW_GLOBAL_VAR REAL* KW_RESTRICT matrices,
                                                    KW_GLOBAL_VAR int* KW_RESTRICT operations,
                                                    KW_GLOBAL_VAR REAL* KW_RESTRICT sizes,
                                                    KW_GLOBAL_VAR REAL* KW_RESTRICT coalescent,
													int start,
                                                    int numOps,
                                                    int totalPatterns) {

    int state = KW_LOCAL_ID_0;
    int patIdx = KW_LOCAL_ID_1;
    int pattern = __umul24(KW_GROUP_ID_0,BASTA_SUM_ACROSS_BLOCK_SIZE) + patIdx;
    int op = pattern + start;
    int maxOp = start + totalPatterns - 1;
    int sameTransIndex = 1;
    KW_LOCAL_MEM REAL sMatrix1[BLOCK_PEELING_SIZE_SCA][PADDED_STATE_COUNT];
    KW_LOCAL_MEM REAL sMatrix2[BLOCK_PEELING_SIZE_SCA][PADDED_STATE_COUNT];
    KW_LOCAL_MEM REAL sPartials1[BASTA_SUM_ACROSS_BLOCK_SIZE][PADDED_STATE_COUNT];
	KW_LOCAL_MEM REAL sPartials2[1][PADDED_STATE_COUNT];
    //    KW_LOCAL_MEM REAL sPartials2[PADDED_STATE_COUNT];
    KW_LOCAL_MEM REAL popSizes[PADDED_STATE_COUNT];

    int desIndex = operations[op * numOps];
    int child1PartialIndex = operations[op * numOps + 1];
    int child1TransIndex = operations[maxOp * numOps + 2];
    int child2PartialIndex = operations[op * numOps + 3];
    int child2TransIndex = operations[maxOp * numOps + 4];
    int accumulation1PartialIndex = operations[op * numOps + 5];
    int accumulation2PartialIndex = operations[op * numOps + 6];
    int intervalNumber = operations[op * numOps + 7];;

    KW_GLOBAL_VAR REAL* KW_RESTRICT partials1 = partials + child1PartialIndex;
    KW_GLOBAL_VAR REAL* KW_RESTRICT partials2 = partials + child2PartialIndex;
    KW_GLOBAL_VAR REAL* KW_RESTRICT partials3 = partials + desIndex;
	KW_GLOBAL_VAR REAL* KW_RESTRICT accumulation1 = partials + accumulation1PartialIndex;
	KW_GLOBAL_VAR REAL* KW_RESTRICT accumulation2 = partials + accumulation2PartialIndex;


    if (pattern < totalPatterns) {
        sPartials1[patIdx][state] = partials1[state];
    } else {
        sPartials1[patIdx][state] = 0;
    }
    REAL sum1 = 0;

    if (pattern < totalPatterns && child2PartialIndex >= 0) {
        sPartials2[patIdx][state] = partials2[state];
    } else {
        sPartials2[patIdx][state] = 0;
    }

    REAL sum2 = 0;


    KW_GLOBAL_VAR REAL* KW_RESTRICT matrix1 = matrices + child1TransIndex;
    KW_GLOBAL_VAR REAL* KW_RESTRICT matrix2 = matrices + child2TransIndex;
    for (int i = 0; i < PADDED_STATE_COUNT; i += BLOCK_PEELING_SIZE_SCA) {
        /* load one row of matrices */
        if (patIdx < BLOCK_PEELING_SIZE_SCA) {
            /* These are all coherent global memory reads. */
            sMatrix1[patIdx][state] = matrix1[patIdx * PADDED_STATE_COUNT + state];
            /* sMatrix now filled with starting in state and ending in i */
            matrix1 += BLOCK_PEELING_SIZE_SCA * PADDED_STATE_COUNT;
            if (!sameTransIndex) {
          		sMatrix2[patIdx][state] = matrix2[patIdx * PADDED_STATE_COUNT + state];
                matrix2 += BLOCK_PEELING_SIZE_SCA * PADDED_STATE_COUNT;
        	}
        }
        KW_LOCAL_FENCE;
        REAL (*secondMatrix)[PADDED_STATE_COUNT] = (sameTransIndex == 1) ? sMatrix1 : sMatrix2;
        	for(int j = 0; j < BLOCK_PEELING_SIZE_SCA; j++) {
                int k = (pattern < totalPatterns && child2PartialIndex >= 0) ? 0 : patIdx;
            	FMA(sMatrix1[j][state], sPartials1[patIdx][i + j], sum1);
                FMA(secondMatrix[j][state], sPartials2[k][i + j], sum2);
        	}

//            if (pattern < totalPatterns && child2PartialIndex >= 0) {
//                for(int j = 0; j < BLOCK_PEELING_SIZE_SCA; j++) {
//            		FMA(secondMatrix[j][state], sPartials2[i + j], sum2);
//        		}
//    		}

        KW_LOCAL_FENCE;
    }


    if (pattern < totalPatterns) {
        partials3[state] = sum1;
    }


    if (patIdx == 0) {
        popSizes[state] = sizes[state];
    }


	if (pattern < totalPatterns && child2PartialIndex >= 0) {
		accumulation1[state] = sum1;
		accumulation2[state] = sum2;
		if (popSizes[state] > 0) {
            partials3[state] = sum1 * sum2 / popSizes[state];
        } else {
            partials3[state] = 0;
        }
	    sPartials1[patIdx][state] = partials3[state];

        KW_LOCAL_FENCE;
        
#ifdef IS_POWER_OF_TWO
	    // parallelized reduction *** only works for powers-of-2 ****
	    for (int i = PADDED_STATE_COUNT / 2; i > 0; i >>= 1) {
	        if (state < i) {
#else
	    for (int i = SMALLEST_POWER_OF_TWO / 2; i > 0; i >>= 1) {
	        if (state < i && state + i < PADDED_STATE_COUNT) {
#endif // IS_POWER_OF_TWO
	            sPartials1[patIdx][state] += sPartials1[patIdx][state + i];
	        }
	        KW_LOCAL_FENCE;
	    }

		REAL denominator = sPartials1[patIdx][0];
		partials3[state] = partials3[state] / denominator;


		coalescent[intervalNumber] = denominator;
    }
}

//KW_GLOBAL_KERNEL void kernelBastaReduceWithinInterval(KW_GLOBAL_VAR REAL* e,
                                                    //KW_GLOBAL_VAR REAL*  f,
                                                    //KW_GLOBAL_VAR REAL*  g,
                                                    //KW_GLOBAL_VAR REAL*  h,
                                                    //KW_GLOBAL_VAR REAL* KW_RESTRICT startPartials1,
                                                    //KW_GLOBAL_VAR REAL* KW_RESTRICT startPartials2,
                                                    //KW_GLOBAL_VAR REAL* KW_RESTRICT endPartials1,
                                                    //KW_GLOBAL_VAR REAL* KW_RESTRICT endPartials2,
													//int intervalNumber,
                                                    //int child2PartialIndex,
                                                    //int renew) {

    //#define SUM_SITES_BLOCK_SIZE_B 64
	//#define SUM_PARTIAL_BLOCK_SIZE_B 4
    //int state = KW_LOCAL_ID_0;
    //int u = KW_LOCAL_ID_0 + KW_GROUP_ID_0 * SUM_SITES_BLOCK_SIZE_B;
    //int y = intervalNumber * PADDED_STATE_COUNT;
	//KW_LOCAL_MEM REAL sPartials1[SUM_PARTIAL_BLOCK_SIZE_B][PADDED_STATE_COUNT];

    //if (state < PADDED_STATE_COUNT) {
		//sPartials1[0][state] = startPartials1[u];
		//sPartials1[1][state] = startPartials1[u] * startPartials1[u];
		//sPartials1[2][state] = endPartials1[u];
		//sPartials1[3][state] = endPartials1[u] * endPartials1[u];
    //} else {
        //sPartials1[0][state] = 0;
		//sPartials1[1][state] = 0;
		//sPartials1[2][state] = 0;
		//sPartials1[3][state] = 0;
    //}

    //if (renew == 0) {
        //e[y + state] = sPartials1[0][state];
        //f[y + state] = sPartials1[1][state];
        //g[y + state] = sPartials1[2][state];
        //h[y + state] = sPartials1[3][state];
    //} else {
        //e[y + state] += sPartials1[0][state];
        //f[y + state] += sPartials1[1][state];
        //g[y + state] += sPartials1[2][state];
        //h[y + state] += sPartials1[3][state];
    //}

	//KW_LOCAL_MEM REAL sPartials2[SUM_PARTIAL_BLOCK_SIZE_B][PADDED_STATE_COUNT];
    //if (child2PartialIndex >= 0) {
    	//if (state < PADDED_STATE_COUNT) {
			//sPartials2[0][state] = startPartials2[u];
			//sPartials2[1][state] = startPartials2[u] * startPartials2[u];
			//sPartials2[2][state] = endPartials2[u];
			//sPartials2[3][state] = endPartials2[u] * endPartials2[u];
    	//} else {
        	//sPartials2[0][state] = 0;
			//sPartials2[1][state] = 0;
			//sPartials2[2][state] = 0;
			//sPartials2[3][state] = 0;
    	///}
	//e[y + state] += sPartials2[0][state];
	//f[y + state] += sPartials2[1][state];
	//g[y + state] += sPartials2[2][state];
	//h[y + state] += sPartials2[3][state];
	//}
//}




KW_GLOBAL_KERNEL void kernelBastaReduceWithinIntervalMerged(KW_GLOBAL_VAR int* KW_RESTRICT operations,
                                                    KW_GLOBAL_VAR REAL* KW_RESTRICT partials,
                                                    KW_GLOBAL_VAR REAL* dBastaMemory,
                                                    int numOps,
                                                    int start,
                                                    int end,
                                                    int numBlocks,
                                                    int kCoalescentBufferLength) {

#define OPS_PER_THREAD 8

    // Thread and block indices
    int state = KW_LOCAL_ID_0;
    int threadId = KW_LOCAL_ID_1;
    int blockY = KW_GROUP_ID_0;

    int halfBlocks = numBlocks / 2;
    int doEF = (blockY < halfBlocks) ? 1 : 0;

    if (!doEF) {blockY = blockY - halfBlocks;}
    int threadGlobalY = blockY * BASTA_SUM_INTERVAL_BLOCK_SIZE + threadId;
    int opStart = start + threadGlobalY * OPS_PER_THREAD;
    int opEnd = opStart + OPS_PER_THREAD;
    int opBlockStart = OPS_PER_THREAD * blockY * BASTA_SUM_INTERVAL_BLOCK_SIZE;
	int opBlockEnd = opBlockStart + OPS_PER_THREAD * BASTA_SUM_INTERVAL_BLOCK_SIZE;
    if (opEnd > end) opEnd = end;
	if (opBlockEnd > end) opBlockEnd = end;


    KW_GLOBAL_VAR REAL* e = dBastaMemory;
    KW_GLOBAL_VAR REAL* f = e + PADDED_STATE_COUNT * kCoalescentBufferLength;
    KW_GLOBAL_VAR REAL* g = f + PADDED_STATE_COUNT * kCoalescentBufferLength;
    KW_GLOBAL_VAR REAL* h = g + PADDED_STATE_COUNT * kCoalescentBufferLength;
// 	KW_LOCAL_MEM int shared_child1PartialIndex[BASTA_SUM_INTERVAL_BLOCK_SIZE * OPS_PER_THREAD];
// 	KW_LOCAL_MEM int shared_child2PartialIndex[BASTA_SUM_INTERVAL_BLOCK_SIZE * OPS_PER_THREAD];
// 	KW_LOCAL_MEM int shared_accumulation1PartialIndex[BASTA_SUM_INTERVAL_BLOCK_SIZE * OPS_PER_THREAD];
// 	KW_LOCAL_MEM int shared_accumulation2PartialIndex[BASTA_SUM_INTERVAL_BLOCK_SIZE * OPS_PER_THREAD];
// 	KW_LOCAL_MEM int shared_segmentKey[BASTA_SUM_INTERVAL_BLOCK_SIZE * OPS_PER_THREAD];
    int currentSegmentKey = -1;
    int carryOutSegmentKey = -1;
    REAL partialA = 0;
    REAL partialB = 0;


    int next_op = opStart;
    int nextSegmentKey = -1;
    REAL nextA_val1 = 0, nextB_val1 = 0;
    REAL nextA_val2 = 0, nextB_val2 = 0;



     if (state < PADDED_STATE_COUNT && next_op < opEnd) {
        int op = next_op;
		int child1PartialIndex = operations[op * numOps + 1];
		int child2PartialIndex = operations[op * numOps + 3];
		int accumulation1PartialIndex = operations[op * numOps + 5];
		int accumulation2PartialIndex = operations[op * numOps + 6];
		int segmentKey = operations[op * numOps + 7];


        KW_GLOBAL_VAR REAL* part1A = (doEF)? (partials + child1PartialIndex):(partials + accumulation1PartialIndex);

        KW_GLOBAL_VAR REAL* part2A = (doEF)? (partials + child2PartialIndex):(partials + accumulation2PartialIndex);

        REAL val1A = part1A[state];
        REAL val2A = 0;

        if (child2PartialIndex >= 0) {
            val2A = part2A[state];
        }

        nextA_val1 = val1A;
        nextB_val1 = val1A * val1A;
        nextA_val2 = val2A;
        nextB_val2 = val2A * val2A;
        nextSegmentKey = segmentKey;
    }

    for (int idx = opStart; idx < opEnd; ++idx) {
        REAL currA_val1 = nextA_val1;
        REAL currB_val1 = nextB_val1;
        REAL currA_val2 = nextA_val2;
        REAL currB_val2 = nextB_val2;

        int segmentKey = nextSegmentKey;

        next_op = idx + 1;
        if (state < PADDED_STATE_COUNT && next_op < opEnd) {
            int op = next_op;
			int child1PartialIndex = operations[op * numOps + 1];
			int child2PartialIndex = operations[op * numOps + 3];
			int accumulation1PartialIndex = operations[op * numOps + 5];
			int accumulation2PartialIndex = operations[op * numOps + 6];
			int segmentKeyNext = operations[op * numOps + 7];

        	KW_GLOBAL_VAR REAL* part1A = (doEF)? (partials + child1PartialIndex):(partials + accumulation1PartialIndex);

        	KW_GLOBAL_VAR REAL* part2A = (doEF)? (partials + child2PartialIndex):(partials + accumulation2PartialIndex);

        	REAL val1A = part1A[state];
        	REAL val2A = 0;

        	if (child2PartialIndex >= 0) {
            	val2A = part2A[state];
        	}

        	nextA_val1 = val1A;
        	nextB_val1 = val1A * val1A;
        	nextA_val2 = val2A;
        	nextB_val2 = val2A * val2A;

            nextSegmentKey = segmentKeyNext;
        } else {
            nextA_val1 = nextB_val1 = 0;
            nextA_val2 = nextB_val2 = 0;
            nextSegmentKey = -1;
        }

        int isNewSegment = (segmentKey != currentSegmentKey) ? 1 : 0;

        if (isNewSegment == 1 && idx != opStart) {
        	int w = currentSegmentKey * PADDED_STATE_COUNT + state;

                if (doEF) {
                    // partialA => e, partialB => f
                    atomicAdd(&e[w], partialA);
                    atomicAdd(&f[w], partialB);
                } else {
                    // partialA => g, partialB => h
                    atomicAdd(&g[w], partialA);
                    atomicAdd(&h[w], partialB);
                }

            partialA = 0;
            partialB = 0;
            }

        partialA += (currA_val1 + currA_val2);
        partialB += (currB_val1 + currB_val2);

        currentSegmentKey = segmentKey;
    }

	KW_LOCAL_FENCE;

	carryOutSegmentKey = currentSegmentKey;
    REAL carryOutA = partialA;
    REAL carryOutB = partialB;


    KW_LOCAL_MEM REAL sCarryOutA[BASTA_SUM_INTERVAL_BLOCK_SIZE][PADDED_STATE_COUNT];
    KW_LOCAL_MEM REAL sCarryOutB[BASTA_SUM_INTERVAL_BLOCK_SIZE][PADDED_STATE_COUNT];

	KW_LOCAL_MEM REAL sSegmentFlags[BASTA_SUM_INTERVAL_BLOCK_SIZE];
	KW_LOCAL_MEM REAL sCarryOutSegmentKeys[BASTA_SUM_INTERVAL_BLOCK_SIZE + 1];


    if (state < PADDED_STATE_COUNT) {
        sCarryOutA[threadId][state] = carryOutA;
        sCarryOutB[threadId][state] = carryOutB;
        sCarryOutSegmentKeys[threadId] = carryOutSegmentKey;
    }

	KW_LOCAL_FENCE;
	if (state == 0 && opStart < end) {
		if (threadId == 0) {
	    	sSegmentFlags[threadId] = 1;
		} else {
	    	int prevSegmentKey = sCarryOutSegmentKeys[threadId - 1];
	    	int currSegmentKey = sCarryOutSegmentKeys[threadId];

	    	if (currSegmentKey != prevSegmentKey) {
	        	sSegmentFlags[threadId] = 1;
	    	} else {
	        	sSegmentFlags[threadId] = 0;
	    	}
		}
	}

	KW_LOCAL_FENCE;

    int n = BASTA_SUM_INTERVAL_BLOCK_SIZE;
    for (int stride = 1; stride < n; stride *= 2) {
        int index = (threadId + 1) * 2 * stride - 1;
        if (index < n) {
            if (sSegmentFlags[index] == 0) {
                sCarryOutA[index][state] += sCarryOutA[index - stride][state];
                sCarryOutB[index][state] += sCarryOutB[index - stride][state];
                if (state == 0) {
                    sSegmentFlags[index] = sSegmentFlags[index - stride];
                }
            }
        }
        KW_LOCAL_FENCE;
    }

    for (int stride = n / 2; stride >= 1; stride /= 2) {
        int index = (threadId + 1) * 2 * stride - 1;
        if (index + stride < n) {
            if (sSegmentFlags[index + stride] == 0) {
                sCarryOutA[index + stride][state] += sCarryOutA[index][state];
                sCarryOutB[index + stride][state] += sCarryOutB[index][state];
                if (state == 0) {
                    sSegmentFlags[index + stride] = sSegmentFlags[index];
                }
            }
        }
        KW_LOCAL_FENCE;
    }


    if (threadId == BASTA_SUM_INTERVAL_BLOCK_SIZE - 1 || sCarryOutSegmentKeys[threadId] != sCarryOutSegmentKeys[threadId + 1]) {
        int reducedKey = sCarryOutSegmentKeys[threadId];
        if (reducedKey >= 0) {
    		int u = reducedKey * PADDED_STATE_COUNT + state;

            if (doEF) {
    			atomicAdd(&e[u], sCarryOutA[threadId][state]);
    			atomicAdd(&f[u], sCarryOutB[threadId][state]);
            } else {
    			atomicAdd(&g[u], sCarryOutA[threadId][state]);
    			atomicAdd(&h[u], sCarryOutB[threadId][state]);
            }
		}
    }
}


KW_GLOBAL_KERNEL void kernelBastaReduceAcrossInterval(KW_GLOBAL_VAR REAL* KW_RESTRICT dBastaMemory,
                                                    KW_GLOBAL_VAR REAL* KW_RESTRICT distance,
                                                    KW_GLOBAL_VAR REAL* KW_RESTRICT dLogL,
                                                    KW_GLOBAL_VAR REAL* KW_RESTRICT sizes,
                                                    KW_GLOBAL_VAR REAL* KW_RESTRICT coalescent,
													int intervalStartsCount,
													int kCoalescentBufferLength) {


        int intervalCount = intervalStartsCount - 1;
        int tid = KW_LOCAL_ID_0;
        int tidTotal = __umul24(KW_GROUP_ID_0, BASTA_SUM_ACROSS_BLOCK_SIZE * PADDED_STATE_COUNT) + tid;
        int state = tid % PADDED_STATE_COUNT;
        int intervalIdx = tid / PADDED_STATE_COUNT;
        int intervalNumber = __umul24(KW_GROUP_ID_0, BASTA_SUM_ACROSS_BLOCK_SIZE) + intervalIdx;
        int u = state + intervalNumber * PADDED_STATE_COUNT;

	    KW_GLOBAL_VAR REAL* e = dBastaMemory;
	    KW_GLOBAL_VAR REAL* f = e + PADDED_STATE_COUNT * kCoalescentBufferLength;
	    KW_GLOBAL_VAR REAL* g = f + PADDED_STATE_COUNT * kCoalescentBufferLength;
	    KW_GLOBAL_VAR REAL* h = g + PADDED_STATE_COUNT * kCoalescentBufferLength;

        KW_LOCAL_MEM REAL sPartials1[BASTA_SUM_ACROSS_BLOCK_SIZE * PADDED_STATE_COUNT];

        if (intervalNumber < intervalCount && (sizes[state] > 0)) {
            sPartials1[tid] = (e[u] * e[u] - f[u] +
                                 g[u] * g[u] - h[u]) * distance[intervalNumber] / sizes[state];
        } else {
            sPartials1[tid] = 0;
        }
        KW_LOCAL_FENCE;


        for (int i = BASTA_SUM_ACROSS_BLOCK_SIZE * PADDED_STATE_COUNT / 2; i > 0; i >>= 1) {
            if (tid < i) {
                sPartials1[tid] += sPartials1[tid + i];
            }
            KW_LOCAL_FENCE;
        }


        REAL temp = - sPartials1[0] / 4;

        if (tidTotal < intervalCount && (coalescent[tidTotal] > 0)) {
            sPartials1[tid] = log(coalescent[tidTotal]);
        } else {
            sPartials1[tid] = 0;
        }

        KW_LOCAL_FENCE;

        for (int i = BASTA_SUM_ACROSS_BLOCK_SIZE * PADDED_STATE_COUNT / 2; i > 0; i >>= 1) {
            if (tid < i) {
                sPartials1[tid] += sPartials1[tid + i];
            }
            KW_LOCAL_FENCE;
        }

        if (tid == 0) {
            temp = temp + sPartials1[0];
            dLogL[KW_GROUP_ID_0] = temp;
        }
    }


#ifdef CUDA
} // extern "C"
#endif //CUDA
